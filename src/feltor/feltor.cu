#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>

#include "draw/host_window.h"
//#include "draw/device_window.cuh"

#include "feltor.cuh"
#include "dg/rk.cuh"
#include "dg/timer.cuh"
#include "dg/karniadakis.cuh"
#include "file/read_input.h"
#include "parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/


int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v, v2;
    std::stringstream title;
    if( argc == 1)
    {
        try{
        v = file::read_input("input.txt");
        }catch( toefl::Message& m){m.display();}
    }
    else if( argc == 2)
    {
        v = file::read_input( argv[1]);
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }

    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow( v2[3], v2[4], "");
    draw::RenderHostData render(v2[1], v2[2]);
    /////////////////////////////////////////////////////////////////////////
    const Parameters p( v);
    p.display( std::cout);

    dg::Grid3d<double > grid( -p.a, p.a,  -p.a, p.a, 0, 2.*M_PI*p.a/p.eps_a, p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER);
    //create RHS 
    eule::Feltor< dg::DVec > feltor( grid, p); 
    eule::Rolkar< dg::DVec > rolkar( grid, p.nu_perp, p.nu_parallel);
    //create initial vector
    dg::Gaussian gauss( p.a - p.posX*p.thickness, 0., p.sigma, p.sigma, p.amp); //gaussian width is in absolute values
    eule::Gradient grad( p.a, p.thickness, p.lnn_inner);

    std::vector<dg::DVec> y0(3, dg::evaluate( gauss, grid)); // n_e' = gaussian
    std::vector<dg::DVec> y1(3, dg::evaluate( grad, grid)); 
    dg::blas1::axpby( 1., y1[0], 1., y0[0]);
    dg::blas1::axpby( 1., y1[1], 1., y0[1]);
    dg::blas1::axpby( 0., y1[2], 0., y0[2]); //set U = 0

    //dg::blas2::symv( feltor.gamma(), y0[0], y0[1]); // n_e = \Gamma_i n_i -> n_i = ( 1+alphaDelta) n_e' + 1
    //dg::blas2::symv( (dg::DVec)dg::create::v2d( grid), y0[1], y0[1]);

    feltor.log( y0, y0, 2); //transform to logarithmic values

    dg::Karniadakis< std::vector<dg::DVec> > ab( y0, y0[0].size(), p.eps_time);

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::HMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    dg::Timer t;
    double time = 0;
    //ab.init( feltor, rolkar, y0, p.dt);
    const double mass0 = feltor.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0, diff = 0;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    unsigned step = 0;
    while ( !glfwWindowShouldClose( w ))
    {
        //transform field to an equidistant grid
        feltor.exp( y0, y1, 2);
        thrust::transform( y1[0].begin(), y1[0].end(), dvisual.begin(), dg::PLUS<double>(-1));

        hvisual = dvisual;
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ne / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //transform phi
        //dg::blas2::gemv( rolkar.laplacianM(), feltor.potential()[0], y1[1]);
        hvisual = y0[2];
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"U_e / "<<colors.scale()<<"\t";
        title << std::fixed; 
        title << " &&   time = "<<time;
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        double x;
        std::cin >> x;
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass_blob0<<"\t";
            E0 = E1;
            E1 = feltor.energy();
            diff = (E1 - E0)/p.dt;
            double diss = feltor.energy_diffusion( );
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<"\n";

            try{ ab( feltor, rolkar, y0);}
            catch( eule::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
