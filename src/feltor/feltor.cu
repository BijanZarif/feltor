#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>

#include "draw/host_window.h"
//#include "draw/device_window.cuh"

#include "feltor.cuh"
#include "bessel.h"
#include "dg/rk.cuh"
#include "dg/timer.cuh"
#include "dg/karniadakis.cuh"
#include "file/read_input.h"
#include "parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/

struct Init
{
    /**
     * @brief 
     *
     * @param state 
     * @param R characteristic radius of dipole
     * @param vel_ratio u_drift/u_dipole
     * @param kz
     */
    Init( double x0, double y0, unsigned state, 
          double R,  double u_dipole, double kz):
        x0_(x0), y0_(y0), s_(state),  R_(R), u_d( u_dipole), kz_(kz){
        g_[0] = 3.83187; //groundstate with uuu=2
        g_[1] = 3.83235; //groundstate with uuu=-1
        g_[2] = 7.016;
        c_[0] = 0.5;
        c_[1] = -1;
        c_[2] = -1;
    }
    double operator()( double x, double y, double z)
    {
        double r = sqrt( (x-x0_)*(x-x0_)+(y-y0_)*(y-y0_));
        double theta = atan2( y-y0_, x-x0_);
        //std::cout << cos(theta)<<std::endl;
        double beta = sqrt(1-(c_[s_]));
        double norm = 1.2965125;
        

        if( r/R_<=1.)
            return u_d*(
                    r/R_*(1+beta*beta/g_[s_]/g_[s_]) 
                    - beta*beta/g_[s_]/g_[s_]*j1(g_[s_]*r/R_)/j1(g_[s_])
                    )*cos(theta)/norm*sin(kz_*z);
        return u_d*cos(theta)
                  *beta*beta/g_[s_]/g_[s_]
                  *bessk1(beta*r/R_)/bessk1(beta)/norm*sin(kz_*z);
    }
    private:
    double x0_, y0_;
    unsigned s_;
    double R_, c_[3], u_d;
    double g_[3];
    double kz_;
};

int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v, v2;
    std::stringstream title;
    if( argc == 1)
    {
        try{
        v = file::read_input("input.txt");
        }catch( toefl::Message& m){m.display();}
    }
    else if( argc == 2)
    {
        v = file::read_input( argv[1]);
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }

    /////////////////////////////////////////////////////////////////////////
    const Parameters p( v);
    p.display( std::cout);
    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow( p.Nz/v2[2]*v2[3], v2[1]*v2[4], "");
    draw::RenderHostData render(v2[1], p.Nz/v2[2]);

    dg::Grid3d<double > grid( -p.a*(1+1e-1), p.a*(1+1e-1),  -p.a*(1+1e-1), p.a*(1+1e-1), 0, 1., p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER);
    //create RHS 
    eule::Feltor< dg::DVec > feltor( grid, p); 
    eule::Rolkar< dg::DVec > rolkar( grid, p.nu_perp, p.nu_parallel, p.a, p.thickness, p.mu_e*4.*M_PI*M_PI*p.R_0*p.R_0);
    //create initial vector
    Init init0( 0., p.a - p.posX*p.thickness, 2, p.sigma, p.amp ,2.*M_PI*p.m_par); //gaussian width is in absolute values
    Init init1( 0., -p.a + p.posX*p.thickness, 2, p.sigma, p.amp ,2.*M_PI*p.m_par); //gaussian width is in absolute values
    Init init2( p.a - p.posX*p.thickness, 0., 2, p.sigma, p.amp ,2.*M_PI*p.m_par); //gaussian width is in absolute values
    Init init3( -p.a + p.posX*p.thickness, 0., 2, p.sigma, p.amp ,2.*M_PI*p.m_par); //gaussian width is in absolute values
    eule::Gradient grad( p.a, p.thickness, p.lnn_inner);

    const dg::HVec gradient( dg::evaluate(grad, grid));
    std::vector<dg::DVec> y0(3, dg::evaluate( init0, grid)); // n_e' = gaussian
    std::vector<dg::DVec> y1(3, dg::evaluate( grad, grid)); 
    dg::blas1::axpby( 1., y1[0], 1., y0[0]);
    dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init1, grid), 1., y0[0]);
    dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init2, grid), 1., y0[0]);
    dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init3, grid), 1., y0[0]);
    dg::blas1::axpby( 1., y1[1], 0., y0[1]);
    dg::blas1::axpby( 0., y1[2], 0., y0[2]); //set U = 0

    //dg::blas2::symv( feltor.gamma(), y0[0], y0[1]); // n_e = \Gamma_i n_i -> n_i = ( 1+alphaDelta) n_e' + 1
    //dg::blas2::symv( (dg::DVec)dg::create::v2d( grid), y0[1], y0[1]);

    feltor.log( y0, y0, 2); //transform to logarithmic values

    dg::Karniadakis< std::vector<dg::DVec> > ab( y0, y0[0].size(), p.eps_time);

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::HMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    dg::Timer t;
    double time = 0;
    
    ab.init( feltor, rolkar, y0, p.dt);
    const double mass0 = feltor.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0, diff = 0;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    unsigned step = 0;
    while ( !glfwWindowShouldClose( w ))
    {
        //transform field to an equidistant grid
        feltor.exp( y0, y1, 2);
        //thrust::transform( y1[0].begin(), y1[0].end(), dvisual.begin(), dg::PLUS<double>(-1));

        hvisual = y1[0];
        dg::blas1::axpby( -1., gradient, 1., hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ne / "<<colors.scale()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }

        thrust::transform( y1[1].begin(), y1[1].end(), dvisual.begin(), dg::PLUS<double>(-1));
        hvisual = dvisual;
        //dg::HVec iris = dg::evaluate( eule::Iris( p.a, p.thickness), grid);
        //dg::blas1::pointwiseDot( iris, hvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ni / "<<colors.scale()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }

        //transform phi
        dg::blas2::gemv( rolkar.laplacianM(), feltor.potential()[0], y1[1]);
        hvisual = y1[1];
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        title <<"phi / "<<colors.scale()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        hvisual = y0[2];
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"Ue / "<<colors.scale()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }


        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        //double x;
        //std::cin >> x;
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass_blob0<<"\t";
            E0 = E1;
            E1 = feltor.energy();
            diff = (E1 - E0)/p.dt;
            double diss = feltor.energy_diffusion( );
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<"\n";

            try{ ab( feltor, rolkar, y0);}
            catch( eule::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
