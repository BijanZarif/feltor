#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG



#include "dg/backend/timer.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/interpolation.cuh"
#include "file/read_input.h"
#include "file/nc_utilities.h"


#include "feltor.cuh"
#include "parameters.h"
#include "geometry.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

const unsigned k = 3;//!< a change in k needs a recompilation

int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v,v3;
    std::string input, geom;
    if( argc != 4)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        v = file::read_input( argv[1]);
        input = file::read_file( argv[1]);
    }
    const eule::Parameters p( v);
    p.display( std::cout);

    ////////////////////////////////set up computations///////////////////////////
    try{ v3 = file::read_input( argv[2]); }
    catch (toefl::Message& m) {  m.display(); 
        geom = file::read_file( argv[2]);
        std::cout << geom << std::endl;
        for( unsigned i = 0; i<v.size(); i++)
        return -1;
    }

    const solovev::GeomParameters gp(v3);
    gp.display( std::cout);
    double Rmin=gp.R_0-(gp.boxscale)*gp.a;
    double Zmin=-(gp.boxscale)*gp.a*gp.elongation;
    double Rmax=gp.R_0+(gp.boxscale)*gp.a; 
    double Zmax=(gp.boxscale)*gp.a*gp.elongation;
    //Make grids
     dg::Grid3d<double > grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER, dg::cylindrical);  
     dg::Grid3d<double > grid_out( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n_out, p.Nx_out, p.Ny_out, p.Nz_out, dg::DIR, dg::DIR, dg::PER, dg::cylindrical);  
     
    //create RHS 
    eule::Feltor<dg::DMatrix, dg::DVec, dg::DVec > feltor( grid, p,gp); 
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p,gp);

    /////////////////////The initial field///////////////////////////////////////////
      //dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    //dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
    //solovev::ZonalFlow init0(gp,p.amp);
    solovev::Nprofile grad(gp); //initial background profile
    
    std::vector<dg::DVec> y0(4, dg::evaluate( grad, grid)), y1(y0); 

    dg::Gaussian gaussian( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
    dg::GaussianZ gaussianZ( M_PI, M_PI, 1);
    y1[1] = feltor.dz().evaluate( gaussian);
    y1[2] = dg::evaluate( gaussianZ, grid);
    dg::blas1::pointwiseDot( y1[1], y1[2], y1[1]);

    y1[1] = dg::evaluate( init0, grid);
    //damp the bath on psi boundaries 
    dg::blas1::pointwiseDot(rolkar.damping(),y1[1], y1[1]);  
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni
    feltor.initializene(y0[1],y0[0]);    

    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0
    
    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( feltor, rolkar, y0, p.dt);
    double time = 0;
    unsigned step = 0;

    /////////////////////////////set up netcdf//////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[3], NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    err = nc_put_att_text( ncid, NC_GLOBAL, "geomfile", geom.size(), geom.data());
    int dim_ids[4], tvarID;
    err = file::define_dimensions( ncid, dim_ids, &tvarID, grid_out);
    solovev::FieldR fieldR(gp);
    solovev::FieldZ fieldZ(gp);
    solovev::FieldP fieldP(gp);
    dg::HVec vecR = dg::evaluate( fieldR, grid_out);
    dg::HVec vecZ = dg::evaluate( fieldZ, grid_out);
    dg::HVec vecP = dg::evaluate( fieldP, grid_out);
    int vecID[3];
    err = nc_def_var( ncid, "BR", NC_DOUBLE, 3, &dim_ids[1], &vecID[0]);
    err = nc_def_var( ncid, "BZ", NC_DOUBLE, 3, &dim_ids[1], &vecID[1]);
    err = nc_def_var( ncid, "BP", NC_DOUBLE, 3, &dim_ids[1], &vecID[2]);
    err = nc_enddef( ncid);
    err = nc_put_var_double( ncid, vecID[0], vecR.data());
    err = nc_put_var_double( ncid, vecID[1], vecZ.data());
    err = nc_put_var_double( ncid, vecID[2], vecP.data());
    err = nc_redef(ncid);

    std::string names[5] = {"electrons", "ions", "Ue", "Ui", "potential"}; 
    int dataIDs[5];
    for( unsigned i=0; i<5; i++){
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[i]);}
    err = nc_def_var( ncid, "energy", NC_DOUBLE, 1, dim_ids, &dataIDs[5]);
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    size_t count[4] = {1., grid_out.Nz(), grid_out.n()*grid_out.Ny(), grid_out.n()*grid_out.Nx()};
    size_t start[4] = {0, 0, 0, 0};
    dg::DVec transfer(  dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::DMatrix interpolate = dg::create::interpolation( grid_out, grid); 
    for( unsigned i=0; i<4; i++)
    {
        dg::blas2::symv( interpolate, y0[i], transferD);
        transferH = transferD;//transfer to host
        err = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    transfer = feltor.potential()[0];
    dg::blas2::symv( interpolate, transfer, transferD);
    transferH = transferD;//transfer to host
    err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
    err = nc_put_vara_double( ncid, tvarID, start, count, &time);

    double E0 = feltor.energy(), energy0 = E0, E1 = 1, diff = 0;
    err = nc_put_vara_double( ncid, dataIDs[5], start, count,&E1);
    err = nc_close(ncid);

    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Timer t;
    t.tic();
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=0; i<p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                return -1;
            }
        }
        time += p.itstp*p.dt;
        start[0] = i;
        err = nc_open(argv[3], NC_WRITE, &ncid);

        for( unsigned j=0; j<4; j++)
        {
            dg::blas2::symv( interpolate, y0[j], transferD);
            transferH = transferD;//transfer to host
            err = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data());
        }
        transfer = feltor.potential()[0];
        dg::blas2::symv( interpolate, transfer, transferD);
        transferH = transferD;//transfer to host
        err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
        //write time data
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
        E1 = feltor.energy()/energy0;
        err = nc_put_vara_double( ncid, dataIDs[5], start, count,&E1);

        err = nc_close(ncid);
#ifdef DG_BENCHMARK
        ti.toc();
        step+=p.itstp;
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}

