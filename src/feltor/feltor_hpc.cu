#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

// #include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/timer.cuh"
#include "dg/backend/xspacelib.cuh"
#include "file/read_input.h"
#include "file/file.h"


#include "feltor.cuh"
#include "bessel.h"
#include "parameters.h"
#include "geometry.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

const unsigned k = 3;//!< a change in k needs a recompilation

int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v,v3;
    std::string input;
    if( argc != 3)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        v = file::read_input( argv[1]);
        input = file::read_file( argv[1]);
    }
    const Parameters p( v);
    p.display( std::cout);

    ////////////////////////////////set up computations///////////////////////////
    try{ v3 = file::read_input( "geometry_params.txt"); }
    catch (toefl::Message& m) {  
        m.display(); 
        for( unsigned i = 0; i<v.size(); i++)
//             std::cout << v3[i] << " ";
//             std::cout << std::endl;
        return -1;}

     const solovev::GeomParameters gp(v3);
    gp.display( std::cout);
    double Rmin=gp.R_0-(gp.boxscale)*gp.a;
    double Zmin=-(gp.boxscale)*gp.a*gp.elongation;
    double Rmax=gp.R_0+(gp.boxscale)*gp.a; 
    double Zmax=(gp.boxscale)*gp.a*gp.elongation;
    //Make grid
     dg::Grid3d<double > grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER);  
     
    //create RHS 
    eule::Feltor< dg::DVec > feltor( grid, p,gp); 
    eule::Rolkar< dg::DVec > rolkar( grid, p,gp);

    
    //with bath
      dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,15.,p.amp);
     //with zonal flow field
//       solovev::ZonalFlow init0(gp,p.amp);
    //with gaussians
//     dg::Gaussian3d init0( p.R_0, p.posY*p.a,    M_PI, p.sigma, p.sigma, M_PI/8.*p.m_par, p.amp );     
//     dg::Gaussian3d init1( p.R_0, -p.a*p.posY,   M_PI, p.sigma, p.sigma, M_PI/8.*p.m_par, p.amp ); 
//     dg::Gaussian3d init2( p.R_0+p.posX*p.a, 0., M_PI, p.sigma, p.sigma, M_PI/8.*p.m_par, p.amp ); 
//     dg::Gaussian3d init3( p.R_0-p.a*p.posX, 0., M_PI, p.sigma, p.sigma, M_PI/8.*p.m_par, p.amp ); 
    
//     solovev::Gradient grad(gp); //background gradient
    solovev::Nprofile grad(gp); //initial profile


    std::vector<dg::DVec> y0(4, dg::evaluate( grad, grid)), y1(y0); 

    dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init0, grid), 1., y0[0]);
    
//     dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init1, grid), 1., y0[0]);
//     dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init2, grid), 1., y0[0]);
//     dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init3, grid), 1., y0[0]);
   
    dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init0, grid), 1., y0[1]);
   
//     dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init1, grid), 1., y0[1]);
//     dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init2, grid), 1., y0[1]);
//     dg::blas1::axpby( 1., (dg::DVec)dg::evaluate(init3, grid), 1., y0[1]);

    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set U = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set U = 0

    feltor.log( y0, y0, 2); //transform to logarithmic values (ne and ni)
    
    dg::blas1::pointwiseDot(rolkar.iris(),y0[0],y0[0]); //is pupil on bath
    dg::blas1::pointwiseDot(rolkar.iris(),y0[1],y0[1]); //is pupil on bath
    
    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( feltor, rolkar, y0, p.dt);
    double time = 0;
    unsigned step = 0;

    /////////////////////////////set up hdf5/////////////////////////////////
    file::T5trunc t5file( argv[2], input);
    std::vector<std::string> names(5); 
    names[0] = "electrons", names[1] = "ions", names[2] = "Ue", names[3] = "Ui";
    names[4] = "potential";
    std::vector<unsigned> dims( 3);
    dims[0] = grid.Nz(), dims[1] = grid.n()*grid.Ny(), dims[2] = grid.n()*grid.Nx();
    std::vector<dg::HVec> output(5); 
    ///////////////////////////////////first output/////////////////////////
    feltor.exp( y0,y0,2); //transform to correct values
    for( unsigned i=0; i<4; i++)
        output[i] = y0[i];
    output[4] = feltor.potential()[0];
    t5file.write( output, names, dims, time );
    t5file.append( feltor.mass(), feltor.mass_diffusion(), feltor.energy(), feltor.energy_diffusion());
    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Timer t;
    t.tic();
    try
    {
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=0; i<p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                break;
            }
            t5file.append( feltor.mass(), feltor.mass_diffusion(), feltor.energy(), feltor.energy_diffusion());
        }
        time += p.itstp*p.dt;
        feltor.exp( y0,y0,2); //transform to correct values
        for( unsigned i=0; i<4; i++)
            output[i] = y0[i];
        output[4] = feltor.potential()[0];
        t5file.write( output, names, dims, time );
#ifdef DG_BENCHMARK
        ti.toc();
        step+=p.itstp;
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
    }
    }
    catch( dg::Fail& fail) { 
        std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
        std::cerr << "Does Simulation respect CFL condition?\n";
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}

