#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG



#include "dg/backend/timer.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/interpolation.cuh"
#include "file/read_input.h"
#include "file/nc_utilities.h"
#include "solovev/geometry.h"

#include "feltor.cuh"
#include "parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

const unsigned k = 3;//!< a change in k needs a recompilation

int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v,v3;
    std::string input, geom;
    if( argc != 4)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        v = file::read_input( argv[1]);
        input = file::read_file( argv[1]);
        
        v3 = file::read_input( argv[2]); 
        geom = file::read_file( argv[2]);
        std::cout << geom << std::endl;
    }
    const eule::Parameters p( v);
    p.display( std::cout);
    const solovev::GeomParameters gp(v3);
    gp.display( std::cout);
    ////////////////////////////////set up computations///////////////////////////
    double Rmin=gp.R_0-p.boxscale*gp.a;
    double Zmin=-p.boxscale*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscale*gp.a; 
    double Zmax=p.boxscale*gp.a*gp.elongation;
    //Make grids
    dg::Grid3d<double > grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER, dg::cylindrical);  
    dg::Grid3d<double > grid_out( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n_out, p.Nx_out, p.Ny_out, p.Nz_out, dg::DIR, dg::DIR, dg::PER, dg::cylindrical);  
     
    //create RHS 
    eule::Feltor<dg::DMatrix, dg::DVec, dg::DVec > feltor( grid, p,gp); 
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p,gp);

/////////////////////The initial field///////////////////////////////////////////
    //initial perturbation
    //dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
//     dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
 solovev::ZonalFlow init0(p, gp);
    //background profile
    solovev::Nprofile grad(p, gp); //initial background profile
    
    std::vector<dg::DVec> y0(4, dg::evaluate( grad, grid)), y1(y0); 
    //For field alongated perturbation
    //dg::CONSTANT gaussianZ( 1.);
//     dg::GaussianZ gaussianZ( M_PI, p.sigma_z, 1);
//     y1[1] = feltor.dz().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 2);
//     dg::blas1::pointwiseDot( y1[1], y1[2], y1[1]);

    y1[1] = dg::evaluate( init0, grid);
    
    //damp initialni on boundaries psimax
    dg::blas1::pointwiseDot(rolkar.damping(),y1[1], y1[1]); 
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni
    dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-1));
    feltor.initializene( y0[1], y0[0]);    
    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0
    
    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( feltor, rolkar, y0, p.dt);
    double time = 0;
    /////////////////////////////set up netcdf//////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[3],NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    err = nc_put_att_text( ncid, NC_GLOBAL, "geomfile", geom.size(), geom.data());
    int dim_ids[4], tvarID;
    err = file::define_dimensions( ncid, dim_ids, &tvarID, grid_out);
    solovev::FieldR fieldR(gp);
    solovev::FieldZ fieldZ(gp);
    solovev::FieldP fieldP(gp);
    dg::HVec vecR = dg::evaluate( fieldR, grid_out);
    dg::HVec vecZ = dg::evaluate( fieldZ, grid_out);
    dg::HVec vecP = dg::evaluate( fieldP, grid_out);
    int vecID[3];
    err = nc_def_var( ncid, "BR", NC_DOUBLE, 3, &dim_ids[1], &vecID[0]);
    err = nc_def_var( ncid, "BZ", NC_DOUBLE, 3, &dim_ids[1], &vecID[1]);
    err = nc_def_var( ncid, "BP", NC_DOUBLE, 3, &dim_ids[1], &vecID[2]);
    err = nc_enddef( ncid);
    err = nc_put_var_double( ncid, vecID[0], vecR.data());
    err = nc_put_var_double( ncid, vecID[1], vecZ.data());
    err = nc_put_var_double( ncid, vecID[2], vecP.data());
    err = nc_redef(ncid);

    std::string names[5] = {"electrons", "ions", "Ue", "Ui", "potential"}; 
    int dataIDs[5], energyID;
    for( unsigned i=0; i<5; i++){
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[i]);}
    err = nc_def_var( ncid, "energy", NC_DOUBLE, 1, dim_ids, &energyID);
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    size_t count[4] = {1., grid_out.Nz(), grid_out.n()*grid_out.Ny(), grid_out.n()*grid_out.Nx()};
    size_t start[4] = {0, 0, 0, 0};
    dg::DVec transfer(  dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::DMatrix interpolate = dg::create::interpolation( grid_out, grid); 
    for( unsigned i=0; i<4; i++)
    {
        dg::blas2::symv( interpolate, y0[i], transferD);
        transferH = transferD;//transfer to host
        err = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    transfer = feltor.potential()[0];
    dg::blas2::symv( interpolate, transfer, transferD);
    transferH = transferD;//transfer to host
    err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
    err = nc_put_vara_double( ncid, tvarID, start, count, &time);

    double E0 = feltor.energy(), energy0 = E0, E1 = 1, diff = 0;
    err = nc_put_vara_double( ncid, energyID, start, count,&E1);
    err = nc_close(ncid);

    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Timer t;
    t.tic();
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=0; i<p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                return -1;
            }
        }
        time += p.itstp*p.dt;
#ifdef DG_BENCHMARK
        ti.toc();
        step+=p.itstp;
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK

        start[0] = i;
        err = nc_open(argv[3], NC_WRITE, &ncid);
        for( unsigned j=0; j<4; j++)
        {
            dg::blas2::symv( interpolate, y0[j], transferD);
            transferH = transferD;//transfer to host
            err = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data());
        }
        transfer = feltor.potential()[0];
        dg::blas2::symv( interpolate, transfer, transferD);
        transferH = transferD;//transfer to host
        err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
        //write time data
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
        E1 = feltor.energy()/energy0;
        err = nc_put_vara_double( ncid, energyID, start, count,&E1);

        err = nc_close(ncid);
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}

