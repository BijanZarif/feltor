#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG



#include "dg/backend/timer.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/interpolation.cuh"
#include "file/read_input.h"
#include "file/nc_utilities.h"


#include "feltor.cuh"
#include "bessel.h"
#include "parameters.h"
#include "geometry.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

const unsigned k = 3;//!< a change in k needs a recompilation

int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v,v3;
    std::string input, geom;
    if( argc != 4)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        v = file::read_input( argv[1]);
        input = file::read_file( argv[1]);
    }
    const eule::Parameters p( v);
    p.display( std::cout);

    ////////////////////////////////set up computations///////////////////////////
    try{ v3 = file::read_input( argv[2]); }
    catch (toefl::Message& m) {  m.display(); 
        geom = file::read_file( argv[2]);
        std::cout << geom << std::endl;
        for( unsigned i = 0; i<v.size(); i++)
        return -1;
    }

    const solovev::GeomParameters gp(v3);
    gp.display( std::cout);
    double Rmin=gp.R_0-(gp.boxscale)*gp.a;
    double Zmin=-(gp.boxscale)*gp.a*gp.elongation;
    double Rmax=gp.R_0+(gp.boxscale)*gp.a; 
    double Zmax=(gp.boxscale)*gp.a*gp.elongation;
    //Make grids
     dg::Grid3d<double > grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER, dg::cylindrical);  
     dg::Grid3d<double > grid_out( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n_out, p.Nx_out, p.Ny_out, p.Nz_out, dg::DIR, dg::DIR, dg::PER, dg::cylindrical);  
     
    //create RHS 
    eule::Feltor<dg::DMatrix, dg::DVec, dg::DVec > feltor( grid, p,gp); 
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p,gp);

    //The initial field
    dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI/p.Nz, p.sigma, p.sigma, p.sigma, p.amp);
//     dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
//       solovev::ZonalFlow init0(gp,p.amp);
    solovev::Nprofile grad(gp); //initial profile
    
    std::vector<dg::DVec> y0(4, dg::evaluate( grad, grid)), y1(y0); 
    //damp the bath on psi boundaries 
    dg::blas1::pointwiseDot(rolkar.dampin(),(dg::DVec)dg::evaluate(init0, grid), y1[1]); //is damping on bath    
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ne
    //without FLR
    //dg::blas1::axpby( 1., y1[0], 1., y0[1]);
    //with FLR
    feltor.initializene(y0[1],y0[0]);    
    feltor.log( y0, y0, 2); 

    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0
    
    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( feltor, rolkar, y0, p.dt);
    double time = 0;
    unsigned step = 0;

    /////////////////////////////set up netcdf//////////////////////////////
    file::NC_Error_Handle h;
    int ncid;
    h = nc_create( argv[2], NC_CLOBBER, &ncid);
    h = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    h = nc_put_att_text( ncid, NC_GLOBAL, "geomfile", geom.size(), geom.data());
    int dim_ids[4], tvarID;
    h = file::define_dimensions( ncid, dim_ids, &tvarID, grid_out);

    std::vector<std::string> names(6); 
    int dataIDs[names.size()];
    names[0] = "electrons", names[1] = "ions", names[2] = "Ue", names[3] = "Ui";
    names[4] = "potential"; 
    names[5] = "energy";
    for( unsigned i=0; i<names.size()-1; i++){
        h = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[i]);}
    nc_def_var( ncid, names[5].data(), NC_DOUBLE, 1, dim_ids, &dataIDs[5]);
    h = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    size_t count[4] = {1., grid.Nz(), grid.n()*grid.Ny(), grid.n()*grid.Nx()};
    size_t start[4] = {0, 0, 0, 0};
    dg::DVec transfer(  dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::DMatrix interpolate = dg::create::interpolation( grid_out, grid); 
    feltor.exp( y0,y0,2); //transform to correct values
    for( unsigned i=0; i<4; i++)
    {
        dg::blas2::symv( interpolate, y0[i], transferD);
        transferH = transferD;//transfer to host
        h = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    transfer = feltor.potential()[0];
    dg::blas2::symv( interpolate, transfer, transferD);
    transferH = transferD;//transfer to host
    h = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
    h = nc_put_vara_double( ncid, tvarID, start, count, &time);
    h = nc_close(ncid);
    ///////////////////////////////////////Timeloop/////////////////////////////////
    double E0 = feltor.energy(), energy0 = E0, E1 = 0, diff = 0;

    dg::Timer t;
    t.tic();
    try
    {
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=0; i<p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                break;
            }
        }
        time += p.itstp*p.dt;
        start[0] = i;
        feltor.exp( y0,y0,2); //transform to correct values
        h = nc_open(argv[2], NC_WRITE, &ncid);

        for( unsigned j=0; j<4; j++)
        {
            dg::blas2::symv( interpolate, y0[j], transferD);
            transferH = transferD;//transfer to host
            h = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data());
        }
        transfer = feltor.potential()[0];
        dg::blas2::symv( interpolate, transfer, transferD);
        transferH = transferD;//transfer to host
        h = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
        //write time data
        h = nc_put_vara_double( ncid, tvarID, start, count, &time);
        E1 = feltor.energy()/energy0;
        h = nc_put_vara_double( ncid, dataIDs[5], start, count,&E1);

        h = nc_close(ncid);
#ifdef DG_BENCHMARK
        ti.toc();
        step+=p.itstp;
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
    }
    }
    catch( dg::Fail& fail) { 
        std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
        std::cerr << "Does Simulation respect CFL condition?\n";
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}

