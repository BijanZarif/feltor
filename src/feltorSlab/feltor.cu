#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"
#include "file/read_input.h"

#include "feltor.cuh"
#include "parameters.h"

#define TORLIM //for toroidal limiter setup
// #define TORSHEATHLIM //for toroidal sheath limiter setup (under construction)
/*
   - reads parameters from input.txt or any other given file, 
   - integrates the Feltor - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/


int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v2;
    std::stringstream title;
    if( argc == 1)
    {
        try{
            v = file::read_input("input.txt");
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else if( argc == 2)
    {
        try{
            v = file::read_input(argv[1]);
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const eule::Parameters p( v);
    p.display( std::cout);

    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow(  v2[2]*v2[3]*p.lx/p.ly, v2[1]*v2[4], "");
    draw::RenderHostData render( v2[1], v2[2]);



    //////////////////////////////////////////////////////////////////////////

    //Make grid
     dg::Grid2d<double > grid( 0., p.lx, 0.,p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);  
    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::DMatrix, dg::DVec, dg::DVec > feltor( grid, p); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //initial perturbation
    //dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Gaussian init0( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp);
//     dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
//     solovev::ZonalFlow init0(p, gp);
//     dg::CONSTANT init0( 0.);
    
    //background profile
//     solovev::Nprofile prof(p, gp); //initial background profile
//     dg::CONSTANT prof(p.bgprofamp );
    //
//     dg::LinearX prof(-p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);
    dg::SinProfX prof(p.nprofileamp, p.bgprofamp,M_PI/(2.*p.lx));
//     const dg::DVec prof =  dg::LinearX( -p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);

    std::vector<dg::DVec> y0(2, dg::evaluate( prof, grid)), y1(y0); 
    

    //no field aligning
    y1[1] = dg::evaluate( init0, grid);
    
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni
    dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); //initialize ni-1
//     dg::blas1::pointwiseDot(rolkar.damping(),y0[1], y0[1]); //damp with gaussprofdamp
    std::cout << "intiialize ne" << std::endl;
    feltor.initializene( y0[1], y0[0]);    
    std::cout << "Done!\n";


    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    std::cout << "intiialize karniadakis" << std::endl;
    karniadakis.init( feltor, rolkar, y0, p.dt);
    std::cout << "Done!\n";
    feltor.energies( y0);//now energies and potential are at time 0

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual),avisual(hvisual);
    dg::HMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    
    const double mass0 = feltor.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0., diff = 0.;
    
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
   
    while ( !glfwWindowShouldClose( w ))
    {

        hvisual = y0[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), (double)-1e14, thrust::maximum<double>() );
//         colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() = 1.0;
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ne / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ne-1 / " << colors.scalemin()<<"\t";

        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //draw ions
        //thrust::transform( y1[1].begin(), y1[1].end(), dvisual.begin(), dg::PLUS<double>(-0.));//ne-1
        hvisual = y0[1];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        //colors.scalemin() = 1.0;        
//         colors.scalemin() = -colors.scalemax();        
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ni / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ni-1 / " << colors.scalemin()<<"\t";

        render.renderQuad(visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        
        //draw potential
        //transform to Vor
//        dvisual=feltor.potential()[0];
//        dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
//        hvisual = y1[1];
         hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );

//         //colors.scalemin() = 1.0;        
//          colors.scalemin() = -colors.scalemax();        
//          colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"Potential / "<< colors.scalemax() << " " << colors.scalemin()<<"\t";

        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw potential
        //transform to Vor
        dvisual=feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
        hvisual = y1[1];
         //hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        //colors.scalemin() = 1.0;        
         colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"Omega / "<< colors.scalemax()<<"\t";

        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);


     
           
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
            step++;
            feltor.energies( y0); //update energetics
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass_blob0<<"\t";
            E1 = feltor.energy();
            diff = (E1 - E0)/p.dt; //
            double diss = feltor.energy_diffusion( );
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<
            " d E/dt = " << diff <<" Lambda =" << diss << "\n";
            
            E0 = E1;

        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
