#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>

#include "draw/host_window.h"

#include "esel.cuh"
#include "dg/rk.cuh"
#include "dg/timer.cuh"
#include "file/read_input.h"
#include "../galerkin/parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/

const unsigned k = 3; //!< a change of k needs a recompilation!

int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v, v2;
    if( argc == 1)
    {
        v = file::read_input("input.txt");
    }
    else if( argc == 2)
    {
        v = file::read_input( argv[1]);
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }

    v2 = file::read_input( "window_params.txt");
    draw::HostWindow w(v2[3], v2[4]);
    w.set_multiplot( v2[1], v2[2]);
    /////////////////////////////////////////////////////////////////////////
    const Parameters p( v);
    p.display( std::cout);
    if( p.k != k)
    {
        std::cerr << "ERROR: k doesn't match: "<<k<<" vs. "<<p.k<<"\n";
        return -1;
    }

    dg::Grid<double > grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    dg::SOL sol( v[22], v[23], v[24], v[25]);
    //create RHS 
    dg::Esel< dg::DVec > test( grid, p.kappa, p.nu, p.tau, p.eps_pol, p.eps_gamma, sol); 
    //create initial vector
    dg::EXPX<double> exp( 1., -1./v[26]); 
    dg::Gaussian gaussian( p.posX*grid.lx(), p.posY*grid.ly(), p.sigma, p.sigma, p.n0); //gaussian width is in absolute values
    std::vector<dg::DVec> y0(2, dg::evaluate( exp, grid)), y1(y0); 
    std::vector<dg::DVec> y0p(2, dg::evaluate( gaussian, grid)); 
    dg::blas1::axpby( 1, y0p, 1, y0);

    //dg::blas2::symv( test.gamma(), y0[0], y0[1]); // n_e = \Gamma_i n_i -> n_i = ( 1+alphaDelta) n_e' + 1
    dg::blas2::symv( (dg::DVec)dg::create::v2d( grid), y0[1], y0[1]);
    assert( p.global);
    assert( p.bc_x == dg::DIR_NEU);

    test.log( y0, y0); //transform to logarithmic values

    dg::AB< k, std::vector<dg::DVec> > ab( y0);
    //dg::TVB< std::vector<dg::DVec> > ab( y0);

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::HMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    dg::Timer t;
    bool running = true;
    double time = 0;
    ab.init( test, y0, p.dt);
    ab( test, y0, y1, p.dt);
    y0.swap( y1); 
    const double mass0 = test.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = test.energy(), energy0 = E0, E1 = 0, diff = 0;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    unsigned step = 0;
    while (running)
    {
        //transform field to an equidistant grid
        {
            //test.exp( y1, y1); //plot logarithmic values
            thrust::transform( y1[0].begin(), y1[0].end(), dvisual.begin(), dg::PLUS<double>(1));
        }

        hvisual = dvisual;
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        w.title() << std::setprecision(2) << std::scientific;
        w.title() <<"ne / "<<colors.scale()<<"\t";
        w.draw( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //transform phi
        //dg::blas2::gemv( test.laplacianM(), test.potential()[0], y1[1]);
        dg::blas1::axpby( 1., test.potential()[0], 0, y1[1]);
        hvisual = y1[1];
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        w.title() <<"omega / "<<colors.scale()<<"\t";
        w.title() << std::fixed; 
        w.title() << " &&   time = "<<time;
        w.draw( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            if( p.global)
            {
                std::cout << "(m_tot-m_0)/m_0: "<< (test.mass()-mass0)/mass_blob0<<"\t";
                E0 = E1;
                E1 = test.energy();
                diff = (E1 - E0)/p.dt;
                double diss = test.energy_diffusion( );
                std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
                std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<"\n";

            }
            try{ ab( test, y0, y1, p.dt);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                running = false;
                break;
            }
            y0.swap( y1); //attention on -O3 ?
        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
        running = running && 
                  !glfwGetKey( GLFW_KEY_ESC) &&
                  glfwGetWindowParam( GLFW_OPENED);
    }
    ////////////////////////////////////////////////////////////////////

    return 0;

}
