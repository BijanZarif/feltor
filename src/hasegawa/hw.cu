#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>

#include "draw/host_window.h"
//#include "draw/device_window.cuh"

#include "hw.cuh"
#include "dg/multistep.h"
#include "dg/backend/timer.cuh"
#include "file/read_input.h"
#include "../toefl/parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/

const unsigned k = 3; //!< a change of k needs a recompilation!

int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v, v2;
    std::stringstream title;
    if( argc == 1)
    {
        v = file::read_input("input.txt");
    }
    else if( argc == 2)
    {
        v = file::read_input( argv[1]);
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }

    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow( v2[3], v2[4], "");
    draw::RenderHostData render(v2[1], v2[2]);
    /////////////////////////////////////////////////////////////////////////
    const Parameters p( v);
    p.display( std::cout);
    if( p.k != k)
    {
        std::cerr << "ERROR: k doesn't match: "<<k<<" vs. "<<p.k<<"\n";
        return -1;
    }

    dg::Grid2d<double > grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    //create RHS 
    dg::HW< dg::DVec > test( grid, p.kappa, p.tau, p.nu, p.eps_pol, (bool)p.global); 
    dg::DVec one( grid.size(), 1.);
    //create initial vector
    dg::Gaussian gaussian( p.posX*grid.lx(), p.posY*grid.ly(), p.sigma, p.sigma, p.n0); //gaussian width is in absolute values
    dg::Vortex vortex( p.posX*grid.lx(), p.posY*grid.ly(), 0, p.sigma, p.n0);
    std::vector<dg::DVec> y0(2, dg::evaluate( vortex, grid)), y1(y0); // n_e' = gaussian
    dg::DVec w2d( dg::create::weights( grid));

    if( p.bc_x == dg::PER && p.bc_y == dg::PER)
    {
        double meanMass = dg::blas2::dot( y0[0], w2d, one)/(double)(p.lx*p.ly);
        std::cout << "Mean Mass is "<<meanMass<<"\n";
        dg::blas1::axpby( -meanMass, one, 1., y0[0]);
        y0[1] = y0[0];
    }
    //dg::AB< k, std::vector<dg::DVec> > ab( y0);
    //dg::TVB< std::vector<dg::DVec> > ab( y0);
    dg::Karniadakis<std::vector<dg::DVec> > ab( y0, y0[0].size(), 1e-9);
    dg::Diffusion<dg::DVec> diffusion( grid, p.nu);

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::HMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    dg::Timer t;
    double time = 0;
    ab.init( test, diffusion, y0, p.dt);
    //ab( test, y0, y1, p.dt);
    //y0.swap( y1); 
    double E0 = test.energy(), energy0 = E0, E1 = 0, diff = 0;
    double Ezf0 = test.zonal_flow_energy(), energyzf0 = Ezf0, Ezf1 = 0, diffzf = 0;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    unsigned step = 0;
    while ( !glfwWindowShouldClose( w ))
    {
        if( p.bc_x == dg::PER && p.bc_y == dg::PER)
        {
            double meanMass = dg::blas2::dot( y0[0], w2d, one)/(double)(p.lx*p.ly);
            std::cout << "Mean Mass is "<<meanMass<<"\n";
        }
        //transform field to an equidistant grid
        dvisual = y0[0];

        hvisual = dvisual;
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ne / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //transform phi
        dg::blas2::gemv( test.laplacianM(), test.potential(), y1[1]);
        hvisual = y1[1];
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"omega / "<<colors.scale()<<"\t";
        title << std::fixed; 
        title << " &&   time = "<<time;
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            {
                if( p.bc_x == dg::PER && p.bc_y == dg::PER)
                {
                    double meanMass = dg::blas2::dot( y0[0], w2d, one)/(double)(p.lx*p.ly);
                    dg::blas1::axpby( -meanMass, one, 1., y0[0]);
                    meanMass = dg::blas2::dot( y0[1], w2d, one)/(double)(p.lx*p.ly);
                    dg::blas1::axpby( -meanMass, one, 1., y0[1]);
                }
                E0 = E1; Ezf0 = Ezf1;
                E1 = test.energy(); Ezf1 = test.zonal_flow_energy();
                diff = (E1 - E0)/p.dt; diffzf = (Ezf1-Ezf0)/p.dt;
                double diss = test.energy_diffusion( ) + test.flux() + test.capital_jot(); double disszf = test.zonal_flow_diffusion() + test.capital_r();
                //std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
                std::cout << "Accuracy:   "<< 2.*(diff-diss)/(diff+diss)<<"\n";
                std::cout << "AccuracyZF: "<< 2.*(diffzf-disszf)/(diffzf+disszf)<<"\n";
                std::cout << diff << " "<< test.energy_diffusion() << " " <<test.flux()<<" "<<test.capital_jot()<<"\n";
                std::cout << diffzf << " "<< test.zonal_flow_diffusion() << " " <<test.capital_r()<<"\n";

            }
            try{ ab( test, diffusion, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
