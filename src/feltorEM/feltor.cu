#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/timer.cuh"
#include "dg/backend/xspacelib.cuh"
#include "file/read_input.h"

#include "feltor.cuh"
#include "bessel.h"
#include "parameters.h"
#include "geometry.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/


int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v, v2,v3;
    std::stringstream title;
    if( argc == 1)
    {
        try{
        v = file::read_input("input.txt");
        }catch( toefl::Message& m){m.display();}
    }
    else if( argc == 2)
    {
        v = file::read_input( argv[1]);
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }

    /////////////////////////////////////////////////////////////////////////
    const Parameters p( v);
    p.display( std::cout);
    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow( p.Nz/v2[2]*v2[3], v2[1]*v2[4], "");
    draw::RenderHostData render(v2[1], p.Nz/v2[2]);

    //////////////////////////////////////////////////////////////////////////
//     dg::Grid3d<double > grid( p.R_0-p.a*(1.05), p.R_0 + p.a*(1.05),  -p.a*(1.05), p.a*(1.05), 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER);
    
    try{ v3 = file::read_input( "geometry_params.txt"); }
    catch (toefl::Message& m) {  
        m.display(); 
        for( unsigned i = 0; i<v.size(); i++)
//             std::cout << v3[i] << " ";
//             std::cout << std::endl;
        return -1;}

    const solovev::GeomParameters gp(v3);
    gp.display( std::cout);
    double Rmin=gp.R_0-(gp.boxscale)*gp.a;
    double Zmin=-(gp.boxscale)*gp.a*gp.elongation;
    double Rmax=gp.R_0+(gp.boxscale)*gp.a; 
    double Zmax=(gp.boxscale)*gp.a*gp.elongation;
    //Make grid
     dg::Grid3d<double > grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, dg::DIR, dg::DIR, dg::PER);  
     
    //create RHS 
    eule::Feltor< dg::DVec > feltor( grid, p,gp); //initialize before rolkar!
    eule::Rolkar< dg::DVec > rolkar( grid, p,gp);


      dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,15.,p.amp);
//       solovev::ZonalFlow init0(gp,p.amp);
    
//     solovev::Gradient grad(gp); //background gradient
    solovev::Nprofile grad(gp); //initial profile

    std::vector<dg::DVec> y0(4, dg::evaluate( grad, grid)), y1(y0); 
    //damp the bath on psi boundaries 
    dg::blas1::pointwiseDot(rolkar.dampin(),(dg::DVec)dg::evaluate(init0, grid), y1[0]); //is damping on bath

    
    dg::blas1::axpby( 1., y1[0], 1., y0[0]);
    dg::blas1::axpby( 1., y1[0], 1., y0[1]);
    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0
//     dg::blas1::pointwiseDot(rolkar.dampout(),y0[1],y0[1]); //is damping on bath
    feltor.log( y0, y0, 2); //transform to logarithmic values (ne and ni)
    
    dg::Karniadakis< std::vector<dg::DVec> > ab( y0, y0[0].size(), p.eps_time);
    ab.init( feltor, rolkar, y0, p.dt);

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::HMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);

    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    
    const double mass0 = feltor.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0, diff = 0;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    while ( !glfwWindowShouldClose( w ))
    {
        //transform field to an equidistant grid
        feltor.exp( y0, y1, 2); //calculate real densities from logdensities

        //plot electrons
        thrust::transform( y1[0].begin(), y1[0].end(), dvisual.begin(), dg::PLUS<double>(0.));//ne-1
        hvisual = dvisual;
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title << std::setprecision(2) << std::scientific;
        title <<"ne / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }

        //draw ions
        thrust::transform( y1[1].begin(), y1[1].end(), dvisual.begin(), dg::PLUS<double>(0.));//ne-1
        hvisual = dvisual;
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title << std::setprecision(2) << std::scientific;
        title <<"ni / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }

        //transform phi
        dg::blas2::gemv( rolkar.laplacianM(), feltor.potential()[0], y1[1]);
        hvisual = y1[1];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0.,thrust::maximum<double>()  );
        colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"phi / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }

        //draw U_e
        hvisual = feltor.uparallel()[0]; //=U_parallel_e
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0.,thrust::maximum<double>()  );
        colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"Ue / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }

        //draw U_i
        hvisual =feltor.uparallel()[1];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>()  );
        colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"Ui / "<<colors.scalemin()<< "  " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }

        //draw a parallel
        hvisual =feltor.aparallel();
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>()  );
        colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"A / "<<colors.scalemin()<< "  " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        //double x;
        //std::cin >> x;
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass_blob0<<"\t";
            E0 = E1;
            E1 = feltor.energy();
            diff = (E1 - E0)/p.dt;
            double diss = feltor.energy_diffusion( );
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<"\n";

            try{ ab( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
