#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"
#include "dg/backend/average.cuh"
#include "file/read_input.h"
#include "solovev/geometry.h"

#include "asela.cuh"
#include "asela/parameters.h"

#define TORLIM //for toroidal limiter setup

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/


int main( int argc, char* argv[])
{
     ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v2,v3;
    std::stringstream title;
    if( argc == 1)
    {
        try{
            v = file::read_input("input.txt");
            v3 = file::read_input( "geometry_params.txt"); 
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else if( argc == 3)
    {
        try{
            v = file::read_input(argv[1]);
            v3 = file::read_input( argv[2]); 
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const eule::Parameters p( v);
    p.display( std::cout);
    const solovev::GeomParameters gp(v3);
    gp.display( std::cout);
    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow( (p.Nz+1)/v2[2]*v2[3], v2[1]*v2[4], "");
    draw::RenderHostData render(v2[1], (p.Nz+1)/v2[2]);



    //////////////////////////////////////////////////////////////////////////
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a; 
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grid
     dg::Grid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, p.bc, p.bc, dg::PER, dg::cylindrical);  
    //create RHS 
    std::cout << "Constructing asela...\n";
    eule::Asela<dg::DDS, dg::DMatrix, dg::DVec, dg::DVec > asela( grid, p,gp); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p,gp);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //initial perturbation
    //dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
//     dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
//     solovev::ZonalFlow init0(p, gp);

    
    //background profile
    solovev::Nprofile prof(p, gp); //initial background profile
    std::vector<dg::DVec> y0(4, dg::evaluate( prof, grid)), y1(y0); 
    
    //field aligning
    //dg::CONSTANT gaussianZ( 1.);
    dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1);
    y1[1] = asela.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 3); //rounds =2 ->2*2-1

    //no field aligning
//     y1[1] = dg::evaluate( init0, grid);
    
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni
    dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-1)); //initialize ni-1
    dg::blas1::pointwiseDot(rolkar.damping(),y0[1], y0[1]); //damp with gaussprofdamp
    asela.initializene( y0[1], y0[0]);    
    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set we = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set wi = 0

    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( asela, rolkar, y0, p.dt);
    karniadakis( asela, rolkar, y0); //now energies and potential are at time 0

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual),avisual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
    dg::ToroidalAverage<dg::HVec> toravg(grid);

    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    const double mass0 = asela.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = asela.energy(), energy0 = E0, E1 = 0, diff = 0;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    
   

    while ( !glfwWindowShouldClose( w ))
    {
        //plot electrons
        hvisual = karniadakis.last()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() = -colors.scalemax();   
        title << std::setprecision(2) << std::scientific;
        title <<"ne-1 / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw ions
        hvisual =  karniadakis.last()[1];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() = -colors.scalemax();   
        title << std::setprecision(2) << std::scientific;
        title <<"ni-1 / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw Potential
        hvisual = asela.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        //transform to Vor
        //dvisual=asela.potential()[0];
        //dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
        //hvisual = y1[1];
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0.,thrust::maximum<double>()  );
//         colors.scalemin() =  (float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        title <<"Phi / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);


        //draw U_e
        hvisual = asela.uparallel()[0]; //=U_parallel_e
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() = -colors.scalemax();   
        title <<"Ue / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw U_i
        hvisual =asela.uparallel()[1];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() = -colors.scalemax();   
        title <<"Ui / "<< colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw a parallel
        hvisual = asela.aparallel();
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (float)thrust::reduce( visual.begin(),visual.end(), 0., thrust::maximum<double>()  );
        colors.scalemin() = - colors.scalemax();
        title <<"A / "<<(float)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<< "  " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
 
        
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        //double x;
        //std::cin >> x;
        for( unsigned i=0; i<p.itstp; i++)
        {
            try{ karniadakis( asela, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
            step++;
            asela.energies( y0); //update energetics
            std::cout << "(m_tot-m_0)/m_0: "<< (asela.mass()-mass0)/mass_blob0<<"\t";
            E1 = asela.energy();
            diff = (E1 - E0)/p.dt; //
            double diss = asela.energy_diffusion( );
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<" d E/dt = " << diff <<" Lambda =" << diss << "\n";
            E0 = E1;

        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
