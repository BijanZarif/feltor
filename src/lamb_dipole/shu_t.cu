#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <thrust/remove.h>
#include <thrust/host_vector.h>

#include "draw/host_window.h"

#include "dg/functors.cuh"
#include "dg/arrvec2d.cuh"
#include "dg/evaluation.cuh"
#include "dg/xspacelib.cuh"
#include "dg/rk.cuh"
#include "dg/typedefs.cuh"

#include "shu.cuh"


using namespace std;
using namespace dg;

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;

const unsigned k = 2;
const double D = 0.01;
const double T = 1.;


double initial( double x, double y){return 2.*sin(x)*sin(y);}
double solution( double x, double y) {return 2.*sin(x)*sin(y)*exp( -2.*T*D);}


int main()
{
    unsigned n, Nx, Ny;
    double eps;
    cout << "Type n, Nx, Ny and eps!\n";
    cin >> n >> Nx >> Ny>>eps;
    const unsigned NT = (unsigned)(D*T*n*n*Nx*Nx/0.01/lx/lx);
    
    Grid<double> grid( 0, lx, 0, ly, n, Nx, Ny, dg::PER, dg::PER);
    DVec w2d( create::w2d( grid));
    const double dt = T/(double)NT;
    /////////////////////////////////////////////////////////////////
    //create CUDA context that uses OpenGL textures in Glfw window
    draw::HostWindow w( 600, 600);
    glfwSetWindowTitle( "Navier Stokes");
    ////////////////////////////////////////////////////////////
    cout << "# of Legendre coefficients: " << n<<endl;
    cout << "# of grid cells:            " << Nx*Ny<<endl;
    cout << "Timestep                    " << dt << endl;
    //cout << "# of timesteps              " << NT << endl;
    cout << "Diffusion                   " << D <<endl;
    dg::Lamb lamb( 0.5*lx, 0.5*ly, 0.2*lx, 1);
    HVec omega = evaluate ( lamb, grid);
    DVec stencil = evaluate( one, grid);
    //DArrVec sol = evaluate< double(&)(double, double), n> ( solution, 0, lx, 0, ly, Nx, Ny);
    DVec y0( omega), y1( y0);
    Shu<DVec> test( grid, D, eps);
    AB< k, DVec > ab( y0);

    ////////////////////////////////glfw//////////////////////////////
    //create visualisation vectors
    DVec visual( grid.size());
    HVec hvisual( grid.size());
    //transform vector to an equidistant grid
    dg::DMatrix equidistant = dg::create::backscatter( grid, LSPACE );
    int running = GL_TRUE;
    draw::ColorMapRedBlueExt colors( 1.);
    ab.init( test, y0, dt);
    while (running)
    {
        //transform field to an equidistant grid
        cout << "Total vorticity is: "<<blas2::dot( stencil, w2d, y0) << "\n";
        cout << "Total enstrophy is: "<<blas2::dot( w2d, y0)<<"\n";
        //compute the color scale
        dg::blas2::mv( equidistant, y0, visual );
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), -1., dg::AbsMax<float>() );
        std::cout << "Color scale " << colors.scale() <<"\n";
        //draw and swap buffers
        hvisual = visual;
        w.draw( hvisual, n*Nx, n*Ny, colors);
        //step 
        ab( test, y0, y1, dt);
        //thrust::swap(y0, y1);
        y0.swap( y1);

        glfwWaitEvents();
        running = !glfwGetKey( GLFW_KEY_ESC) &&
                    glfwGetWindowParam( GLFW_OPENED);
    }
    ////////////////////////////////////////////////////////////////////
    /*
    cout << "Total vorticity is: "<< blas2::dot( stencil, w2d, y0) << "\n";
    cout << "Total enstrophy  is "<<blas2::dot( y0, w2d, y0)<<"\n";
    blas1::axpby( 1., sol.data(), -1., y0);
    hipDeviceSynchronize();
    cout << "Distance to solution "<<sqrt( blas2::dot( w2d, y0))<<endl; //don't forget sqrt when comuting errors
    */

    return 0;

}
