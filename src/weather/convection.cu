#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>

#include "draw/host_window.h"

#include "convection.cuh"
#include "dg/rk.cuh"
#include "dg/timer.cuh"

#include "file/read_input.h"

struct InitDens
{
    InitDens( double alpha, double zeta): alpha_( alpha), zeta_(zeta){}

    double operator()( double x, double y)
    {
        return tanh( (zeta_-y)/alpha_);
    }
  private:
    double alpha_, zeta_;
};

struct Parameters
{
    unsigned n, k,  Nx, Ny;
    double dt; 
    double eps_lap;
    double lx, ly;

    double eps, R_l, L, P, R, zeta;
    double n0; 
    unsigned itstp;
    unsigned maxout;
    Parameters( const std::vector<double>& v)
    {
        n = (unsigned)v[1];
        k = (unsigned)v[4];
        Nx = (unsigned)v[2];
        Ny = (unsigned)v[3];
        dt = v[5];
        eps_lap = v[6];
        lx = v[7];
        ly = v[8];
        R = v[9];
        P = v[10];
        L = v[11];
        eps = v[12];
        R_l = v[13];
        zeta = v[14];
        itstp = v[15];
        maxout = v[16];
    }
    void display( std::ostream& os = std::cout ) const
    {
        os <<  "Algorithmic parameters are: \n"
           <<  "    n = "<<n<<"\n"
           <<  "    Nx = "<<Nx<<"\n"
           <<  "    Ny = "<<Ny<<"\n"
           <<  "    k = "<<k<<"\n"
           <<  "    dt = "<<dt<<"\n"
           <<  "    eps_lap = "<<eps_lap<<"\n";
        os <<  "Boundary parameters are: \n"
           <<  "    lx = "<<lx<<"\n"
           <<  "    ly = "<<ly<<"\n";
        os <<  "Physical Parameters are: \n"
           <<  "    R = "<<R<<"\n"
           <<  "    P = "<<P<<"\n"
           <<  "    L = "<<L<<"\n"
           <<  "    eps = "<<eps<<"\n"
           <<  "    R_l = "<<R_l<<"\n"
           <<  "    zeta = "<<zeta<<"\n";
        os <<  "Steps between output: "<<itstp<<"\n"
           <<  "Number of outputs:    "<<maxout<<std::endl; //implicit flush!
    }
};

const unsigned k = 3;

int main(int argc, char* argv[])
{
    std::vector<double> v, v2;
    if( argc==1)
        try{ v=file::read_input( "input.txt");}catch( toefl::Message& m){m.display();}
    else if( argc==2)
        try{ v=file::read_input( argv[1]);}catch( toefl::Message& m){m.display();}
    else
    {
        std::cerr << "ERROR: Too many arguments! \n Usage: "<<argv[0]<<" [filename]\n";
        return -1;
    }
    try{ v2=file::read_input( "window_params.txt");}catch( toefl::Message& m){m.display();}
    draw::HostWindow w( v2[3], v2[4]);
    w.set_multiplot( v2[1], v2[2]);

    const Parameters p(v);
    if( p.k != k)
    {
        std::cerr << "ERROR: k doesn't match "<<k<<" vs. "<<p.k<<"\n";
        return -1;
    }
    Params params; 
    params.eps = p.eps, params.P = p.P;
    params.R = p.R, params.L = p.L, params.R_l = p.R_l, params.zeta = p.zeta;

    dg::Grid<double> grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, dg::PER, dg::DIR);
    Convection< dg::DVec> convect( grid, params, p.eps_lap);
    //initial conditions
    dg::Gaussian g1( 0.4*p.lx, 0.3*p.ly, 5, 5, p.n0);
    dg::Gaussian g2( 0.7*p.lx, 0.35*p.ly, 5, 5,  p.n0);
    std::vector<dg::DVec> y0( 3);
    y0[0] = y0[1] = y0[2] = dg::evaluate( g1, grid);
    dg::blas1::axpby( 1., y0[0], 1., (dg::DVec)dg::evaluate( g2, grid), y0[0]);
    y0[1] = dg::evaluate( InitDens(0.001, p.zeta), grid);
    dg::blas1::axpby( 1., y0[2], -1, y0[2]);
    //init timestepper
    dg::AB< k, std::vector<dg::DVec> > ab(y0);
    ab.init( convect, y0, p.dt);
    std::vector<dg::DVec> y1( y0);
    ab( convect, y0, y1, p.dt);
    y0.swap(y1);
    unsigned step = 0;
    double time = 0;
    //visualization
    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::HMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    bool running = true;
    while ( running) 
    {
        dg::blas1::axpby( 1., y0[0], 1., convect.background(), dvisual);
        hvisual = dvisual;
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scale() = p.R/2.;

        w.title() << std::setprecision(2) << std::scientific;
        w.title() <<"temp / "<<colors.scale()<<"\t";
        w.draw( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //steps
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            try{ ab( convect, y0, y1, p.dt);}
            catch( Fail& fail){
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does simulation respect CFL condition?\n";
                running = false;
                break;
            }
        }
        y0.swap( y1);
        running = running &&
            !glfwGetKey( GLFW_KEY_ESC) &&
            glfwGetWindowParam( GLFW_OPENED);
    }



    
}
