#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <vector>
#include <string>

#include "dg/xspacelib.cuh"
#include "file/read_input.h"
#include "file/file.h"

#include "galerkin/parameters.h"

double X( double x, double y) {return x;}
double Y( double x, double y) {return y;}
template< class container>
void log( const container& y, container& target)
{
    thrust::transform( y.begin(), y.end(), target.begin(), dg::LN<double>());
}

template <class container>
struct Vesqr
{
    Vesqr( const dg::Grid<double>& grid, double kappa): dx( grid.size()), dy(dx), one( grid.size(), 1.), w2d( dg::create::w2d(grid)), binv( evaluate( dg::LinearX( kappa, 1.), grid)), arakawa(grid){}
    const container& operator()( const container& phi)
    {
        dg::blas2::gemv( arakawa.dx(), phi, dx);
        dg::blas2::gemv( arakawa.dy(), phi, dy);
        dg::blas1::pointwiseDot( binv, dx, dx);
        dg::blas1::pointwiseDot( binv, dy, dy);
        dg::blas1::pointwiseDot( dx, dx, dx);
        dg::blas1::pointwiseDot( dy, dy, dy);
        dg::blas1::axpby( 1., dx, 1.,  dy);
        return dy;

    }
  private:
    container dx, dy, one, w2d, binv;    
    dg::ArakawaX<container> arakawa;

};

int main( int argc, char* argv[])
{
    if( argc != 3)
    {
        std::cerr << "Usage: "<<argv[0]<<" [input.h5] [output.dat]\n";
        return -1;
    }
    std::ofstream os( argv[2]);
    std::cout << argv[1]<< " -> "<<argv[2]<<std::endl;

    std::string in;
    file::T5rdonly t5file( argv[1], in);
    const unsigned num_out = t5file.get_size();
    const Parameters p( file::read_input( in), 0);
    //p.display();
    dg::Grid<double> grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);

    dg::HVec input_h( grid.size());
    dg::HVec input0( input_h), input1(input0), ln0( input0), ln1(input0);
    dg::HVec visual( input0);
    std::vector<double> mass, energy, diffusion, dissipation;
    if( p.global)
    {
        t5file.get_xfile( mass, "mass");
        t5file.get_xfile( energy, "energy");
        t5file.get_xfile( diffusion, "diffusion");
        t5file.get_xfile( dissipation, "dissipation");
    }

    dg::HVec xvec = dg::evaluate( X, grid);
    dg::HVec yvec = dg::evaluate( Y, grid);
    dg::HVec one = dg::evaluate( dg::one, grid);
    dg::HVec w2d = dg::create::w2d( grid);
    dg::HMatrix equi = dg::create::backscatter( grid);

    double mass_, posX, posY, velX, velY;
    double posX_max, posY_max;
    double posX_old = 0, posY_old = 0;
    double deltaT = p.dt*p.itstp;
    Vesqr<dg::HVec> vesqr( grid, p.kappa);
    os << "#Time(1) posX(2) posY(3) velX(4) velY(5) mass(6) diff(7) (m_tot-m_0)/m_0(8) "
       << "Ue(9) Ui(10) Uphi(11) Utot(12) (U_tot-U_0)/U_0(13) diss(14) posX_max(15) posY_max(16) \n";
    for( unsigned idx=1; idx<=num_out; idx++)
    {
        t5file.get_field( input_h, "electrons", idx);
        input0 = input_h;
        t5file.get_field( input_h, "ions", idx);
        input1 = input_h;
        if( p.global)
        {
            log( input0, ln0), log( input1, ln1);
            double Ue = dg::blas2::dot( input0, w2d, ln0);
            double Ui = p.tau*dg::blas2::dot( input1, w2d, ln1);
            double Uphi = energy[(idx-1)*p.itstp] - Ue - Ui;
            thrust::transform( input0.begin(), input0.end(), input0.begin(), dg::PLUS<double>(-1));
        }
        mass_ = dg::blas2::dot( one, w2d, input0 ); 

        posX = dg::blas2::dot( xvec, w2d, input0)/mass_ - p.posX*p.lx;
        posY = dg::blas2::dot( yvec, w2d, input0)/mass_ - p.posY*p.ly;
        velX = (posX - posX_old)/deltaT;
        velY = (posY - posY_old)/deltaT;
        posX_old = posX;
        posY_old = posY;
        //output
        os << t5file.get_time( idx);//(1)
        os << " "<<posX << " " << posY << " "<<velX<<" "<<velY;//(2-5)
        os << " "<<mass[(idx-1)*p.itstp] << " "<<diffusion[(idx-1)*p.itstp];//(6,7)
        os << " "<< (mass[(idx-1)*p.itstp]-mass[0])/(mass[0]-grid.lx()*grid.ly());//blob mass is mass[] - Area (8)
        os << " "<<Ue<<" "<<Ui<<" "<<Uphi<<" "<<energy[(idx-1)*p.itstp]; //(9-12)
        os << " "<<(energy[(idx-1)*p.itstp]-energy[0])/energy[0];//(13)
        os << " "<<dissipation[(idx-1)*p.itstp]; //(14)
        //get the maximum amplitude position
        dg::blas2::gemv( equi, input0, visual);
        unsigned position = thrust::distance( visual.begin(), thrust::max_element( visual.begin(), visual.end()) );
        unsigned Nx = p.Nx*p.n; 
        const double hx = grid.hx()/(double)grid.n();
        const double hy = grid.hy()/(double)grid.n();
        posX_max = hx*(1./2. + (double)(position%Nx))-p.posX*p.lx;
        posY_max = hy*(1./2. + (double)(position/Nx))-p.posY*p.ly;
        os << " "<<posX_max<<" "<<posY_max;
        os <<"\n";
    }
    os.close();
    return 0;
}

