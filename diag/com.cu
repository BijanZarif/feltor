#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <vector>
#include <string>

#include "dg/xspacelib.cuh"
#include "file/read_input.h"
#include "file/file.h"

#include "galerkin/parameters.h"

double X( double x, double y) {return x;}
double Y( double x, double y) {return y;}
template< class container>
void log( const container& y, container& target)
{
    thrust::transform( y.begin(), y.end(), target.begin(), dg::LN<double>());
}

int main( int argc, char* argv[])
{
    if( argc != 3)
    {
        std::cerr << "Usage: "<<argv[0]<<" [input.h5] [output.dat]\n";
        return -1;
    }
    //open file for com - data
    //std::string outputfile( argv[1]);
    //outputfile.erase( outputfile.end()-2, outputfile.end());
    //outputfile+="com";
    //std::ofstream os( outputfile.c_str());
    std::ofstream os( argv[2]);

    std::string in;
    file::T5rdonly t5file( argv[1], in);
    const unsigned num_out = t5file.get_size();
    const Parameters p( file::read_input( in));
    p.display();
    dg::Grid<double> grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);

    dg::HVec input_h( grid.size());
    dg::DVec input0( input_h), input1(input0), ln0( input0), ln1(input0);
    std::vector<double> mass, energy, diffusion, dissipation;
    t5file.get_xfile( mass, "mass");
    t5file.get_xfile( energy, "energy");
    t5file.get_xfile( diffusion, "diffusion");
    t5file.get_xfile( dissipation, "dissipation");

    dg::DVec xvec = dg::evaluate( X, grid);
    dg::DVec yvec = dg::evaluate( Y, grid);
    dg::DVec one = dg::evaluate( dg::one, grid);
    dg::DVec w2d = dg::create::w2d( grid);

    double mass_, posX, posY, velX, velY;
    double posX_old = 0, posY_old = 0;
    double deltaT = p.dt*p.itstp;
    os << "#Time(1) posX(2) posY(3) velX(4) velY(5) mass(6) diff(7) (m_tot-m_0)/m_0(8) "
       << "Ue(9) Ui(10) Uphi(11) Utot(12) (U_tot-U_0)/U_0(13) diss(14) \n";
    for( unsigned idx=1; idx<=num_out; idx++)
    {
        t5file.get_field( input_h, "electrons", idx);
        input0 = input_h;
        t5file.get_field( input_h, "electrons", idx);
        input1 = input_h;
        log( input0, ln0), log( input1, ln1);
        double Ue = dg::blas2::dot( input0, w2d, ln0);
        double Ui = p.tau*dg::blas2::dot( input1, w2d, ln1);
        double Uphi = energy[(idx-1)*p.itstp] - Ue - Ui;
        if( p.global)
            thrust::transform( input0.begin(), input0.end(), input0.begin(), dg::PLUS<double>(-1));
        mass_ = dg::blas2::dot( one, w2d, input0 ); 

        posX = dg::blas2::dot( xvec, w2d, input0)/mass_ - p.posX*p.lx;
        posY = dg::blas2::dot( yvec, w2d, input0)/mass_ - p.posY*p.ly;
        velX = (posX - posX_old)/deltaT;
        velY = (posY - posY_old)/deltaT;
        posX_old = posX;
        posY_old = posY;
        //output
        os << t5file.get_time( idx);//(1)
        os << " "<<posX << " " << posY << " "<<velX<<" "<<velY;//(2-5)
        os << " "<<mass[(idx-1)*p.itstp] << " "<<diffusion[(idx-1)*p.itstp];//(6,7)
        os << " "<< (mass[(idx-1)*p.itstp]-mass[0])/(mass[0]-grid.lx()*grid.ly());//blob mass is mass[] - Area (8)
        os << " "<<Ue<<" "<<Ui<<" "<<Uphi<<" "<<energy[(idx-1)*p.itstp]; //(9-12)
        os << " "<<(energy[(idx-1)*p.itstp]-energy[0])/energy[0];//(13)
        os << " "<<dissipation[(idx-1)*p.itstp]; //(14)
        os <<"\n";
    }
    os.close();
    return 0;
}

