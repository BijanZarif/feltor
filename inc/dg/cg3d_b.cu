#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "backend/timer.cuh"
#include "backend/evaluation.cuh"
#include "backend/derivatives.cuh"
#include "cg.h"

#include "backend/typedefs.cuh"


const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
const double lz = 1.;

dg::bc bcx = dg::DIR;
double initial( double x, double y, double z) {return sin(0);}
double fct(double x, double y, double z){ return sin(y)*sin(x)*sin(2.*M_PI*z);}
double laplace_fct( double x, double y, double z) { return 2*sin(y)*sin(x)*sin(2.*M_PI*z);}

//const double lx = 2./3.*M_PI;
//double fct(double x, double y){ return sin(y)*sin(3.*x/4.);}
//double laplace_fct( double x, double y) { return 25./16.*sin(y)*sin(3.*x/4.);}
//dg::bc bcx = dg::DIR_NEU;

int main()
{
    dg::Timer t;
    unsigned n, Nx, Ny, Nz; 
    std::cout << "Type n, Nx, Ny and Nz\n";
    std::cin >> n >> Nx >> Ny>> Nz;
    std::cout << "Type in eps\n";
    double eps = 1e-6; 
    std::cin >> eps;

    std::cout << "TEST 3D VERSION\n";
    dg::Grid3d<double> g3d( 0, lx, 0, ly, 0, lz, n, Nx, Ny, Nz, bcx, dg::PER);
    dg::HVec w3d = dg::create::weights( g3d);
    dg::HVec v3d = dg::create::inv_weights( g3d);
    dg::HVec x3 = dg::evaluate( initial, g3d);
    dg::HVec b3 = dg::evaluate ( laplace_fct, g3d);
    dg::blas2::symv( w3d, b3, b3);

    dg::DVec w3d_d(w3d), v3d_d(v3d), x3_d(x3), b3_d(b3);

    dg::HMatrix A3 = dg::create::laplacianM_perp( g3d, dg::not_normed); //this function is deprecated
    dg::DMatrix A3_d(A3);
    dg::CG<dg::HVec > pcg3_host( x3, g3d.size());
    dg::CG<dg::DVec > pcg3_d( x3_d, g3d.size());
    t.tic();
    std::cout << "Number of pcg iterations "<< pcg3_d( A3_d, x3_d, b3_d, v3d_d, eps, sqrt(lz))<<std::endl;
    t.toc();
    std::cout << "... for a precision of "<< eps<<std::endl;
    std::cout << "... on the device took "<< t.diff()<<"s\n";
    t.tic();
    std::cout << "Number of pcg iterations "<< pcg3_host( A3, x3, b3, v3d, eps, sqrt(lz))<<std::endl;
    t.toc();
    std::cout << "... for a precision of "<< eps<<std::endl;
    std::cout << "... on the host took   "<< t.diff()<<"s\n";
    //compute error
    const dg::HVec solution3 = dg::evaluate ( fct, g3d);
    dg::HVec error3( solution3);
    dg::blas1::axpby( 1.,x3,-1.,error3);

    double eps3 = dg::blas2::dot(w3d , error3);
    double norm3 = dg::blas2::dot(w3d , solution3);
    std::cout << "L2 Norm of relative error is:  " <<sqrt( eps3/norm3)<<std::endl;


    return 0;
}
