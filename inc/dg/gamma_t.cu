#include "hip/hip_runtime.h"
#include <iostream>

#include "blas.h"

#include "gamma.cuh"
#include "xspacelib.cuh"

#include "cg.cuh"

const unsigned n=3;
const double eps = 1e-4;
const double alpha = -0.5; 
double lhs( double x, double y){ return sin(x)*sin(y);}
double rhs( double x, double y){ return (1.-2.*alpha)*sin(x)*sin(y);}
//double rhs( double x, double y){ return lhs(x,y);}
int main()
{
    
    unsigned Nx, Ny; 
    std::cout << "Type Nx and Ny\n";
    std::cin >> Nx >> Ny;
    dg::Grid<double, n> grid( 0, 2.*M_PI, 0, 2.*M_PI, Nx, Ny, dg::DIR, dg::DIR);
    dg::W2D<double, n> w2d( grid.hx(), grid.hy());
    dg::V2D<double, n> v2d( grid.hx(), grid.hy());
    dg::DVec rho = dg::evaluate( rhs, grid);
    dg::DVec sol = dg::evaluate( lhs, grid);
    dg::DVec x(rho.size(), 0.);
    //dg::DVec x(rho);

    dg::DMatrix A = dg::create::laplacianM( grid, dg::normed, dg::XSPACE); 
    dg::Gamma< dg::DMatrix, dg::W2D<double, n> > gamma1( A, w2d, alpha);

    dg::CG< dg::DVec > cg(x, x.size());
    dg::blas2::symv( w2d, rho, rho);
    unsigned number = cg( gamma1, x, rho, v2d, eps);
    dg::blas1::axpby( 1., sol, -1., x);

    std::cout << "number of iterations:  "<<number<<std::endl;
    std::cout << "error " << sqrt( dg::blas2::dot( w2d, x))<<std::endl;




    return 0;
}



