#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "matrix_traits_thrust.h"
#include "timer.cuh"
#include "evaluation.cuh"
#include "cg.cuh"
#include "derivatives.cuh"

#include "typedefs.cuh"


//leo3 can do 350 x 350 but not 375 x 375
const double ly = 2.*M_PI;

const double eps = 1e-6; //# of pcg iterations increases very much if 
 // eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus

const double lx = M_PI;
double fct(double x, double y){ return sin(y)*sin(x);}
double derivative( double x, double y){return cos(x)*sin(y);}
double laplace_fct( double x, double y) { return 2*sin(y)*sin(x);}
dg::bc bcx = dg::DIR;
//const double lx = 2./3.*M_PI;
//double fct(double x, double y){ return sin(y)*sin(3.*x/4.);}
//double laplace_fct( double x, double y) { return 25./16.*sin(y)*sin(3.*x/4.);}
//dg::bc bcx = dg::DIR_NEU;
double initial( double x, double y) {return sin(0);}


int main()
{
    dg::Timer t;
    unsigned n, Nx, Ny; 
    std::cout << "Type n, Nx and Ny\n";
    std::cin >> n >> Nx >> Ny;
    dg::Grid2d<double> grid( 0., lx, 0, ly, n, Nx, Ny, bcx, dg::PER);
    const dg::HVec s2d_h = dg::create::weights( grid);
    const dg::DVec s2d_d( s2d_h);
    const dg::HVec t2d_h = dg::create::precond( grid);
    const dg::DVec t2d_d( t2d_h);
    std::cout<<"Expand initial condition\n";
    dg::HVec x = dg::evaluate( initial, grid);

    std::cout << "Create symmetric Laplacian\n";
    t.tic();
    dg::DMatrix dA = dg::create::laplacianM( grid, dg::not_normed, dg::forward); 
    dg::DMatrix DX = dg::create::dx( grid);
    dg::HMatrix A = dA;
    t.toc();
    std::cout<< "Creation took "<<t.diff()<<"s\n";

    dg::CG< dg::DVec > pcg( x, n*n*Nx*Ny);
    dg::CG< dg::HVec > pcg_host( x, n*n*Nx*Ny);

    std::cout<<"Expand right hand side\n";
    const dg::HVec solution = dg::evaluate ( fct, grid);
    const dg::DVec deriv = dg::evaluate( derivative, grid);
    dg::HVec b = dg::evaluate ( laplace_fct, grid);
    //compute S b
    dg::blas2::symv( s2d_h, b, b);

    //copy data to device memory
    t.tic();
    const dg::DVec dsolution( solution);
    dg::DVec db( b), dx( x);
    dg::DVec db_(b), dx_(x);
    dg::HVec b_(b), x_(x);
    t.toc();
    std::cout << "Allocation and copy to device "<<t.diff()<<"s\n";
    //////////////////////////////////////////////////////////////////////
    std::cout << "# of polynomial coefficients: "<< n <<std::endl;
    std::cout << "# of 2d cells                 "<< Nx*Ny <<std::endl;
    
    t.tic();
    std::cout << "Number of pcg iterations "<< pcg( dA, dx, db, t2d_d, eps)<<std::endl;
    t.toc();
    std::cout << "... for a precision of "<< eps<<std::endl;
    std::cout << "... on the device took "<< t.diff()<<"s\n";
    t.tic();
    dg::cg( dA, dx_, db_, t2d_d, eps, dx_.size());
    t.toc();
    std::cout << "... with function took "<< t.diff()<<"s\n";
    t.tic();
    std::cout << "Number of pcg iterations "<< pcg_host( A, x, b, t2d_h, eps)<<std::endl;
    t.toc();
    std::cout << "... for a precision of "<< eps<<std::endl;
    std::cout << "... on the host took   "<< t.diff()<<"s\n";
    t.tic();
    dg::cg( A, x_, b_, t2d_h, eps, x_.size());
    t.toc();
    std::cout << "... with function took "<< t.diff()<<"s\n";
    dg::DVec derror( dsolution);
    dg::HVec  error(  solution);
    dg::blas1::axpby( 1.,dx,-1.,derror);
    dg::blas1::axpby( 1., x,-1., error);

    double normerr = dg::blas2::dot( s2d_d, derror);
    double norm = dg::blas2::dot( s2d_d, dsolution);
    std::cout << "L2 Norm of relative error is:               " <<sqrt( normerr/norm)<<std::endl;
    dg::blas2::gemv( DX, dsolution, derror);
    dg::blas1::axpby( 1., deriv, -1., derror);
    normerr = dg::blas2::dot( s2d_d, derror); 
    norm = dg::blas2::dot( s2d_d, deriv);
    std::cout << "L2 Norm of relative error in derivative is: " <<sqrt( normerr/norm)<<std::endl;
    //both functiona and derivative converge with order P 

    return 0;
}
