#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "timer.cuh"
#include "cusp_eigen.h"
#include "evaluation.cuh"
#include "cg.cuh"
#include "derivatives.cuh"
#include "preconditioner.cuh"

#include "typedefs.cuh"

const unsigned n = 3; //global relative error in L2 norm is O(h^P)

//leo3 can do 350 x 350 but not 375 x 375
const double lx = 2.*M_PI;
const double ly = 2.*M_PI;

const double eps = 1e-6; //# of pcg iterations increases very much if 
 // eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus

typedef dg::T2D<double, n> Preconditioner;
typedef dg::S2D<double, n> Postconditioner;

double fct(double x, double y){ return sin(y)*sin(x);}
double laplace_fct( double x, double y) { return 2*sin(y)*sin(x);}
double initial( double x, double y) {return sin(0);}

using namespace std;

int main()
{
    dg::Timer t;
    unsigned Nx, Ny; 
    std::cout << "Type Nx and Ny\n";
    std::cin >> Nx >> Ny;
    dg::Grid<double, n> grid( 0, lx, 0, ly, Nx, Ny, dg::DIR, dg::DIR);
    dg::S2D<double,n > s2d( grid);
    cout<<"Expand initial condition\n";
    dg::HVec x = dg::expand( initial, grid);

    cout << "Create Laplacian\n";
    t.tic();
    dg::DMatrix dA = dg::create::laplacianM( grid, dg::not_normed, dg::LSPACE); 
    dg::HMatrix A = dA;
    /*
    dg::dgtensor<double, n>( dg::create::laplace1d_dir<double, n>( Ny, hy), 
                               dg::S1D<double, n>( hx),
                               dg::S1D<double, n>( hy),
                               dg::create::laplace1d_per<double, n>( Nx, hx)); //dir does also work but is slow
                               */
    t.toc();
    cout<< "Creation took "<<t.diff()<<"s\n";

    //create conjugate gradient and one eigen Cholesky
    dg::CG< dg::DVec > pcg( x, n*n*Nx*Ny);
    dg::CG< dg::HVec > pcg_host( x, n*n*Nx*Ny);
    //dg::SimplicialCholesky sol;
    //sol.compute( A);

    cout<<"Expand right hand side\n";
    const dg::HVec solution = dg::expand ( fct, grid);
    dg::HVec b = dg::expand ( laplace_fct, grid);
    //compute S b
    dg::blas2::symv( s2d, b, b);
    hipDeviceSynchronize();

    //copy data to device memory
    t.tic();
    const dg::DVec dsolution( solution);
    dg::DVec db( b), dx( x);
    t.toc();
    cout << "Allocation and copy to device "<<t.diff()<<"s\n";
    //////////////////////////////////////////////////////////////////////
    cout << "# of polynomial coefficients: "<< n <<endl;
    cout << "# of 2d cells                 "<< Nx*Ny <<endl;
    
    t.tic();
    cout << "Number of pcg iterations "<< pcg( dA, dx, db, Preconditioner(grid), eps)<<endl;
    t.toc();
    cout << "... for a precision of "<< eps<<endl;
    cout << "... on the device took "<< t.diff()<<"s\n";
    t.tic();
    cout << "Number of pcg iterations "<< pcg_host( A, x, b, Preconditioner(grid.hx(), grid.hy()), eps)<<endl;
    t.toc();
    cout << "... for a precision of "<< eps<<endl;
    cout << "... on the host took   "<< t.diff()<<"s\n";
    //t.tic();
    //cout << "Success (1) "<< sol.solve( x.data().data(), b.data().data(), n*n*Nx*Ny)<<endl;
    //t.toc();
    //cout << "Cholesky took          "<< t.diff()<<"s\n";
    //compute error
    dg::DVec derror( dsolution);
    dg::HVec  error(  solution);
    dg::blas1::axpby( 1.,dx,-1.,derror);
    dg::blas1::axpby( 1., x,-1., error);

    double normerr = dg::blas2::dot( s2d, derror);
    cout << "L2 Norm2 of CG Error is        " << normerr << endl;
    double normerr2= dg::blas2::dot( s2d,  error);
    cout << "L2 Norm2 of Cholesky Error is  " << normerr2 << endl;
    double norm = dg::blas2::dot( s2d, dsolution);
    cout << "L2 Norm of relative error is   " <<sqrt( normerr/norm)<<endl;
    cout << "L2 Norm of relative error is   " <<sqrt( normerr2/norm)<<endl;

    return 0;
}
