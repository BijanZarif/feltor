#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "backend/evaluation.cuh"
#include "arakawa.h"
#include "blas.h"
#include "backend/typedefs.cuh"

#include "backend/timer.cuh"

using namespace std;
using namespace dg;

const double lx = 2*M_PI;
const double ly = 2*M_PI;
//const double lx = 1.;
//const double ly = 1.;


//choose some mean function (attention on lx and ly)
//THESE ARE NOT PERIODIC
/*
double left( double x, double y) { return sin(x)*cos(y);}
double right( double x, double y){ return exp(0.1*(x+y)); }
double jacobian( double x, double y) 
{
    return exp( x-M_PI)*(sin(x)+cos(x))*sin(y) * exp(y-M_PI)*sin(x)*(sin(y) + cos(y)) - sin(x)*exp(x-M_PI)*cos(y) * cos(x)*sin(y)*exp(y-M_PI); 
}
*/

dg::bc bcx = dg::PER;
dg::bc bcy = dg::PER;
double left( double x, double y) {return sin(x)*cos(y);}
double right( double x, double y) {return cos(x)*sin(y);}
double jacobian( double x, double y) 
{
    return cos(x)*cos(y)*cos(x)*cos(y) - sin(x)*sin(y)*sin(x)*sin(y); 
}
////These are for comparing to FD arakawa results
//double left( double x, double y) {return sin(2.*M_PI*(x-hx/2.));}
//double right( double x, double y) {return y;}
//double jacobian( double x, double y) {return 2.*M_PI*cos(2.*M_PI*(x-hx/2.));}

int main()
{
    Timer t;
    unsigned n, Nx, Ny;
    cout << "Type n, Nx and Ny! \n";
    cin >> n >> Nx >> Ny;
    Grid2d<double> grid( 0, lx, 0, ly, n, Nx, Ny, dg::PER, dg::PER);
    //S2D<double > s2d( grid);
    DVec w2d = create::w2d( grid);
    cout << "# of 2d cells                     " << Nx*Ny <<endl;
    cout << "# of Legendre nodes per dimension "<< n <<endl;
    DVec lhs = evaluate ( left, grid), jac(lhs);
    DVec rhs = evaluate ( right,grid);
    const DVec sol = evaluate( jacobian, grid );
    DVec eins = evaluate( one, grid );
    cout<< setprecision(2);


    ArakawaX<dg::DMatrix, DVec> arakawa( grid);
    unsigned multi=20;
    t.tic(); 
    for( unsigned i=0; i<multi; i++)
        arakawa( lhs, rhs, jac);
    t.toc();
    cout << "\nArakawa took "<<t.diff()*1000/(double)multi<<"ms\n\n";

    cout << scientific;
    cout << "Mean     Jacobian is "<<blas2::dot( eins, w2d, jac)<<"\n";
    cout << "Mean rhs*Jacobian is "<<blas2::dot( rhs, w2d, jac)<<"\n";
    cout << "Mean   n*Jacobian is "<<blas2::dot( lhs, w2d, jac)<<"\n";
    blas1::axpby( 1., sol, -1., jac);
    cout << "Distance to solution "<<sqrt(blas2::dot( w2d, jac))<<endl; //don't forget sqrt when comuting errors

    //periocid bc       |  dirichlet in x per in y
    //n = 1 -> p = 2    |        1.5
    //n = 2 -> p = 1    |        1
    //n = 3 -> p = 3    |        3
    //n = 4 -> p = 3    |        3
    //n = 5 -> p = 5    |        5
    // quantities are all conserved to 1e-15 for periodic bc
    // for dirichlet bc these are not better conserved than normal jacobian

    return 0;
}
