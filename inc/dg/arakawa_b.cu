#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "evaluation.cuh"
#include "arakawa.cuh"
#include "blas.h"
#include "typedefs.cuh"

#include "timer.cuh"

using namespace std;
using namespace dg;

const double lx = M_PI;
const double ly = M_PI;
//const double lx = 1.;
//const double ly = 1.;


//choose some mean function (attention on lx and ly)
//THESE ARE NOT PERIODIC
double left( double x, double y) { return sin(x)*cos(y);}
double right( double x, double y){ return exp(0.1*(x+y)); }
/*
double left( double x, double y) {return sin(x)*exp(x-M_PI)*sin(y);}
double right( double x, double y) {return sin(x)*sin(y)*exp(y-M_PI);}
*/
double jacobian( double x, double y) 
{
    return exp( x-M_PI)*(sin(x)+cos(x))*sin(y) * exp(y-M_PI)*sin(x)*(sin(y) + cos(y)) - sin(x)*exp(x-M_PI)*cos(y) * cos(x)*sin(y)*exp(y-M_PI); 
}

/*
double left( double x, double y) {return sin(x)*cos(y);}
double right( double x, double y) {return cos(x)*sin(y);}
double jacobian( double x, double y) 
{
    return cos(x)*cos(y)*cos(x)*cos(y) - sin(x)*sin(y)*sin(x)*sin(y); 
}
*/
////These are for comparing to FD arakawa results
//double left( double x, double y) {return sin(2.*M_PI*(x-hx/2.));}
//double right( double x, double y) {return y;}
//double jacobian( double x, double y) {return 2.*M_PI*cos(2.*M_PI*(x-hx/2.));}

int main()
{
    Timer t;
    unsigned n, Nx, Ny;
    cout << "Type n, Nx and Ny! \n";
    cin >> n >> Nx >> Ny;
    Grid2d<double> grid( 0, lx, 0, ly, n, Nx, Ny, dg::PER, dg::PER);
    //S2D<double > s2d( grid);
    DVec w2d = create::w2d( grid);
    cout << "# of 2d cells                     " << Nx*Ny <<endl;
    cout << "# of Legendre nodes per dimension "<< n <<endl;
    DVec lhs = evaluate ( left, grid), jac(lhs);
    DVec rhs = evaluate ( right,grid);
    const DVec sol = evaluate( jacobian, grid );
    DVec eins = evaluate( one, grid );
    cout<< setprecision(2);


    ArakawaX<DVec> arakawa( grid);
    t.tic(); 
    for( unsigned i=0; i<20; i++)
        arakawa( lhs, rhs, jac);
    t.toc();
    cout << "\nArakawa took "<<t.diff()/0.02<<"ms\n\n";

    cout << scientific;
    cout << "Mean     Jacobian is "<<blas2::dot( eins, w2d, jac)<<"\n";
    cout << "Mean rhs*Jacobian is "<<blas2::dot( rhs, w2d, jac)<<"\n";
    cout << "Mean   n*Jacobian is "<<blas2::dot( lhs, w2d, jac)<<"\n";
    blas1::axpby( 1., sol, -1., jac);
    cout << "Distance to solution "<<sqrt(blas2::dot( w2d, jac))<<endl; //don't forget sqrt when comuting errors

    //periocid bc       |  dirichlet in x per in y
    //n = 1 -> p = 2    |        1.5
    //n = 2 -> p = 1    |        1
    //n = 3 -> p = 3    |        3
    //n = 4 -> p = 3    |        3
    //n = 5 -> p = 5    |        5
    // quantities are all conserved to 1e-15 for periodic bc
    // for dirichlet bc these are not better conserved than normal jacobian

    return 0;
}
