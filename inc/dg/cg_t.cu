#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "evaluation.cuh"
#include "cg.cuh"
#include "laplace.cuh"
#include "preconditioner.cuh"
#include "typedefs.cuh"

unsigned n = 3; //global relative error in L2 norm is O(h^P)
unsigned N = 200;  //more N means less iterations for same error

const double lx = 2.*M_PI;

const double eps = 1e-7; //# of pcg iterations increases very much if 
 // eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus


typedef dg::T1D<double> Preconditioner;


double sine(double x){ return sin( x);}
double initial( double x) {return sin(0);}
dg::bc bcx = dg::PER;

using namespace std;
int main()
{
    cout << "Type n and N\n";
    cin >> n >> N;
    dg::Grid1d<double > g( 0, lx, n, N, bcx);
    dg::HVec x = dg::expand( initial, g);
    dg::DMatrix A = dg::create::laplace1d( g); 

    dg::CG< dg::DVec > cg( x, x.size());
    dg::HVec b = dg::expand ( sine, g);
    dg::HVec error(b);
    const dg::HVec solution(b);

    //copy data to device memory
    dg::DVec dx( x), db( b), derror( error);
    const dg::DVec dsolution( solution);

    cout << "# of polynomial coefficients: "<< n <<endl;
    cout << "# of intervals                "<< N <<endl;
    //compute S b
    dg::blas2::symv( dg::S1D<double>(g), db, db);
    std::cout << "Number of pcg iterations "<< cg( A, dx, db, Preconditioner(g), eps)<<endl;
    std::cout << "Number of cg iterations "<< cg( A, dx, db, eps)<<endl;
    cout << "For a precision of "<< eps<<endl;
    //compute error
    dg::blas1::axpby( 1.,dx,-1.,derror);
    /*
    //and Ax
    DArrVec dbx(dx);
    dg::blas2::symv(  A, dx.data(), dbx.data());

    cout<< dx <<endl;
    */

    double eps = dg::blas2::dot( dg::S1D<double>(g), derror);
    cout << "L2 Norm2 of Error is " << eps << endl;
    double norm = dg::blas2::dot( dg::S1D<double>(g), dsolution);
    std::cout << "L2 Norm of relative error is "<<sqrt( eps/norm)<<std::endl;
    //Fehler der Integration des Sinus ist vernachlässigbar (vgl. evaluation_t)



    return 0;
}
