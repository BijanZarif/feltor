#include "hip/hip_runtime.h"
#include <iostream>

#include "blas.h"

#include "helmholtz.h"
#include "xspacelib.cuh"
#include "multistep.h"

#include "cg.h"
template< class container>
struct Diffusion
{
    Diffusion( const dg::Grid2d<double>& g, double nu):
        w2d( dg::create::w2d( g)), v2d( dg::create::v2d(g)) { 
        dg::Matrix Laplacian_ = dg::create::laplacianM( g, dg::normed, dg::XSPACE); 
        cusp::blas::scal( Laplacian_.values, -nu);
        Laplacian = Laplacian_;
        }
    void operator()( const container& x, container& y)
    {
        //dg::blas1::axpby( 0., x, 0., y);
        dg::blas2::gemv( Laplacian, x, y);
    }
    const container& weights(){return w2d;}
    const container& precond(){return v2d;}
  private:
    const container w2d, v2d;
    dg::DMatrix Laplacian;
};

const double eps = 1e-4;
const double alpha = -0.5; 
double lhs( double x, double y){ return sin(x)*sin(y);}
double rhs( double x, double y){ return (1.-2.*alpha)*sin(x)*sin(y);}
//double rhs( double x, double y){ return lhs(x,y);}
int main()
{
    
    unsigned n, Nx, Ny; 
    std::cout << "Type n, Nx and Ny\n";
    std::cin >> n>> Nx >> Ny;
    dg::Grid2d<double> grid( 0, 2.*M_PI, 0, 2.*M_PI, n, Nx, Ny, dg::DIR, dg::PER);
    const dg::DVec w2d = dg::create::w2d( grid);
    const dg::DVec v2d = dg::create::v2d( grid);
    const dg::DVec rho = dg::evaluate( rhs, grid);
    const dg::DVec sol = dg::evaluate( lhs, grid);
    dg::DVec x(rho.size(), 0.), rho_(rho);

    dg::DMatrix A = dg::create::laplacianM( grid, dg::normed, dg::XSPACE); 
    dg::GammaInv< dg::DMatrix, dg::DVec > gamma1inv( A, w2d, v2d, alpha);

    std::cout << "FIRST METHOD:\n";
    dg::CG< dg::DVec > cg(x, x.size());
    dg::blas2::symv( w2d, rho, rho_);
    unsigned number = cg( gamma1inv, x, rho_, v2d, eps);

    std::cout << "SECOND METHOD:\n";
    dg::Helmholtz2d <dg::DVec> diff( w2d, grid.size(), eps);
    dg::Maxwell< dg::DMatrix, dg::DVec > maxwell( A, dg::DVec(grid.size(), 1.),w2d, v2d, alpha);
    dg::DVec x_(rho.size(), 0.);
    diff( gamma1inv, x_, rho);

    std::cout << "THIRD METHOD:\n";
    dg::DVec x__(rho.size(), 0.);
    Diffusion<dg::DVec> diffusion( grid, 1.);
    dg::detail::Implicit<Diffusion<dg::DVec> > implicit( alpha, diffusion);
    dg::blas2::symv( diffusion.weights(), rho, rho_);
    number = cg( implicit, x__, rho_, diffusion.precond(), eps);

    //Evaluation
    dg::blas1::axpby( 1., sol, -1., x);
    dg::blas1::axpby( 1., sol, -1., x_);
    dg::blas1::axpby( 1., sol, -1., x__);

    std::cout << "number of iterations:  "<<number<<std::endl;
    std::cout << "error1 " << sqrt( dg::blas2::dot( w2d, x))<<std::endl;
    std::cout << "error2 " << sqrt( dg::blas2::dot( w2d, x_))<<std::endl;
    std::cout << "error3 " << sqrt( dg::blas2::dot( w2d, x__))<<std::endl;




    return 0;
}



