#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "backend/timer.cuh"
#include "blas.h"
#include "backend/derivatives.h"
#include "backend/derivativesX.h"
#include "backend/evaluation.cuh"

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
double function(double x, double y){ return sin(y)*sin(x);}

int main()
{
    dg::Timer t;
    unsigned n, Nx, Ny; 
    std::cout << "Type n, Nx and Ny\n";
    std::cin >> n >> Nx >> Ny;
    dg::Grid2d<double> grid( 0., lx, 0, ly, n, Nx, Ny);
    dg::GridX2d gridX( 0., lx, 0, ly, 0.2, 0., n, Nx, Ny);
    const dg::DVec w2d = dg::create::weights( grid);
    std::cout<<"Evaluate a function on the grid\n";
    t.tic();
    dg::DVec x = dg::evaluate( function, grid);
    t.toc();
    std::cout<<"Evaluation of a function took    "<<t.diff()<<"s\n";
    double gbytes=x.size()*8/1e9;
    t.tic();
    double norm = dg::blas2::dot( w2d, x);
    t.toc();
    std::cout<<"DOT took                         " <<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";
    dg::DVec y(x);
    dg::DMatrix M = dg::create::dx( grid, dg::centered);
    t.tic();
    dg::blas2::symv( M, x, y);
    t.toc();
    std::cout<<"centered x derivative took       "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    M = dg::create::dx( grid, dg::forward);
    t.tic();
    dg::blas2::symv( M, x, y);
    t.toc();
    std::cout<<"forward x derivative took        "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    M = dg::create::dy( grid, dg::forward);
    t.tic();
    dg::blas2::symv( M, x, y);
    t.toc();
    std::cout<<"forward y derivative took        "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    M = dg::create::dy( grid, dg::centered);
    t.tic();
    dg::blas2::symv( M, x, y);
    t.toc();
    std::cout<<"centered y derivative took       "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    M = dg::create::jumpX( grid);
    t.tic();
    dg::blas2::symv( M, x, y);
    t.toc();
    std::cout<<"jump X took                      "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    t.tic();
    dg::blas1::axpby( 1., y, -1., x);
    t.toc();
    std::cout<<"AXPBY took                       "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    t.tic();
    dg::blas1::pointwiseDot( y, x, x);
    t.toc();
    std::cout<<"pointwiseDot took                "<<t.diff()<<"s\t" <<x.size()*8/1e9/t.diff()<<"GB/s\n";
    t.tic();
    norm = dg::blas2::dot( w2d, y);
    t.toc();
    std::cout<<"DOT(w,y) took                    " <<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    t.tic();
    norm = dg::blas2::dot( x, w2d, y);
    t.toc();
    std::cout<<"DOT(x,w,y) took                  " <<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";
    norm++;//get rid of compiler warning

    dg::Composite<dg::DMatrix> Mat = dg::create::dx( gridX, dg::centered);
    t.tic();
    dg::blas2::symv( Mat, x, y);
    t.toc();
    std::cout<<"centered x Xderivative took       "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    Mat = dg::create::dx( gridX, dg::forward);
    t.tic();
    dg::blas2::symv( Mat, x, y);
    t.toc();
    std::cout<<"forward x Xderivative took        "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    Mat = dg::create::dy( gridX, dg::forward);
    t.tic();
    dg::blas2::symv( Mat, x, y);
    t.toc();
    std::cout<<"forward y Xderivative took        "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    Mat = dg::create::dy( gridX, dg::centered);
    t.tic();
    dg::blas2::symv( Mat, x, y);
    t.toc();
    std::cout<<"centered y Xderivative took       "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";

    Mat = dg::create::jumpX( gridX);
    t.tic();
    dg::blas2::symv( Mat, x, y);
    t.toc();
    std::cout<<"jump X took                      "<<t.diff()<<"s\t"<<gbytes/t.diff()<<"GB/s\n";


    return 0;
}
