#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/print.h>

#include "evaluation.cuh"
#include "dz.cuh"
#include "functions.h"
#include "../blas2.h"
#include "../functors.h"
#include "interpolation.cuh"


struct Field
{
    Field( double R_0, double I_0):R_0(R_0), I_0(I_0){}
    void operator()( const std::vector<dg::HVec>& y, std::vector<dg::HVec>& yp)
    {
        for( unsigned i=0; i<y[0].size(); i++)
        {
            double gradpsi = ((y[0][i]-R_0)*(y[0][i]-R_0) + y[1][i]*y[1][i])/I_0/I_0;
            yp[2][i] = y[0][i]*sqrt(1 + gradpsi);
            yp[0][i] = y[0][i]*y[1][i]/I_0;
            yp[1][i] = -y[0][i]*y[0][i]/I_0 + R_0/I_0*y[0][i] ;
        }
    }
    private:
    double R_0, I_0;
};

double R_0 = 10;
double I_0 = 40;
double func(double R, double Z, double phi)
{
    double r2 = (R-R_0)*(R-R_0)+Z*Z;
    return r2*sin(phi);
}
double deri(double R, double Z, double phi)
{
    double r2 = (R-R_0)*(R-R_0)+Z*Z;
    return I_0/R/sqrt(I_0*I_0 + r2)* r2*cos(phi);
}


int main()
{
    Field field( R_0, I_0);
    std::cout << "Type n, Nx, Ny, Nz\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;
    dg::Grid3d<double> g3d( R_0 - 1, R_0+1, -1, 1, 0, 2.*M_PI, n, Nx, Ny, Nz);
    const dg::DVec w3d = dg::create::w3d( g3d);
    dg::DZ<dg::DVec> dz( field, g3d);

    dg::DVec function = dg::evaluate( func, g3d), derivative(function);
    const dg::DVec solution = dg::evaluate( deri, g3d);
    dz( function, derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    double norm = dg::blas2::dot( w3d, solution);
    std::cout << "Norm Solution "<<sqrt( norm)<<"\n";
    std::cout << "Relative Difference Is "<< sqrt( dg::blas2::dot( derivative, w3d, derivative)/norm )<<"\n";


    
    return 0;
}
