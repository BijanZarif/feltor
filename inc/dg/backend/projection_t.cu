#include "hip/hip_runtime.h"
#include <iostream>
#include <cusp/print.h>
#include "typedefs.cuh"
#include "projection.cuh"
#include "evaluation.cuh"
#include "blas.h"

double sine( double x){ return sin(x);}
double sine( double x, double y){return sin(x)*sin(y);}

int main()
{
    //Projection might not be correct any more due to layout change
    std::cout << "TEST 1D\n";
    unsigned n_old = 4, n_new = 3, N = 10, Nf = 1;
    dg::Grid1d<double> go ( 0, M_PI, n_old, N);
    dg::Grid1d<double> gn ( 0, M_PI, n_new, N*Nf);
    cusp::coo_matrix<int, double, cusp::host_memory> proj = dg::create::projection1d( go, gn);
    thrust::host_vector<double> v = dg::evaluate( sine, go);
    thrust::host_vector<double> w1do = dg::create::weights( go);
    thrust::host_vector<double> w1dn = dg::create::weights( gn);
    dg::HVec oneo( go.size(), 1.);
    dg::HVec onen( gn.size(), 1.);
    thrust::host_vector<double> w( gn.size());
    dg::blas2::gemv( proj, v, w);
    std::cout << "Original vector  "<<dg::blas2::dot( oneo, w1do, v) << "\n";
    std::cout << "Projected vector "<<dg::blas2::dot( onen, w1dn, w) << "\n";
    std::cout << "Difference       "<<dg::blas2::dot( oneo, w1do, v) - dg::blas2::dot( onen, w1dn, w) << "\n"<<std::endl;

    /*
    std::cout << "TEST KRONECKER PRODUCT\n";
    dg::create::detail::HelperMatrix<double> m1(2,2), m2(2,2);
    for( unsigned i=0; i<2;i++)
        for( unsigned j=0; j<2;j++)
            m1(i,j) = 2*i+j+1;
    m2(0,0) = 0; m2( 0, 1) = 5; m2( 1,0) = 6, m2( 1,1) = 7;
    std::cout << "M1 \n"<<m1 << "Times\n"<<m2;
    std::cout << "Is \n"<<dg::create::detail::kronecker( m1, m2);
    std::cout << "(Compare Wikipedia for correctness!)\n"<<std::endl;
    */
    /*
    std::cout << "TEST GCD AND LCM\n";
    std::cout << "gcd of 1071 and 462 is "<<dg::gcd( 1071, 462)<<" (21)\n";
    std::cout << "lcm of 1071 and 462 is "<<dg::lcm( 1071, 462)<<" (23562)\n"<<std::endl;
    */

    std::cout << "TEST 2D\n";
    n_old = 7, n_new = 3, N = 20, Nf = 1;
    dg::Grid2d<double> g2o (0, M_PI, 0, M_PI, n_old, N, N);
    dg::Grid2d<double> g2n (0, M_PI, 0, M_PI, n_new, N, N*Nf);
    cusp::coo_matrix<int, double, cusp::host_memory> proj2d = dg::create::projection2d( g2o, g2n);
    const dg::HVec sinO = dg::evaluate( sine, g2o), 
                   sinN = dg::evaluate( sine, g2n);
    dg::HVec w2do = dg::create::weights( g2o);
    dg::HVec w2dn = dg::create::weights( g2n);
    dg::HVec sinP( g2n.size());
    dg::blas2::gemv( proj2d, sinO, sinP);
    std::cout << "Original vector  "<<dg::blas2::dot( sinO, w2do, sinO) << "\n";
    std::cout << "Projected vector "<<dg::blas2::dot( sinP, w2dn, sinP) << "\n";
    std::cout << "Evaluated vector "<<dg::blas2::dot(sinN, w2dn, sinN) << "\n";
    std::cout << "Difference       "<<dg::blas2::dot( sinO, w2do, sinO) - dg::blas2::dot( sinP, w2dn, sinP) << "\n" << std::endl;

    std::cout << "TEST OF DIFFERENCE\n";
    dg::DifferenceNorm<dg::HVec> diff( g2o, g2n);
    std::cout << "Information loss due to projection:\n";
    std::cout << diff( sinO, sinP)<<" (should converge to zero) \n";
    std::cout << "Difference between two grid evaluations:\n";
    std::cout << diff( sinO, sinN)<<" (should converge to zero!) \n";
    std::cout << "Difference between projection and evaluation      \n";
    dg::blas1::axpby( 1., sinN, -1., sinP);
    std::cout << dg::blas2::dot( sinP, w2dn, sinP)<<" (smaller than above)\n";


    return 0;
}
