#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/ell_matrix.h>

#include "blas.h"
#include "dx.cuh"
#include "evaluation.cuh"
#include "typedefs.cuh"
#include "weights.cuh"


unsigned n = 3;
unsigned N = 40;
const double lx = 2*M_PI;

/*
double function( double x) { return sin(x);}
double derivative( double x) { return cos(x);}
bc bcx = PER;
double function (double  x) {return x*(x-2*M_PI)*exp(x);}
double derivative( double x) { return (2.*x-2*M_PI)*exp(x) + function(x);}
bc bcx = DIR;
*/
/*
double function( double x) { return cos(x);}
double derivative( double x) { return -sin(x);}
bc bcx = NEU;
*/
double function( double x) { return sin(3./4.*x);}
double derivative( double x) { return 3./4.*cos(3./4.*x);}
dg::bc bcx = dg::DIR_NEU;
/*
double function( double x) { return cos(3./4.*x);}
double derivative( double x) { return -3./4.*sin(3./4.*x);}
bc bcx = NEU_DIR;
*/

int main ()
{
    std::cout << "Note the supraconvergence!\n";
    std::cout << "Type in n an Nx!\n";
    std::cin >> n>> N;
    std::cout << "# of Legendre nodes " << n <<"\n";
    std::cout << "# of cells          " << N <<"\n";
    dg::Grid1d<double> g( 0, lx, n, N);
    const double hx = lx/(double)N;
    //cusp::ell_matrix< int, double, cusp::host_memory> hm = dg::create::dx_symm_normed<double>( n, N, hx, bcx);
    //cusp::ell_matrix< int, double, cusp::host_memory> hm = dg::create::dx_minus_normed<double>( n, N, hx, bcx);
    cusp::ell_matrix< int, double, cusp::host_memory> hm = dg::create::dx_plus_normed<double>( n, N, hx, bcx);
    dg::HVec hv = dg::evaluate( function, g);
    dg::HVec hw = hv;
    const dg::HVec hu = dg::evaluate( derivative, g);

    dg::blas2::symv( hm, hv, hw);
    dg::blas1::axpby( 1., hu, -1., hw);
    
    std::cout << "Distance to true solution: "<<sqrt(dg::blas2::dot( dg::create::weights(g), hw) )<<"\n";
    //for periodic bc | dirichlet bc
    //n = 1 -> p = 2      2
    //n = 2 -> p = 1      1
    //n = 3 -> p = 3      3
    //n = 4 -> p = 3      3
    //n = 5 -> p = 5      5


    
    return 0;
}
