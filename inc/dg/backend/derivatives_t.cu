#include "hip/hip_runtime.h"
#include <iostream>
#include "blas.h"
#include "derivatives.h"
#include "evaluation.cuh"
#include "typedefs.cuh"
#include "sparseblockmat.cuh"

const double lx = 2*M_PI;
/*
double function( double x, double y, double z) { return sin(3./4.*z);}
double derivative( double x, double y, double z) { return 3./4.*cos(3./4.*z);}
dg::bc bcz = dg::DIR_NEU;
*/
double function(   double x, double y, double z) { return sin(x);}
double derivative( double x, double y, double z) { return cos(x);}
double siny(   double x, double y, double z) { return sin(y);}
double cosy(   double x, double y, double z) { return cos(y);}
double sinz(   double x, double y, double z) { return sin(z);}
double cosz(   double x, double y, double z) { return cos(z);}
dg::bc bcx = dg::PER;
dg::bc bcy = dg::PER;
dg::bc bcz = dg::PER;


int main()
{
    unsigned n, Nx, Ny, Nz;
    std::cout << "Note the supraconvergence!\n";
    std::cout << "Type in n, Nx and Ny and Nz!\n";
    std::cin >> n >> Nx >> Ny >> Nz;
    dg::Grid3d<double> g( 0, lx, 0, lx, 0., lx, n, Nx, Ny, Nz, bcx, bcy, bcz);
    //dg::Grid2d<double> g( 0, lx, 0, lx, n, Nx, Ny, bcx, dg::PER);
    dg::SparseBlockMatDevice dx = dg::create::dx( g, bcx, dg::centered);
    dg::DVec v = dg::evaluate( function, g);
    dg::DVec w = v;
    const dg::DVec u = dg::evaluate( derivative, g);

    const dg::DVec w3d = dg::create::weights( g);
    dg::blas2::symv( dx, v, w);
    dg::blas1::axpby( 1., u, -1., w);
    std::cout << "DX(symm):  Distance to true solution: "<<sqrt(dg::blas2::dot(w, w3d, w))<<"\n";
    //for periodic bc | dirichlet bc
    //n = 1 -> p = 2      2
    //n = 2 -> p = 1      1
    //n = 3 -> p = 3      3
    //n = 4 -> p = 3      3
    //n = 5 -> p = 5      5

    std::cout << "TEST DY and DZ\n";
    {
    const dg::DVec func = dg::evaluate( siny, g);
    const dg::DVec deri = dg::evaluate( cosy, g);

    dg::SparseBlockMatDevice dy = dg::create::dy( g); 
    dg::DVec temp( func);
    dg::blas2::gemv( dy, func, temp);
    dg::blas1::axpby( 1., deri, -1., temp);
    std::cout << "DY(symm):  Distance to true solution: "<<sqrt(dg::blas2::dot(temp, w3d, temp))<<"\n";
    }
    {
    const dg::DVec func = dg::evaluate( sinz, g);
    const dg::DVec deri = dg::evaluate( cosz, g);

    dg::SparseBlockMatDevice dz = dg::create::dz( g); 
    dg::DVec temp( func);
    dg::blas2::gemv( dz, func, temp);
    dg::blas1::axpby( 1., deri, -1., temp);
    std::cout << "DZ(symm):  Distance to true solution: "<<sqrt(dg::blas2::dot(temp, w3d, temp))<<"\n";
    }
    return 0;
}
