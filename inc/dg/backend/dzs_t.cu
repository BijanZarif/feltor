#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/print.h>

#include "evaluation.cuh"
#include "dzs.cuh"
#include "functions.h"
#include "../blas2.h"
#include "../functors.h"
#include "../cg.h"
#include "interpolation.cuh"

struct InvB
{
    InvB( double R_0, double I_0):R_0(R_0), I_0(I_0){}
    double operator()( double R, double Z, double phi)
    {
        return 2.*sqrt(2.)*R/sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(R-R_0))*cos(M_PI*Z))/R_0;
    }
    private:
    double R_0, I_0;
};
struct B
{
    B( double R_0, double I_0):R_0(R_0), I_0(I_0){}
    double operator()( double R, double Z, double phi)
    {
        return R_0*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(R-R_0))*cos(M_PI*Z))/2./sqrt(2.)/R;
    }
    private:
    double R_0, I_0;
};
struct LnB
{
    LnB( double R_0, double I_0):R_0(R_0), I_0(I_0){}
    double operator()( double R, double Z, double phi)
    {
        double invB = 2.*sqrt(2.)*R/sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(R-R_0))*cos(M_PI*Z))/R_0;
        return log(1./invB);
    }
    private:
    double R_0, I_0;
};

struct Field
{
    Field( double R_0, double I_0):R_0(R_0), I_0(I_0){}
    void operator()( const std::vector<dg::HVec>& y, std::vector<dg::HVec>& yp)
    {
        for( unsigned i=0; i<y[0].size(); i++)
        {
        double B = R_0*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(y[0][i]-R_0))*cos(M_PI*y[1][i]))/2./sqrt(2.)/y[0][i];
        double dldp=y[0][i]*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(y[0][i]-R_0))*cos(M_PI*y[1][i]))/2./sqrt(2)/I_0;
        yp[2][i] = B/dldp;            
        yp[0][i] = -B*M_PI*y[0][i]*cos(M_PI*(y[0][i]-R_0)/2.)*sin(M_PI*y[1][i]/2)/2./I_0/dldp;
        yp[1][i] =  B*M_PI*y[0][i]*sin(M_PI*(y[0][i]-R_0)/2.)*cos(M_PI*y[1][i]/2)/2./I_0/dldp ;
        }
    }
    void operator()( const dg::HVec& y, dg::HVec& yp)
    {
        double B = R_0*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(y[0]-R_0))*cos(M_PI*y[1]))/2./sqrt(2.)/y[0];
        double dldp = y[0]*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(y[0]-R_0))*cos(M_PI*y[1]))/2./sqrt(2.)/I_0;
        yp[2] = B/dldp;            
        yp[0] = -B*M_PI*y[0]*cos(M_PI*(y[0]-R_0)/2.)*sin(M_PI*y[1]/2)/2./I_0/dldp;
        yp[1] =  B*M_PI*y[0]*sin(M_PI*(y[0]-R_0)/2.)*cos(M_PI*y[1]/2)/2./I_0/dldp ;
    }
    private:
    double R_0, I_0;
};

struct FieldP
{
    FieldP( double R_0, double I_0):R_0(R_0), I_0(I_0){}
    void operator()( const std::vector<dg::HVec>& y, std::vector<dg::HVec>& yp)
    {
        for( unsigned i=0; i<y[0].size(); i++)
        {
        double B = R_0*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(y[0][i]-R_0))*cos(M_PI*y[1][i]))/2./sqrt(2.)/y[0][i];
        double dldp=y[0][i]*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(y[0][i]-R_0))*cos(M_PI*y[1][i]))/2./sqrt(2)/I_0;
        yp[2][i] = B*dldp;            
        yp[0][i] = -B*M_PI*y[0][i]*cos(M_PI*(y[0][i]-R_0)/2.)*sin(M_PI*y[1][i]/2)/2./I_0;
        yp[1][i] =  B*M_PI*y[0][i]*sin(M_PI*(y[0][i]-R_0)/2.)*cos(M_PI*y[1][i]/2)/2./I_0 ;
        }
    }
    void operator()( const dg::HVec& y, dg::HVec& yp)
    {
        double B = R_0*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(y[0]-R_0))*cos(M_PI*y[1]))/2./sqrt(2.)/y[0];
        double dldp = y[0]*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(y[0]-R_0))*cos(M_PI*y[1]))/2./sqrt(2.)/I_0;
        yp[2] = B*dldp;            
        yp[0] = -B*M_PI*y[0]*cos(M_PI*(y[0]-R_0)/2.)*sin(M_PI*y[1]/2)/2./I_0;
        yp[1] =  B*M_PI*y[0]*sin(M_PI*(y[0]-R_0)/2.)*cos(M_PI*y[1]/2)/2./I_0 ;
    }
    private:
    double R_0, I_0;
};
double R_0 = 10;
double I_0 = 20; //I0=20 and R=10 means q=2

double divb(double R, double Z, double phi)
{
    double fac1 = sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(R-R_0))*cos(M_PI*Z));
    double z1 = cos(M_PI*0.5*(R-R_0))*(32.*I_0*I_0+5.*M_PI*M_PI)+
                M_PI*M_PI* cos(M_PI*3.*(R-R_0)/2.)+
                M_PI*R*sin(M_PI*3.*(R-R_0)/2.) ;
    double z2 = cos(M_PI*0.5*(R-R_0)) + 
                cos(M_PI*3*(R-R_0)/2) + 
                M_PI*R*sin(M_PI*0.5*(R-R_0));
    double nenner = fac1*fac1*fac1*2.*sqrt(2.)*R;
    double divb = -M_PI*(z1*sin(M_PI*Z*0.5)-z2*M_PI*M_PI*sin(M_PI*Z*3./2.))/(nenner);
    return divb;
}
double funcadj(double R, double Z, double phi)
{
    double psi = cos(M_PI*0.5*(R-R_0))*cos(M_PI*Z*0.5);
    return psi*(R*R*sin(Z*Z)+Z*Z*cos(R*Z*phi));
}
double funcNEU(double R, double Z, double phi)
{
    double psi = cos(M_PI*0.5*(R-R_0))*cos(M_PI*Z*0.5);
    return -psi*cos(phi);

}
double deriNEU(double R, double Z, double phi)
{
    double dldp = R*sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(R-R_0))*cos(M_PI*Z))/2./sqrt(2.)/I_0;
    double psi = cos(M_PI*0.5*(R-R_0))*cos(M_PI*Z*0.5);
    double invB = 2.*sqrt(2.)*R/sqrt(8.*I_0*I_0+ M_PI*M_PI-M_PI*M_PI* cos(M_PI*(R-R_0))*cos(M_PI*Z))/R_0;
    return psi*sin(phi)/dldp/invB;
    
}
double cut(double R, double Z, double phi)
{
    double psip = 0.5*((R-R_0)*(R-R_0)+Z*Z);
    if (psip >0.5*((0.9)*(0.9))) return 0.;
    if (psip <0.5*((0.1)*(0.1))) return 0.;
    return 1.;
}
int main()
{
    Field field( R_0, I_0);
    InvB invb(R_0, I_0);
    B Bfield(R_0, I_0);
    LnB lnB(R_0, I_0);

    std::cout << "Type n, Nx, Ny, Nz\n";
    //std::cout << "Note, that function is resolved exactly in R,Z for n > 2\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;
    std::cout << "q = " << I_0/R_0 << std::endl;
    double z0 = 0, z1 = 2.*M_PI;
    //double z0 = M_PI/2., z1 = 3./2.*M_PI;
    double bscale=5.0;
    double Rmin=R_0 - bscale;
    double Rmax=R_0 + bscale;
    double Zmin= - bscale;
    double Zmax= +bscale;
    dg::Grid3d<double> g3d( Rmin,Rmax, Zmin, Zmax, z0, z1,  n, Nx, Ny, Nz,dg::NEU, dg::NEU, dg::PER,dg::cylindrical);
    
    dg::DVec cutongrid = dg::evaluate( cut, g3d);

    const dg::DVec w3d = dg::create::weights( g3d);
    const dg::DVec v3d = dg::create::inv_weights( g3d);


// double hs = g3d.hz();
    double hs = g3d.hz()*(R_0+1)*(sqrt(2)/I_0/2);

    std::cout << "hz = " <<  g3d.hz() << std::endl;
    std::cout << "hs = " << hs << std::endl;
    dg::DZ<dg::DMatrix, dg::DVec> dzs( field, g3d, hs, 1e-4, dg::DefaultLimiter(), dg::NEU);
    dg::DVec func = dg::evaluate(funcNEU, g3d),dzsf(func),dzsfa(func);
    const dg::DVec funca = dg::evaluate(funcadj, g3d);
    const dg::DVec soldzsf = dg::evaluate(deriNEU, g3d);
    const dg::DVec Bfeld = dg::evaluate(Bfield, g3d);
    dg::DVec one = dg::evaluate( dg::one, g3d),dzsone(one),dzsTone(one),dzsTB(one);
    dzs.set_boundaries( dg::PER, 0, 0);

    dzs( one, dzsone); //dz(f)
    dzs( func, dzsf); //dz(f)
    dzs( funca, dzsfa); //dz(f)
    dzs.centeredT( one, dzsTone); //dz(f)
    dzs.centeredT( Bfeld, dzsTB); //dz(f)
    //cut
//     dg::blas1::pointwiseDot(cutongrid,dzsone,dzsone);
//     dg::blas1::pointwiseDot(cutongrid,dzsf,dzsf);
//     dg::blas1::pointwiseDot(cutongrid,dzsTone,dzsTone);
//     dg::blas1::pointwiseDot(cutongrid,func,func);

    double normdzsone  =dg::blas2::dot(dzsone, w3d,dzsone);
    double normdzsTone =dg::blas2::dot(dzsTone, w3d,dzsTone);
    double normdzsTB =dg::blas2::dot(dzsTB, w3d,dzsTB);
    double normonedzsf = dg::blas2::dot(one, w3d,dzsfa);
    double normfdzsone = dg::blas2::dot(funca, w3d,dzsone);
    double normfdzsTone = dg::blas2::dot(funca, w3d,dzsTone);
    
    std::cout << "--------------------testing dzs" << std::endl;
    double normsoldzsf = dg::blas2::dot( w3d, soldzsf);
    std::cout << "|| Solution ||   "<<sqrt( normsoldzsf)<<"\n";
    double errdzsf =dg::blas2::dot( w3d, dzsf);
    std::cout << "|| Derivative || "<<sqrt( errdzsf)<<"\n";
    dg::blas1::axpby( 1.,soldzsf, -1.,dzsf);
    errdzsf=dg::blas2::dot( w3d,dzsf);
    std::cout << "Relative Difference in dzs is "<< sqrt( errdzsf/normsoldzsf )<<"\n"; 
    
    std::cout << "--------------------testing dzs and dzsT " << std::endl;
    std::cout << "|| dzs(1) ||      "<<sqrt( normdzsone)<<"\n";
    std::cout << "|| dzsT(1) ||      "<<sqrt( normdzsTone)<<"\n";
    std::cout << "|| dzsT(B) ||      "<<sqrt( normdzsTB)<<"\n";
    
    std::cout << "--------------------testing adjointness " << std::endl;
    std::cout << "<1,dzs(f)>   = "<< normonedzsf <<"\n";
    std::cout << "<f,dzs(1)>   = "<< normfdzsone <<"\n";
    std::cout << "-<dzsT(1),f> = "<< -normfdzsTone<<"\n";
    std::cout << "Diff         = "<< normonedzsf+normfdzsTone<<"\n";   
    std::cout << "-------------------- " << std::endl;
    
    return 0;

}
