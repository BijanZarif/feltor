#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <mpi.h>

#include "blas.h"

#include "mpi_evaluation.h"
#include "mpi_derivatives.h"
//#include "mpi_matrix.h"
#include "mpi_precon.h"
#include "mpi_init.h"
#include "../average.h"
#include "timer.cuh"

const double lx = 2.*M_PI;
const double ly = M_PI;

double function( double x, double y) {return cos(x)*sin(y);}
double pol_average( double x, double y) {return cos(x)*2./M_PI;}

dg::bc bcx = dg::PER; 
dg::bc bcy = dg::PER;

int main(int argc, char* argv[])
{
    MPI_Init( &argc, &argv);
    int rank;
    unsigned n, Nx, Ny; 

    MPI_Comm comm;
    mpi_init2d( bcx, bcy, n, Nx, Ny, comm);
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);

    dg::MPI_Grid2d g( 0, lx, 0, ly, n, Nx, Ny, bcx, bcy, comm);
    dg::Timer t;
 

    std::cout << "constructing polavg" << std::endl;
    dg::PoloidalAverage<dg::MHVec,dg::HVec > pol(g);
    std::cout << "constructing polavg end" << std::endl;
    dg::MHVec vector = dg::evaluate( function ,g), average_y( vector);
    const dg::MHVec solution = dg::evaluate( pol_average, g);
    std::cout << "Averaging ... \n";
    t.tic();
    pol( vector, average_y);
    t.toc();
    std::cout << "Assembly of average vector took:      "<<t.diff()<<"s\n";

    dg::blas1::axpby( 1., solution, -1., average_y, vector);
    std::cout << "Distance to solution is: "<<        sqrt(dg::blas2::dot( vector, dg::create::weights( g), vector))<<std::endl;

    MPI_Finalize();
    return 0;
}
