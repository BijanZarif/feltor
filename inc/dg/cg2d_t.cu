#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "evaluation.cuh"
#include "cg.cuh"
#include "tensor.cuh"
#include "derivatives.cuh"
#include "typedefs.cuh"

const unsigned n = 3; //global relative error in L2 norm is O(h^P)
const unsigned Nx = 20;  //more N means less iterations for same error
const unsigned Ny = 20;  //more N means less iterations for same error
const double lx = 2.*M_PI;
const double ly = 2.*M_PI;

const double eps_ = 1e-6; //# of pcg iterations increases very much if 
 // eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus

double fct(double x, double y){ return sin(y)*sin(x);}
double laplace_fct( double x, double y) { return 2*sin(y)*sin(x);}
double initial( double x, double y) {return sin(0);}
double fct(double x, double y, double z){ return sin(y)*sin(x);}
double laplace_fct( double x, double y, double z) { return 2*sin(y)*sin(x);}
double initial( double x, double y, double z) {return sin(0);}

using namespace std;

int main()
{
    dg::Grid2d<double> grid( 0, lx, 0, ly,n, Nx, Ny, dg::PER, dg::PER);
    //dg::S2D<double > s2d( grid);
    //dg::T2D<double > t2d( grid);
    dg::HVec s2d = dg::create::s2d( grid);
    dg::HVec t2d = dg::create::t2d( grid);
    cout<<"Expand initial condition\n";
    dg::HVec x = dg::expand( initial, grid);

    cout << "Create Laplacian\n";
    dg::HMatrix A = dg::create::laplacianM( grid, dg::not_normed, dg::LSPACE, dg::symmetric); 
    dg::CG<dg::HVec > pcg( x, n*n*Nx*Ny);
    cout<<"Expand right hand side\n";
    dg::HVec b = dg::expand ( laplace_fct, grid);
    const dg::HVec solution = dg::expand ( fct, grid);
    //////////////////////////////////////////////////////////////////////
    cout << "# of polynomial coefficients: "<< n <<endl;
    cout << "# of 2d cells                 "<< Nx*Ny <<endl;
    //compute S b
    dg::blas2::symv( s2d, b, b);
    cout << "Number of pcg iterations "<< pcg( A, x, b, t2d, eps_)<<endl;
    //std::cout << "Number of cg iterations "<< pcg( A, x, b, dg::Identity<double>(), eps)<<endl;
    cout << "For a precision of "<< eps_<<endl;
    //compute error
    dg::HVec error( solution);
    dg::blas1::axpby( 1.,x,-1.,error);

    dg::HVec Ax(x), res( b);
    dg::blas2::symv(  A, x, Ax);
    dg::blas1::axpby( 1.,Ax,-1.,res);

    double xnorm = dg::blas2::dot( s2d, x);
    cout << "L2 Norm2 of x0 is              " << xnorm << endl;
    double eps = dg::blas2::dot(s2d , error);
    cout << "L2 Norm2 of Error is           " << eps << endl;
    double norm = dg::blas2::dot(s2d , solution);
    cout << "L2 Norm2 of Solution is        " << norm << endl;
    double normres = dg::blas2::dot( s2d, res);
    cout << "L2 Norm2 of Residuum is        " << normres << endl;
    cout << "L2 Norm of relative error is   " <<sqrt( eps/norm)<<endl;
    //Fehler der Integration des Sinus ist vernachlässigbar (vgl. evaluation_t)

    return 0;
}
