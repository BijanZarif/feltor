#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <cusp/print.h>
#include <cusp/hyb_matrix.h>

#include "timer.cuh"
#include "xspacelib.cuh"
#include "cg.cuh"

const unsigned n = 4; //global relative error in L2 norm is O(h^P)
//as a rule of thumb with n=4 the true error is err = 1e-3 * eps as long as eps > 1e3*err

const double lx = M_PI;
const double ly = M_PI;
//const double eps = 1e-3; //# of pcg iterations increases very much if 
 // eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus

double initial( double x, double y) {return 0.;}
double pol( double x, double y) {return 1. + sin(x)*sin(y); } //must be strictly positive
//double pol( double x, double y) {return 1.; }

double rhs( double x, double y) { return 2.*sin(x)*sin(y)*(sin(x)*sin(y)+1)-sin(x)*sin(x)*cos(y)*cos(y)-cos(x)*cos(x)*sin(y)*sin(y);}
//double rhs( double x, double y) { return 2.*sin( x)*sin(y);}
double sol(double x, double y)  { return sin( x)*sin(y);}

using namespace std;

//replace DVec with HVec and DMatrix with HMAtrix to compute on host vs device
typedef dg::DVec Vector;
//typedef dg::DMatrix Matrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> Matrix;
int main()
{
    dg::Timer t;
    unsigned Nx, Ny; 
    double eps;
    cout << "Type Nx and Ny and epsilon! \n";
    cin >> Nx; 
    cin >> Ny; //more N means less iterations for same error
    cin >> eps;
    dg::Grid<double, n> grid( 0, lx, 0, ly, Nx, Ny, dg::DIR, dg::DIR);
    dg::V2D<double, n> v2d( grid.hx(), grid.hy());
    dg::W2D<double, n> w2d( grid.hx(), grid.hy());
    //create functions A(chi) x = b
    Vector x =    dg::evaluate( initial, grid);
    Vector b =    dg::evaluate( rhs, grid);
    Vector chi =  dg::evaluate( pol, grid);
    const Vector solution = dg::evaluate( sol, grid);
    Vector error( solution);


    cout << "Create Polarisation object!\n";
    t.tic();
    dg::Polarisation2dX<double, n, dg::HVec> pol( grid);
    t.toc();
    cout << "Creation of polarisation object took: "<<t.diff()<<"s\n";
    cout << "Create Polarisation matrix!\n";
    t.tic();
    cusp::csr_matrix<int, double, cusp::device_memory> B = pol.create(chi);
    Matrix A = B; 
    t.toc();
    cout << "Creation of polarisation matrix took: "<<t.diff()<<"s\n";
    //dg::Matrix Ap= dg::create::laplacian( grid, dg::not_normed); 
    //cout << "Polarisation matrix: "<< endl;
    //cusp::print( A);
    //cout << "Laplacian    matrix: "<< endl;
    //cusp::print( Ap);
    cout << "Create conjugate gradient!\n";
    t.tic();
    dg::CG<Vector > pcg( x, n*n*Nx*Ny);
    t.toc();
    cout << "Creation of conjugate gradient took: "<<t.diff()<<"s\n";

    cout << "# of polynomial coefficients: "<< n <<endl;
    cout << "# of 2d cells                 "<< Nx*Ny <<endl;
    //compute W b
    dg::blas2::symv( w2d, b, b);
    hipDeviceSynchronize();
    t.tic();
    std::cout << "Number of pcg iterations "<< pcg( A, x, b, v2d, eps)<<endl;
    t.toc();
    cout << "For a precision of "<< eps<<endl;
    cout << "Took "<<t.diff()<<"s\n";
    //compute error
    dg::blas1::axpby( 1.,x,-1., error);

    double err = dg::blas2::dot( v2d, error);
    cout << "L2 Norm2 of Error is " << err << endl;
    double norm = dg::blas2::dot( v2d, solution);
    std::cout << "L2 Norm of relative error is "<<sqrt( err/norm)<<std::endl;

    return 0;
}

