#include "hip/hip_runtime.h"
#include <iostream>
#include "projection.cuh"
#include "evaluation.cuh"
#include "blas.h"

double sine( double x){ return sin(x);}
double sine( double x, double y){return sin(x)*sin(y);}

int main()
{
    std::cout << "TEST 1D\n";
    unsigned n_old = 4, n_new = 4, N = 10, Nf = 3;
    dg::Grid1d<double> g  ( 0, 2.*M_PI, n_old, N);
    dg::Grid1d<double> gn ( 0, 2.*M_PI, n_new, N*Nf);
    cusp::coo_matrix<int, double, cusp::host_memory> proj = dg::create::projection1d( g, gn);
    thrust::host_vector<double> v = dg::evaluate( sine, g);
    thrust::host_vector<double> w1d = dg::create::w1d( g);
    thrust::host_vector<double> w1dn = dg::create::w1d( gn);
    thrust::host_vector<double> w( gn.size());
    dg::blas2::gemv( proj, v, w);
    std::cout << "Original vector  "<<dg::blas2::dot( v, w1d, v) << "\n";
    std::cout << "Projected vector "<<dg::blas2::dot( w, w1dn, w) << "\n";
    std::cout << "Difference       "<<dg::blas2::dot( v, w1d, v) - dg::blas2::dot( w, w1dn, w) << "\n"<<std::endl;

    std::cout << "TEST KRONECKER PRODUCT\n";
    dg::create::detail::HelperMatrix<double> m1(2,2), m2(2,2);
    for( unsigned i=0; i<2;i++)
        for( unsigned j=0; j<2;j++)
            m1(i,j) = 2*i+j+1;
    m2(0,0) = 0; m2( 0, 1) = 5; m2( 1,0) = 6, m2( 1,1) = 7;
    std::cout << "M1 \n"<<m1 << "Times\n"<<m2;
    std::cout << "Is \n"<<dg::create::detail::kronecker( m1, m2);
    std::cout << "(Compare Wikipedia for correctness!)\n"<<std::endl;

    std::cout << "TEST 2D\n";
    dg::Grid<double> g2 (0, 2.*M_PI, 0, 2.*M_PI, n_old, N, N);
    dg::Grid<double> g2n (0, 2.*M_PI, 0, 2.*M_PI, n_new, N*Nf, N*Nf);
    cusp::coo_matrix<int, double, cusp::host_memory> proj2d = dg::create::projection2d( g2, g2n);
    thrust::host_vector<double> v2 = dg::evaluate( sine, g2);
    thrust::host_vector<double> w2d = dg::create::w2d( g2);
    thrust::host_vector<double> w2dn = dg::create::w2d( g2n);
    thrust::host_vector<double> w2( g2n.size());
    dg::blas2::gemv( proj2d, v2, w2);
    /*
    std::cout << "Original vector  "<<dg::blas2::dot( v2, w2d, v2) << "\n";
    for( unsigned i=0; i<v2.size(); i++)
        std::cout << v2[i]<<" ";
    std::cout << "\n"<< std::endl;
    for( unsigned i=0; i<w2.size(); i++)
        std::cout << w2[i]<<" ";
    std::cout << std::endl;
    */
    std::cout << "Projected vector "<<dg::blas2::dot( w2, w2dn, w2) << "\n";
    std::cout << "Difference       "<<dg::blas2::dot( v2, w2d, v2) - dg::blas2::dot( w2, w2dn, w2) << "\n";
    return 0;
}
