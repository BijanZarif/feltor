#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/print.h>
#include <cusp/csr_matrix.h>
#include "dg/backend/xspacelib.cuh"
#include "file/read_input.h"
// #include "file/nc_utilities.h"

#include "backend/evaluation.cuh"
#include "backend/timer.cuh"
#include "blas.h"
#include "dz.h"
#include "backend/functions.h"
#include "functors.h"
#include "elliptic.h"
#include "cg.h"
// #include "draw/host_window.h"
#include "../../src/heat/geometry_g.h"
#include "../../src/heat/parameters.h"


int main( )
{

    /////////////////initialize params////////////////////////////////
     std::vector<double> v,v2,v3;

        try{
            v = file::read_input("../../src/heat/input.txt");
            v3 = file::read_input( "../../src/heat/geometry_params_g.txt"); 
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }

    const eule::Parameters p( v);
//     p.display( std::cout);
    const solovev::GeomParameters gp(v3);
//     gp.display( std::cout);

    //////////////////////////////////////////////////////////////////////////
    
    double Rmin=gp.R_0-1.0*gp.a;
    double Zmin=-1.0*gp.a*gp.elongation;
    double Rmax=gp.R_0+1.0*gp.a; 
    double Zmax=1.0*gp.a*gp.elongation;
    /////////////////////////////////////////////initialze fields /////////////////////
    
    solovev::Field field(gp);
    solovev::InvB invb(gp);
    solovev::GradLnB gradlnB(gp);
    solovev::LnB lnB(gp);
    solovev::FieldR bR_(gp);
    solovev::FieldZ bZ_(gp);
    solovev::FieldP bPhi_(gp);
    solovev::FuncNeu funcNEU(gp.R_0,gp.I_0);
    solovev::FuncNeu2 funcNEU2(gp.R_0,gp.I_0);
    solovev::DeriNeu deriNEU(gp.R_0,gp.I_0);
    solovev::DeriNeu2 deriNEU2(gp.R_0,gp.I_0);
    solovev::DeriNeuT2 deriNEUT2(gp.R_0,gp.I_0);
    solovev::DeriNeuT deriNEUT(gp.R_0,gp.I_0);
    solovev::Divb divb(gp.R_0,gp.I_0);
    solovev::B Bfield(gp);
    
    std::cout << "Type n, Nx, Ny, Nz\n";
    //std::cout << "Note, that function is resolved exactly in R,Z for n > 2\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;
    unsigned Nxn = Nx;
    unsigned Nyn = Ny;
    unsigned Nzn = Nz;

    double rk4eps;
    std::cout << "Type RK4 eps (1e-8)\n";
    std::cin >> rk4eps;
    double z0 = 0, z1 = 2.*M_PI;
    for (unsigned i=0;i<6;i++) { 

        Nzn = unsigned(Nz*pow(2,i));
        Nxn = (unsigned)ceil(Nx*pow(2,(double)(i*2./n)));
        Nyn = (unsigned)ceil(Ny*pow(2,(double)(i*2./n)));



        dg::Grid3d<double> g3d( Rmin,Rmax, Zmin,Zmax, z0, z1,  n,Nxn ,Nyn, Nzn,dg::DIR, dg::DIR, dg::PER,dg::cylindrical);
        dg::Grid2d<double> g2d( Rmin,Rmax, Zmin,Zmax,  n, Nxn ,Nyn);

        std::cout << "NR = " << Nxn << std::endl;
        std::cout << "NZ = " << Nyn<< std::endl;
        std::cout << "Nphi = " << Nzn << std::endl;
//            Nxn = (unsigned)ceil(Nxn*pow(2,(double)(2./n)));
//     Nyn = (unsigned)ceil( Nyn*pow(2,(double)(2./n)));

//        dg::Grid3d<double> g3d( Rmin,Rmax, Zmin,Zmax, z0, z1,  n, Nx, Ny, Nz*pow(2,i),dg::DIR, dg::DIR, dg::PER,dg::cylindrical);
//     dg::Grid2d<double> g2d( Rmin,Rmax, Zmin,Zmax,  n, Nx, Ny); 
    const dg::DVec w3d = dg::create::weights( g3d);
    const dg::DVec w2d = dg::create::weights( g2d);
    const dg::DVec v3d = dg::create::inv_weights( g3d);

    std::cout << "computing dzDIR" << std::endl;
    dg::FieldAligned<dg::IDMatrix, dg::DVec>    
        dzFA( field, g3d, rk4eps, dg::DefaultLimiter(), dg::DIR);
    std::cout << "computing dzNEU" << std::endl;
    dg::FieldAligned<dg::IDMatrix, dg::DVec> 
        dzNUFA( field, g3d, rk4eps, dg::DefaultLimiter(), dg::NEU);

    dg::DZ< dg::FieldAligned<dg::IDMatrix, dg::DVec>, dg::DMatrix, dg::DVec> 
        dz ( dzFA, field, g3d, dg::not_normed, dg::centered), 
        dzNU ( dzNUFA, field, g3d, dg::not_normed, dg::centered);

//     dg::DZ<dg::DMatrix, dg::DVec> dzNEU( field, g3d, g3d.hz(), rk4eps, dg::DefaultLimiter(), dg::NEU);
    
//     dg::Grid3d<double> g3dp( Rmin,Rmax, Zmin,Zmax, z0, z1,  n, Nx, Ny, 1);
    
//     dg::DZ<dg::DMatrix, dg::DVec> dz2d( field, g3dp, g3d.hz(), rk4eps, dg::DefaultLimiter(), dg::NEU);
    dg::DVec boundary=dg::evaluate( dg::zero, g3d);
    
    dg::DVec function = dg::evaluate( funcNEU, g3d) ,
                        temp( function),
                        temp2( function),
                        temp3( function),
                        derivative(function),
                        derivativeRZPhi(function),
                        diffRZPhi(function),
                        derivativef(function),
                        derivativeb(function),
                        derivativeones(function),
                        derivative2(function),
                        inverseB( dg::evaluate(invb, g3d)),
                        derivativeT(function),
                        logB( dg::evaluate(lnB, g3d)),
                        derivativeT2(function),
                        derivativeTones(function),
                        derivativeTdz(function),
                        functionTinv(dg::evaluate( dg::zero, g3d)),
                        functionTinv2(dg::evaluate( dg::zero, g3d)),
                        dzTdz(function),
                        dzTdzb(function),
                        dzTdzf(function),
                        dzTdzbd(function),
                        dzTdzfd(function),
                        dzTdzfb(function),
                        dzTdzfbd(function),
                        dzz(function),
                        divbsol(dg::evaluate(divb, g3d)),
                        divbT(function),
                        divBT(function),
                        lambda(function),
                        omega(function),
                        dzTdz2(function);


    dg::DVec ones = dg::evaluate( dg::one, g3d);
    const dg::DVec function2 = dg::evaluate( funcNEU2, g3d);
    const dg::DVec solution = dg::evaluate( deriNEU, g3d);
    const dg::DVec solutionT = dg::evaluate( deriNEUT, g3d);
    const dg::DVec solutiondzz = dg::evaluate( deriNEU2, g3d);
    const dg::DVec solutiondzTdz = dg::evaluate( deriNEUT2, g3d);

    const dg::DVec bhatR = dg::evaluate( bR_, g3d);
    const dg::DVec bhatZ = dg::evaluate( bZ_, g3d);
    const dg::DVec bhatPhi = dg::evaluate(bPhi_, g3d);
//     const dg::DVec Bfield_ = dg::evaluate(Bfield, g3d);
    const dg::DVec gradlnB_ = dg::evaluate(gradlnB, g3d);
//     dg::DMatrix dR(dg::create::dx( g3d, g3d.bcx(),dg::normed,dg::centered));
//     dg::DMatrix dZ(dg::create::dy( g3d, g3d.bcy(),dg::normed,dg::centered));
//     dg::DMatrix dphi(dg::create::dz( g3d, g3d.bcz(), dg::normed,dg::centered));
    
//     dz.set_boundaries( dg::PER, 0, 0);
    //direct gradpar method
//     dg::blas2::gemv( dR, function, temp); //d_R src
//     dg::blas2::gemv( dZ, function, temp2);  //d_Z src
//     dg::blas2::gemv( dphi, function, temp3);  //d_phi src
//     dg::blas1::pointwiseDot( bhatR, temp, temp); // b^R d_R src
//     dg::blas1::pointwiseDot( bhatZ, temp2, temp2); // b^Z d_Z src
//     dg::blas1::pointwiseDot( bhatPhi, temp3, temp3); // b^phi d_phi src
//     dg::blas1::axpby( 1., temp, 1., temp2 ); // b^R d_R src +  b^Z d_Z src
//     dg::blas1::axpby( 1., temp3, 1., temp2,derivativeRZPhi ); // b^R d_R src +  b^Z d_Z src + b^phi d_phi src
// 
//     dg::GeneralEllipticSym<dg::DMatrix, dg::DVec, dg::DVec> ellipticsym( g3d, dg::normed, dg::forward);
//     ellipticsym.set_x(bhatR);
//     ellipticsym.set_y(bhatZ );
//     ellipticsym.set_z(bhatPhi);
//     
//     
  
    dzNU( function, derivative); //dz(f)

//     dzNU.forward( function, derivativef); //dz(f)
//     dzNU.backward( function, derivativeb); //dz(f)

//     dz( ones, derivativeones); //dz(f)
//     dz( function2, derivative2); //dz(f)
//     //compute dzz
//     dz( inverseB, lambda); //gradpar 1/B
//     dg::blas1::pointwiseDivide(lambda,  inverseB, lambda); //-dz lnB
//     dz(function,omega); //dz T
//     dg::blas1::pointwiseDot(omega, lambda, omega);            //- dz lnB dz T
//     dg::blas1::pointwiseDot(omega, gradlnB_, omega);            //- dz lnB dz T
//     dg::blas1::axpby(1.0, omega, 0., dzz,dzz);    
    //     dg::blas1::axpby(-1.0, omega, 0., dzz,dzz);    

    
//     dzNU.forward(derivativeb,temp);
//     dzNU.backward(derivativef,omega);
//     dg::blas1::axpby( -1.0, omega, -0.0, temp,dzz);
//     dz( derivative, dzz); //dz(dz(f))

//     dz.dzz(function,dzz);       
//     dzNU( function, derivative); //dz(f)
//     dg::blas1::pointwiseDot(derivative, gradlnB_, omega);            //- dz lnB dz T
// 
//dz^2 T 
//     dg::blas1::axpby( -1.0, omega, 1.,dzz, dzTdzfbd);

//     
//     
//     dz.centeredT(function, derivativeT); //dz(f)
// 
//     //divB
//     dg::blas1::pointwiseDivide(ones,  inverseB, temp2); //B
//     dz.centeredT(temp2, divBT); // dzT B
// 
//     
//     dz.centeredT( function2, derivativeT2); //dz(f)
//     dz.centeredT( ones, derivativeTones); //dz(f)
    //B dz f/B
//     dg::blas1::pointwiseDot( inverseB, function, temp);
//     dz( temp, derivativeTdz);
//     dg::blas1::pointwiseDot( derivativeTdz, Bfield_, derivativeTdz);
    //oder dz f - f dzlnB
//     dz( function, derivativeTdz);
//     dg::blas1::pointwiseDot(function,gradlnB_,temp);
//     dg::blas1::axpby(- 1.0, temp, 1., derivativeTdz,derivativeTdz);


    
    //     dg::blas1::pointwiseDivide( derivativeTdz, inverseB, derivativeTdz);
//     
//     dz.centeredT( derivative, dzTdz); //dzT(dz(f))
//     
//     //overwrite with sym from adjoint dg
//     ellipticsym.symv(function,dzTdz);
//     dg::blas1::scal(dzTdz,-1.0);
// //     dz.centeredT(ones,divbT);
    dz.forwardT( derivativef, dzTdzf);  //dzT(dz(f))
//     dz.backwardT( derivativeb, dzTdzb); //dzT(dz(f))

//     //centered
//     dz.centeredTD(derivative,dzTdzfbd);
//     dz.forwardTD( derivativef, dzTdzfd); //dzT(dz(f))
//     dz.backwardTD( derivativeb, dzTdzbd); //dzT(dz(f))

//     //arithmetic average
//     dg::blas1::axpby(0.5,dzTdzb,0.5,dzTdzf,dzTdzfb);
//     dg::blas1::axpby(0.5,dzTdzbd,0.5,dzTdzfd,dzTdzfbd); 
    dz.symv(function,dzTdzfb);
    dg::blas1::pointwiseDot(v3d,dzTdzfb,dzTdzfb);
//     dz.centeredT( derivative2, dzTdz2); //dzT(dz(f))
//     dg::blas1::pointwiseDivide(ones,  inverseB, temp2); //B
//     dz.centeredT( ones, divbT);
//     
//     double normdzdz =dg::blas2::dot(derivative2, w3d,derivative2);
//     double normdz1dz =dg::blas2::dot(derivativeones, w3d,derivative2);
//     double normdivBT =dg::blas2::dot(divBT, w3d,divBT);
//     double normdivbT =dg::blas2::dot(divbT, w3d,divbT);
//     double normdivb =dg::blas2::dot(divbsol, w3d,divbsol); 
//     double normdzTf = dg::blas2::dot(derivativeT2, w3d, function2);
//     double normdzT_1 = dg::blas2::dot(derivativeT2, w3d, ones);
//     double normdzT1 = dg::blas2::dot(derivativeTones, w3d, function2);
//     double normfdz = dg::blas2::dot(function2, w3d, derivative2);
//     double norm1dz = dg::blas2::dot(ones, w3d, derivative2);
//     double normfdzTdz = dg::blas2::dot(function2, w3d, dzTdz2);
//     double norm1dzTdz = dg::blas2::dot(ones, w3d, dzTdz2);
//     
//     double norm1dzTB = dg::blas2::dot(ones, w3d, divBT);
//     double normBdz1 = dg::blas2::dot(temp2, w3d, derivativeones);
//     double normfdz1 = dg::blas2::dot(function2, w3d, derivativeones);
// 
    std::cout << "--------------------testing dz" << std::endl;
    double norm = dg::blas2::dot( w3d, solution);
    std::cout << "|| Solution ||   "<<sqrt( norm)<<"\n";
    double err =dg::blas2::dot( w3d, derivative);
    std::cout << "|| Derivative || "<<sqrt( err)<<"\n";
    dg::blas1::axpby( 1., solution, -1., derivative);
    err =dg::blas2::dot( w3d, derivative);
    std::cout << "Relative Difference in DZ is "<< sqrt( err/norm )<<"\n"; 
   
//     std::cout << "--------------------testing dz with RZPhi method" << std::endl;
//     std::cout << "|| Solution ||   "<<sqrt( norm)<<"\n";
//     double errRZPhi =dg::blas2::dot( w3d, derivativeRZPhi);
//     std::cout << "|| Derivative || "<<sqrt( errRZPhi)<<"\n";
//     dg::blas1::axpby( 1., solution, -1., derivativeRZPhi);
//     errRZPhi =dg::blas2::dot( w3d, derivativeRZPhi);    
//     std::cout << "Relative Difference in DZ is "<< sqrt( errRZPhi/norm )<<"\n"; 
//     
//     std::cout << "--------------------testing dzT" << std::endl;
//     std::cout << "|| divbsol ||  "<<sqrt( normdivb)<<"\n";
//     std::cout << "|| divbT  ||   "<<sqrt( normdivbT)<<"\n";
//     dg::blas1::axpby( 1., divbsol, -1., divbT);
//     normdivbT =dg::blas2::dot(divbT, w3d,divbT);
//     std::cout << "Relative Difference in DZT is   "<<sqrt( normdivbT)<<"\n";
//     std::cout << "-------------------- " << std::endl;
//     std::cout << "|| divB || "<<sqrt( normdivBT)<<"\n";
// 
//     
//     std::cout << "-------------------- " << std::endl;
//     double normT = dg::blas2::dot( w3d, solutionT);
//     std::cout << "|| SolutionT  ||  "<<sqrt( normT)<<"\n";
//     double errT =dg::blas2::dot( w3d, derivativeT);
//     std::cout << "|| DerivativeT || "<<sqrt( errT)<<"\n";
//     dg::blas1::axpby( 1., solutionT, -1., derivativeT);
//     errT =dg::blas2::dot( w3d, derivativeT);
//     std::cout << "Relative Difference in DZT is "<< sqrt( errT/normT )<<"\n"; 
//     dg::blas1::axpby( 1., derivative, -1., derivativeT,omega);
//     double errTdiffdzdzT =dg::blas2::dot( w3d, omega);
//     std::cout << "Relative Difference in DZT to DZ is "<< sqrt( errTdiffdzdzT/norm )<<"\n";   
//     std::cout << "--------------------testing dzT with dz" << std::endl;
//     std::cout << "|| SolutionT ||     "<<sqrt( normT)<<"\n";
//     double errTdz =dg::blas2::dot( w3d, derivativeTdz);
//     std::cout << "|| DerivativeTdz || "<<sqrt( errTdz)<<"\n";
//     dg::blas1::axpby( 1., solutionT, -1., derivativeTdz);
//     errTdz =dg::blas2::dot( w3d, derivativeTdz);
//     std::cout << "Relative Difference in DZT is "<< sqrt( errTdz/normT )<<"\n"; 
//     std::cout << "--------------------testing dzTdz " << std::endl;
  
    double normdzTdz = dg::blas2::dot( w3d, solutiondzTdz);
//     std::cout << std::setprecision(16);
//     std::cout << "is the norm of the testfunction zero ? =       "<<sqrt( normdzTdz)<<"\n";
//     for (unsigned i=0; i<g3d.size();i++){
//         std::cout << "solvalue " << solutiondzTdz[i]<< std::endl;
//     }
//     double normnormdzTdz = dg::blas2::dot(ones, w3d, solutiondzTdz);
//     std::cout << "is the norm of the testfunction zero ? =       "<< normnormdzTdz<<"\n";


//     std::cout << "|| SolutionT ||      "<<sqrt( normdzTdz)<<"\n";
//     double errdzTdz =dg::blas2::dot( w3d,dzTdz);
//     std::cout << "|| DerivativeTdz ||  "<<sqrt( errdzTdz)<<"\n";
//     dg::blas1::axpby( 1., solutiondzTdz, -1., dzTdz);
//     errdzTdz =dg::blas2::dot( w3d, dzTdz);
//     std::cout << "Relative Difference in DZT is "<< sqrt( errdzTdz/normdzTdz )<<"\n";   
    
    std::cout << "--------------------testing dzTdzfb " << std::endl;
    std::cout << "|| SolutionT ||      "<<sqrt( normdzTdz)<<"\n";
    double errdzTdzfb =dg::blas2::dot( w3d,dzTdzfb);
    std::cout << "|| DerivativeTdz ||  "<<sqrt( errdzTdzfb)<<"\n";
    dg::blas1::axpby( 1., solutiondzTdz, -1., dzTdzfb);
    errdzTdzfb =dg::blas2::dot( w3d, dzTdzfb);
    std::cout << "Relative Difference in DZT is "<< sqrt( errdzTdzfb/normdzTdz )<<"\n";
//   
//     std::cout << "--------------------testing dzTdzfb with direct method" << std::endl;
//     std::cout << "|| SolutionT ||      "<<sqrt( normdzTdz)<<"\n";
//     double errdzTdzfbd =dg::blas2::dot( w3d,dzTdzfbd);
//     std::cout << "|| DerivativeTdz ||  "<<sqrt( errdzTdzfbd)<<"\n";
//     dg::blas1::axpby( 1., solutiondzTdz, -1., dzTdzfbd);
//     errdzTdzfbd =dg::blas2::dot( w3d, dzTdzfbd);
//     std::cout << "Relative Difference in DZT is "<< sqrt( errdzTdzfbd/normdzTdz )<<"\n";
    

//     std::cout << "--------------------testing dzTdz with dzz" << std::endl;
//     double normdzz = dg::blas2::dot( w3d, solutiondzz);
// 
//     std::cout << "|| Solution ||      "<<sqrt( normdzz)<<"\n";
//     double errdzz =dg::blas2::dot( w3d,dzz);
//     std::cout << "|| dzz ||  "<<sqrt( errdzz)<<"\n";
//     dg::blas1::axpby( 1., solutiondzz, -1., dzz);
//     errdzz =dg::blas2::dot( w3d, dzz);
//     std::cout << "Relative Difference in DZT is "<< sqrt( errdzz/normdzz )<<"\n";   
//     
//     std::cout << "--------------------testing adjointness " << std::endl;
//     std::cout << "<f,dz(f)>   = "<< normfdz<<"\n";
//     std::cout << "-<dzT(f),f> = "<< -normdzTf<<"\n";
//     std::cout << "Diff        = "<< normfdz+normdzTf<<"\n";     
//     std::cout << "-------------------- " << std::endl;
// 
//     std::cout << "<B,dz(1)>   = "<< normBdz1<<"\n";
//     std::cout << "-<dzT(B),1> = "<< -norm1dzTB<<"\n";
//     std::cout << "Diff        = "<< normBdz1+norm1dzTB<<"\n";     
//     std::cout << "-------------------- " << std::endl;
//     
//     std::cout << "<f,dz(1)>   = "<< normfdz1<<"\n";
//     std::cout << "-<dzT(f),1> = "<< -normdzT_1<<"\n";
//     std::cout << "Diff        = "<< normfdz1+normdzT_1<<"\n";   
//     std::cout << "-------------------- " << std::endl;
//     
//     std::cout << "<1,dz(f)>   = "<< norm1dz<<"\n";
//     std::cout << "-<dzT(1),f> = "<< -normdzT1<<"\n";
//     std::cout << "Diff        = "<< norm1dz+normdzT1<<"\n";   
//     std::cout << "-------------------- " << std::endl;
//   
//     std::cout << "<f,dzT(dz(f))> = "<< normfdzTdz<<"\n";
//     std::cout << "-<dz(f),dz(f)> = "<< -normdzdz<<"\n";
//     std::cout << "Diff           = "<< normfdzTdz+normdzdz<<"\n";     
//     std::cout << "-------------------- " << std::endl;
//    
//     std::cout << "<1,dzT(dz(f))> = "<< norm1dzTdz<<"\n";
//     std::cout << "-<dz(1),dz(f)> = "<< -normdz1dz<<"\n";
//     std::cout << "Diff           = "<< norm1dzTdz+normdz1dz<<"\n";    
//     
// 
//     std::cout << "--------------------testing GeneralElliptic with inversion " << std::endl; 
//    //set up the parallel diffusion
//     dg::GeneralEllipticSym<dg::DMatrix, dg::DVec, dg::DVec> elliptic( g3d, dg::not_normed, dg::forward);
//     elliptic.set_x(bhatR);
//     elliptic.set_y(bhatZ );
//     elliptic.set_z(bhatPhi);
    
    
    double eps =1e-8;   
    dg::Invert< dg::DVec> invert( dg::evaluate(dg::zero,g3d), w3d.size(), eps );  
    std::cout << "MAX # iterations = " << w3d.size() << std::endl;
// 
//    const dg::DVec rhs = dg::evaluate( solovev::DeriNeuT2( gp.R_0, gp.I_0), g3d);
// // 
//     std::cout << " # of iterations "<< invert( elliptic, functionTinv, rhs ) << std::endl; //is dzTdz 
//   
    double normf = dg::blas2::dot( w3d, function);
// 
//     std::cout << "Norm analytic Solution  "<<sqrt( normf)<<"\n";
    double errinvT =dg::blas2::dot( w3d, functionTinv);
//     std::cout << "Norm numerical Solution "<<sqrt( errinvT)<<"\n";
// 
//     dg::blas1::axpby( 1., function, +1.,functionTinv);
//     errinvT =dg::blas2::dot( w3d, functionTinv);
//     std::cout << "Relative Difference is  "<< sqrt( errinvT/normf )<<"\n";
//     
    std::cout << "--------------------testing dzT" << std::endl; 
    std::cout << " # of iterations "<< invert( dzNU, functionTinv2,solutiondzTdz ) << std::endl; //is dzTdz
    std::cout << "Norm analytic Solution  "<<sqrt( normf)<<"\n";
    double errinvT2 =dg::blas2::dot( w3d, functionTinv2);
    std::cout << "Norm numerical Solution "<<sqrt( errinvT2)<<"\n";
    dg::blas1::axpby( 1., function, -1.,functionTinv2);
    errinvT2 =dg::blas2::dot( w3d, functionTinv2);
    std::cout << "Relative Difference is  "<< sqrt( errinvT2/normf )<<"\n";

//write netcdf
//     file::NC_Error_Handle err;
//     int ncid,tvarID;
//     err = nc_create( "out3.nc",NC_NETCDF4|NC_CLOBBER, &ncid);
//     dg::DVec transferD( dg::evaluate(dg::zero, g3d));
//     dg::HVec transferH( dg::evaluate(dg::zero, g3d));
// 
//     int dim_ids[4];
//     err = file::define_dimensions( ncid, dim_ids, &tvarID, g3d);
//     std::string names[3] = {"TG","TD","TA"}; 
//     int dataIDs[3]; 
//     size_t start[4] = {0, 0, 0, 0};
//     size_t count[4] = {1, g3d.Nz(), g3d.n()*g3d.Ny(), g3d.n()*g3d.Nx()};
//     err = nc_def_var( ncid, names[0].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[0]);  
//     err = nc_def_var( ncid, names[1].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[1]);  
//     err = nc_def_var( ncid, names[2].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[2]);
//     err = nc_enddef( ncid);
//     err = nc_open("out3.nc", NC_WRITE, &ncid);
//     transferD=dzTdz;
//     transferH =transferD;
//     err = nc_put_vara_double( ncid, dataIDs[0], start, count, transferH.data());
//     transferD=dzTdzfbd;
//     transferH =transferD;
//     err = nc_put_vara_double( ncid, dataIDs[1], start, count, transferH.data());    
//     transferD=dzTdzfb;
//     transferH =transferD;
//     err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data());    
//      err = nc_close(ncid);
    }
    
//     std::cout << "make Plot" << std::endl;
//     //make equidistant grid from dggrid
//     dg::HVec hvisual;
//     //allocate mem for visual
//     dg::HVec visual;
//     dg::HMatrix equigrid = dg::create::backscatter(g3d);               
// 
//     //evaluate on valzues from devicevector on equidistant visual hvisual vector
//     visual = dg::evaluate( dg::one, g3d);
//     //Create Window and set window title
//     GLFWwindow* w = draw::glfwInitAndCreateWindow( 100*Nz, 700, "");
//     draw::RenderHostData render(7 , 1*Nz);  
//     //create a colormap
//     draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
//     dg::DMatrix jump( dg::create::jump2d( g3d, g3d.bcx(), g3d.bcy(), dg::not_normed));
//     dg::blas2::symv( jump, ones, lambda);

//     std::stringstream title;
//     title << std::setprecision(10) << std::scientific;

//     while (!glfwWindowShouldClose( w ))
//     {
//         hvisual = divBT;
//         dg::blas2::gemv( equigrid, hvisual, visual);        
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"divB"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
// 
//         }
//         hvisual = derivativeT;         
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"dzT(f)"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         hvisual = derivative;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"dz(f)"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         dg::blas1::axpby(1.0,derivative,-1.0,derivativeT,omega);
//         hvisual = omega;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"diff"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         hvisual = dzTdz;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"dzTdzfb"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         hvisual = dzTdz;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"dzTdz"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         hvisual = functionTinv2;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"dzz"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         title << std::fixed; 
//         glfwSetWindowTitle(w,title.str().c_str());
//         title.str("");
//         glfwSwapBuffers(w);
//         glfwWaitEvents();
//     }
// 
//     glfwTerminate();
    return 0;
}
