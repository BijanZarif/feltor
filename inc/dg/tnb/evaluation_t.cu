#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "evaluation.cuh"
#include "weights.cuh"

#include "blas.h"

double function( double x)
{
    return exp(x);
}

double function( double x, double y)
{
        return exp(x)*exp(y);
}
double function( double x, double y, double z)
{
        return exp(x)*exp(y)*exp(z);
}

const double lx = 2;
const double ly = 2;
const double lz = 2;

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;

using namespace std;
int main()
{
    unsigned n;
    cout << "Type # of polynomial coefficients ( 1, 2,...,20)!\n";
    cin >> n;
    unsigned N, Nx, Ny, Nz;
    cout << "# of polynomial coefficients is: "<< n<<endl;
    cout << "Type # of grid cells (e.g. 10, 100)! ( Nx = N, Ny = 2N, Nz = 10*N)\n";
    cin >> N;
    cout << "# of grid cells is: "<< N<<endl;
    Nx = N; Ny = 2*N; Nz = 10*N;

    dg::Grid1d<double> g1d( 0, lx, n, N);
    dg::Grid2d<double> g2d( 0, lx,0, ly,n, Nx, Ny);
    dg::Grid3d<double> g3d( 0, lx,0, ly,0, lz, n, Nx, Ny, Nz);

    //test evaluation and expand functions
    HVec h_v = dg::expand( function, g1d);
    HVec h_x = dg::evaluate( function, g1d);
    HVec h_m = dg::expand( function, g2d);
    HVec h_n = dg::evaluate( function, g2d);
    HVec h_z = dg::evaluate( function, g3d);
    HVec w3d = dg::create::w3d( g3d);

    //test preconditioners
    dg::blas2::symv( 1., dg::create::s1d(g1d), h_v, 0., h_v);

    double norm = dg::blas2::dot( h_v, dg::create::t1d(g1d), h_v);
    double normX = dg::blas2::dot( h_x, dg::create::w1d(g1d), h_x);
    //double norm2 = dg::blas2::dot( dg::S2D<double>(g2d), h_m);
    double norm2 = dg::blas2::dot( dg::create::w2d(g2d), h_m);
    double norm2X = dg::blas2::dot( dg::create::w2d(g2d), h_n);
    double norm3X = dg::blas2::dot( h_z, w3d, h_z);

    cout << "Square normalized 1D norm "<< norm <<"\n";
    cout << "Square normalized 1DXnorm "<< normX <<"\n";
    double solution = (exp(4.) -exp(0))/2.;
    cout << "Correct square norm is    "<<solution<<endl;
    cout << "Square normalized 2D norm "<< norm2 <<"\n";
    cout << "Square normalized 2DXnorm "<< norm2X<<"\n";
    double solution2 = (exp(4.)-exp(0))/2.*(exp(4.) -exp(0))/2.;
    cout << "Correct square norm is    "<<solution2<<endl;

    cout << "Square normalized 3DXnorm   "<< norm3X<<"\n";
    double solution3 = solution2*solution;
    cout << "Correct square norm is      "<<solution3<<endl;
    return 0;
} 
