#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>

#include "dg/backend/xspacelib.cuh"
#include "dg/functors.h"
#include "file/read_input.h"

#include "dg/backend/timer.cuh"
//#include "guenther.h"
#include "solovev.h"
#include "conformal.h"
#include "orthogonal.h"
#include "dg/ds.h"
#include "init.h"

#include "file/nc_utilities.h"

thrust::host_vector<double> periodify( const thrust::host_vector<double>& in, const dg::Grid2d<double>& g)
{
    thrust::host_vector<double> out(g.size());
    for( unsigned i=0; i<g.Ny()-1; i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((i*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.Ny()-1; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((0*g.n() + k)*g.Nx() + j)*g.n()+l];
    return out;
}

double sineX( double x, double y) {return sin(x)*sin(y);}
double cosineX( double x, double y) {return cos(x)*sin(y);}
double sineY( double x, double y) {return sin(x)*sin(y);}
double cosineY( double x, double y) {return sin(x)*cos(y);}
typedef dg::FieldAligned< conformal::RingGrid3d<dg::HVec> , dg::IHMatrix, dg::HVec> DFA;
//typedef dg::FieldAligned< orthogonal::RingGrid3d<dg::HVec> , dg::IHMatrix, dg::HVec> DFA;

int main( int argc, char* argv[])
{
    std::cout << "Type n, Nx, Ny, Nz\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;   
    std::vector<double> v, v2;
    try{ 
        if( argc==1)
        {
            v = file::read_input( "geometry_params_Xpoint.txt"); 
        }
        else
        {
            v = file::read_input( argv[1]); 
        }
    }
    catch (toefl::Message& m) {  
        m.display(); 
        for( unsigned i = 0; i<v.size(); i++)
            std::cout << v[i] << " ";
            std::cout << std::endl;
        return -1;}
    //write parameters from file into variables
    solovev::GeomParameters gp(v);
    solovev::Psip psip( gp); 
    std::cout << "Psi min "<<psip(gp.R_0, 0)<<"\n";
    std::cout << "Type psi_0 and psi_1\n";
    double psi_0, psi_1;
    std::cin >> psi_0>> psi_1;
    gp.display( std::cout);
    dg::Timer t;
    //solovev::detail::Fpsi fpsi( gp, -10);
    std::cout << "Constructing conformal grid ... \n";
    t.tic();
    conformal::RingGrid3d<dg::HVec> g3d(gp, psi_0, psi_1, n, Nx, Ny,Nz, dg::DIR);
    conformal::RingGrid2d<dg::HVec> g2d = g3d.perp_grid();
    //orthogonal::RingGrid3d<dg::HVec> g3d(gp, psi_0, psi_1, n, Nx, Ny,Nz, dg::DIR);
    //orthogonal::RingGrid2d<dg::HVec> g2d = g3d.perp_grid();
    dg::Grid2d<double> g2d_periodic(g2d.x0(), g2d.x1(), g2d.y0(), g2d.y1(), g2d.n(), g2d.Nx(), g2d.Ny()+1); 
    t.toc();
    std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    int ncid;
    file::NC_Error_Handle err;
    err = nc_create( "test.nc", NC_NETCDF4|NC_CLOBBER, &ncid);
    int dim3d[2];
    err = file::define_dimensions(  ncid, dim3d, g2d_periodic);
    int coordsID[2], onesID, defID, divBID;
    err = nc_def_var( ncid, "x_XYP", NC_DOUBLE, 2, dim3d, &coordsID[0]);
    err = nc_def_var( ncid, "y_XYP", NC_DOUBLE, 2, dim3d, &coordsID[1]);
    //err = nc_def_var( ncid, "z_XYP", NC_DOUBLE, 3, dim3d, &coordsID[2]);
    err = nc_def_var( ncid, "psi", NC_DOUBLE, 2, dim3d, &onesID);
    err = nc_def_var( ncid, "deformation", NC_DOUBLE, 2, dim3d, &defID);
    err = nc_def_var( ncid, "divB", NC_DOUBLE, 2, dim3d, &divBID);

    thrust::host_vector<double> psi_p = dg::pullback( psip, g2d);
    //g.display();
    err = nc_put_var_double( ncid, onesID, periodify(psi_p, g2d_periodic).data());
    dg::HVec X( g2d.size()), Y(X); //P = dg::pullback( dg::coo3, g);
    for( unsigned i=0; i<g2d.size(); i++)
    {
        X[i] = g2d.r()[i];
        Y[i] = g2d.z()[i];
    }

    dg::HVec temp0( g2d.size()), temp1(temp0);
    dg::HVec w3d = dg::create::weights( g2d);

    err = nc_put_var_double( ncid, coordsID[0], periodify(X, g2d_periodic).data());
    err = nc_put_var_double( ncid, coordsID[1], periodify(Y, g2d_periodic).data());
    //err = nc_put_var_double( ncid, coordsID[2], g.z().data());

    //dg::blas1::pointwiseDivide( g2d.g_xy(), g2d.g_xx(), temp0);
    dg::blas1::pointwiseDivide( g2d.g_yy(), g2d.g_xx(), temp0);
    const dg::HVec ones = dg::evaluate( dg::one, g2d);
    dg::blas1::axpby( 1., ones, -1., temp0, temp0);
    X=temp0;
    err = nc_put_var_double( ncid, defID, periodify(X, g2d_periodic).data());

    std::cout << "Construction successful!\n";

    //compute error in volume element
    //const dg::HVec f_ = g2d.f1();
    const dg::HVec f_ = g2d.f();
    dg::blas1::pointwiseDot( g2d.g_xx(), g2d.g_yy(), temp0);
    dg::blas1::pointwiseDot( g2d.g_xy(), g2d.g_xy(), temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    //dg::blas1::transform( temp0, temp0, dg::SQRT<double>());
    //dg::blas1::pointwiseDot( f_, f_, temp1);
    temp1 = ones;
    dg::blas1::axpby( 0.0, temp1, 1.0, g2d.g_xx(),  temp1);
    dg::blas1::pointwiseDot( temp1, temp1, temp1);
    dg::blas1::axpby( 1., temp1, -1., temp0, temp0);
    double error = sqrt( dg::blas2::dot( temp0, w3d, temp0)/dg::blas2::dot( temp1, w3d, temp1));
    std::cout<< "Rel Error in Determinant is "<<error<<"\n";

    dg::blas1::pointwiseDot( g2d.g_xx(), g2d.g_yy(), temp0);
    dg::blas1::pointwiseDot( g2d.g_xy(), g2d.g_xy(), temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    //dg::blas1::pointwiseDot( temp0, g.g_pp(), temp0);
    dg::blas1::transform( temp0, temp0, dg::SQRT<double>());
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::axpby( 1., temp0, -1., g2d.vol(), temp0);
    error = sqrt(dg::blas2::dot( temp0, w3d, temp0)/dg::blas2::dot( g2d.vol(), w3d, g2d.vol()));
    std::cout << "Rel Consistency  of volume is "<<error<<"\n";

    //temp0=g.r();
    //dg::blas1::pointwiseDivide( temp0, g.g_xx(), temp0);
    dg::blas1::pointwiseDot( f_, f_, temp0);
    dg::blas1::axpby( 0.0,temp0 , 1.0, g2d.g_xx(), temp0);
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    //dg::blas1::axpby( 1., temp0, -1., g2d.vol(), temp0);
    dg::blas1::axpby( 1., ones, -1., g2d.vol(), temp0);
    error=sqrt(dg::blas2::dot( temp0, w3d, temp0))/sqrt( dg::blas2::dot(g2d.vol(), w3d, g2d.vol()));
    std::cout << "Rel Error of volume form is "<<error<<"\n";

    solovev::conformal::FieldY fieldY(gp);
    //solovev::ConformalField fieldY(gp);
    dg::HVec fby = dg::pullback( fieldY, g2d);
    dg::blas1::pointwiseDot( fby, f_, fby);
    dg::blas1::pointwiseDot( fby, f_, fby);
    //for( unsigned k=0; k<Nz; k++)
        //for( unsigned i=0; i<n*Ny; i++)
        //    for( unsigned j=0; j<n*Nx; j++)
        //        //by[k*n*n*Nx*Ny + i*n*Nx + j] *= g.f_x()[j]*g.f_x()[j];
        //        fby[i*n*Nx + j] *= g.f_x()[j]*g.f_x()[j];
    //dg::HVec fby_device = fby;
    dg::blas1::scal( fby, 1./gp.R_0);
    temp0=g2d.r();
    dg::blas1::pointwiseDot( temp0, fby, fby);
    dg::blas1::pointwiseDivide( ones, g2d.vol(), temp0);
    dg::blas1::axpby( 1., temp0, -1., fby, temp1);
    error= dg::blas2::dot( temp1, w3d, temp1)/dg::blas2::dot(fby,w3d,fby);
    std::cout << "Rel Error of g.g_xx() is "<<sqrt(error)<<"\n";
    const dg::HVec vol = dg::create::volume( g3d);
    dg::HVec ones3d = dg::evaluate( dg::one, g3d);
    double volume = dg::blas1::dot( vol, ones3d);

    std::cout << "TEST VOLUME IS:\n";
    if( psi_0 < psi_1) gp.psipmax = psi_1, gp.psipmin = psi_0;
    else               gp.psipmax = psi_0, gp.psipmin = psi_1;
    solovev::Iris iris( gp);
    //dg::CylindricalGrid<dg::HVec> g3d( gp.R_0 -2.*gp.a, gp.R_0 + 2*gp.a, -2*gp.a, 2*gp.a, 0, 2*M_PI, 3, 2200, 2200, 1, dg::PER, dg::PER, dg::PER);
    dg::CartesianGrid2d g2dC( gp.R_0 -1.2*gp.a, gp.R_0 + 1.2*gp.a, -1.2*gp.a, 1.2*gp.a, 1, 1e3, 1e3, dg::PER, dg::PER);
    dg::HVec vec  = dg::evaluate( iris, g2dC);
    dg::HVec R  = dg::evaluate( dg::coo1, g2dC);
    dg::HVec g2d_weights = dg::create::volume( g2dC);
    double volumeRZP = 2.*M_PI*dg::blas2::dot( vec, g2d_weights, R);
    std::cout << "volumeXYP is "<< volume<<std::endl;
    std::cout << "volumeRZP is "<< volumeRZP<<std::endl;
    std::cout << "relative difference in volume is "<<fabs(volumeRZP - volume)/volume<<std::endl;
    std::cout << "Note that the error might also come from the volume in RZP!\n"; //since integration of jacobian is fairly good probably

    /////////////////////////TEST 3d grid//////////////////////////////////////
    std::cout << "Start DS test!"<<std::endl;
    const dg::HVec vol3d = dg::create::volume( g3d);
    t.tic();
    DFA fieldaligned( conformal::Field( gp, g3d.x(), g3d.f_x()), g3d, gp.rk4eps, dg::NoLimiter()); 
    //DFA fieldaligned( orthogonal::Field( gp, g2d, g2d.f2_xy()), g3d, gp.rk4eps, dg::NoLimiter()); 

    dg::DS<DFA, dg::DMatrix, dg::HVec> ds( fieldaligned, conformal::Field(gp, g3d.x(), g3d.f_x()), dg::normed, dg::centered);
    //dg::DS<DFA, dg::DMatrix, dg::HVec> ds( fieldaligned, orthogonal::Field(gp, g2d, g2d.f2_xy()), dg::normed, dg::centered);
    t.toc();
    std::cout << "Construction took "<<t.diff()<<"s\n";
    dg::HVec B = dg::pullback( solovev::InvB(gp), g3d), divB(B);
    dg::HVec lnB = dg::pullback( solovev::LnB(gp), g3d), gradB(B);
    dg::HVec gradLnB = dg::pullback( solovev::GradLnB(gp), g3d);
    dg::blas1::pointwiseDivide( ones3d, B, B);
    dg::HVec function = dg::pullback( solovev::FuncNeu(gp), g3d), derivative(function);
    ds( function, derivative);

    ds.centeredT( B, divB);
    double norm =  sqrt( dg::blas2::dot(divB, vol3d, divB));
    std::cout << "Divergence of B is "<<norm<<"\n";

    ds.centered( lnB, gradB);
    std::cout << "num. norm of gradLnB is "<<sqrt( dg::blas2::dot( gradB,vol3d, gradB))<<"\n";
    norm = sqrt( dg::blas2::dot( gradLnB, vol3d, gradLnB) );
    std::cout << "ana. norm of gradLnB is "<<norm<<"\n";
    dg::blas1::axpby( 1., gradB, -1., gradLnB, gradLnB);
    X = divB;
    err = nc_put_var_double( ncid, divBID, periodify(X, g2d_periodic).data());
    double norm2 = sqrt(dg::blas2::dot(gradLnB, vol3d,gradLnB));
    std::cout << "rel. error of lnB is    "<<norm2/norm<<"\n";
    err = nc_close( ncid);



    return 0;
}
