#include "hip/hip_runtime.h"
#include <iostream>

#include "file/read_input.h"
#include "file/nc_utilities.h"

#include "dg/backend/timer.cuh"
#include "dg/backend/grid.h"
#include "dg/elliptic.h"
#include "dg/cg.h"

#include "solovev.h"
#include "flux.h"

int main(int argc, char**argv)
{
    std::cout << "Type n, Nx, Ny, Nz\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;   
    std::cout << "Type psi_0 and psi_1\n";
    double psi_0, psi_1;
    std::cin >> psi_0>> psi_1;
    std::vector<double> v, v2;
    try{ 
        if( argc==1)
        {
            v = file::read_input( "geometry_params_Xpoint.txt"); 
        }
        else
        {
            v = file::read_input( argv[1]); 
        }
    }
    catch (toefl::Message& m) {  
        m.display(); 
        for( unsigned i = 0; i<v.size(); i++)
            std::cout << v[i] << " ";
            std::cout << std::endl;
        return -1;}
    //write parameters from file into variables
    solovev::GeomParameters gp(v);
    gp.display( std::cout);
    dg::Timer t;
    solovev::Psip psip( gp); 
    std::cout << "Psi min "<<psip(gp.R_0, 0)<<"\n";
    std::cout << "Constructing grid ... \n";
    t.tic();
    flux::RingGrid3d<dg::DVec> g3d(gp, psi_0, psi_1, n, Nx, Ny,Nz, dg::DIR);
    flux::RingGrid2d<dg::DVec> g2d = g3d.perp_grid();
    dg::Elliptic<flux::RingGrid3d<dg::DVec>, dg::DMatrix, dg::DVec> pol( g3d, dg::not_normed, dg::centered);

    t.toc();
    std::cout << "Construction took "<<t.diff()<<"s\n";
    ///////////////////////////////////////////////////////////////////////////
    int ncid;
    file::NC_Error_Handle ncerr;
    ncerr = nc_create( "testE.nc", NC_NETCDF4|NC_CLOBBER, &ncid);
    int dim2d[2];
    ncerr = file::define_dimensions(  ncid, dim2d, g2d);
    int coordsID[2], psiID, functionID, function2ID;
    ncerr = nc_def_var( ncid, "x_XYP", NC_DOUBLE, 2, dim2d, &coordsID[0]);
    ncerr = nc_def_var( ncid, "y_XYP", NC_DOUBLE, 2, dim2d, &coordsID[1]);
    ncerr = nc_def_var( ncid, "psi", NC_DOUBLE, 2, dim2d, &psiID);
    ncerr = nc_def_var( ncid, "deformation", NC_DOUBLE, 2, dim2d, &functionID);
    ncerr = nc_def_var( ncid, "divB", NC_DOUBLE, 2, dim2d, &function2ID);

    dg::HVec X( g2d.size()), Y(X); //P = dg::pullback( dg::coo3, g);
    for( unsigned i=0; i<g2d.size(); i++)
    {
        X[i] = g2d.r()[i];
        Y[i] = g2d.z()[i];
    }
    ncerr = nc_put_var_double( ncid, coordsID[0], X.data());
    ncerr = nc_put_var_double( ncid, coordsID[1], Y.data());
    ///////////////////////////////////////////////////////////////////////////
    dg::DVec x =    dg::pullback( dg::zero, g3d);
    const dg::DVec b =    dg::pullback( solovev::EllipticDirPerM(gp, psi_0, psi_1), g3d);
    const dg::DVec chi =  dg::pullback( solovev::Bmodule(gp), g3d);
    const dg::DVec solution = dg::pullback( solovev::FuncDirPer(gp, psi_0, psi_1 ), g3d);
    const dg::DVec vol3d = dg::create::volume( g3d);
    pol.set_chi( chi);
    //compute error
    dg::DVec error( solution);
    const double eps = 1e-10;
    dg::Invert<dg::DVec > invert( x, n*n*Nx*Ny*Nz, eps);
    std::cout << "eps \t # iterations \t error \t hx_max\t hy_max \t time/iteration \n";
    std::cout << eps<<"\t";
    t.tic();
    unsigned number = invert(pol, x,b);// vol3d, v3d );
    std::cout <<number<<"\t";
    t.toc();
    dg::blas1::axpby( 1.,x,-1., solution, error);
    double err = dg::blas2::dot( vol3d, error);
    const double norm = dg::blas2::dot( vol3d, solution);
    std::cout << sqrt( err/norm) << "\t";
    dg::DVec gyy = g2d.g_xx(), gxx=g2d.g_yy(), vol = g2d.vol();
    dg::blas1::transform( gxx, gxx, dg::SQRT<double>());
    dg::blas1::transform( gyy, gyy, dg::SQRT<double>());
    dg::blas1::pointwiseDot( gxx, vol, gxx);
    dg::blas1::pointwiseDot( gyy, vol, gyy);
    dg::blas1::scal( gxx, g2d.hx());
    dg::blas1::scal( gyy, g2d.hy());
    std::cout << *thrust::max_element( gxx.begin(), gxx.end()) << "\t";
    std::cout << *thrust::max_element( gyy.begin(), gyy.end()) << "\t";
    std::cout<<t.diff()/(double)number<<"s"<<std::endl;

    dg::blas1::transfer( error, X );
    ncerr = nc_put_var_double( ncid, psiID, X.data());
    dg::blas1::transfer( x, X );
    ncerr = nc_put_var_double( ncid, functionID, X.data());
    dg::blas1::transfer( solution, X );
    ncerr = nc_put_var_double( ncid, function2ID, X.data());
    ncerr = nc_close( ncid);


    return 0;
}
