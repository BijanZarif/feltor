#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "timer.cuh"
#include "evaluation.cuh"
#include "operators.cuh"
#include "blas.h"


double function( double x)
{
    return exp(x);
}


const unsigned n = 3;
const unsigned N = 1e5;

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;
typedef dg::ArrVec1d< double, n, HVec>  HArrVec;
typedef dg::ArrVec1d< double, n, DVec>  DArrVec;


using namespace std;
int main()
{
    cout << "Array size is:             "<< n<<endl;
    cout << "Number of intervals is:    "<< N <<endl;
    double h = 1./(double)N;
    dg::Timer t;

    t.tic();
    HArrVec h_v = dg::expand< double(&) (double), n>( function, 0, 1, N);
    t.toc(); 
    cout << "Expansion on host took         "<< t.diff()<<"s\n";

    t.tic();
    DArrVec d_v( h_v.data());
    t.toc(); 
    cout << "Copy of data host2device took  "<< t.diff()<<"s\n\n";
    t.tic();
    dg::blas2::symv(  dg::S1D<double, n>(h), d_v.data(), d_v.data());
    t.toc(); 
    cout << "symv took on device           "<< t.diff()<<"s\n";
    t.tic();
    dg::blas2::symv(  dg::S1D<double, n>(h), h_v.data(), h_v.data());
    t.toc(); 
    cout << "symv took on host             "<< t.diff()<<"s\n";

    double norm;
    t.tic();
    norm = dg::blas2::dot( dg::T1D<double, n>(h), d_v.data());
    t.toc(); 
    cout << "ddot(v,v) took on device       "<< t.diff()<<"s\n";

    t.tic();
    norm = dg::blas2::dot( dg::T1D<double, n>(h), h_v.data());
    t.toc(); 
    cout << "ddot(v,v) took on host         "<< t.diff()<<"s\n\n";
    cout<< "Square normalized norm "<< norm <<"\n";
    double solution = (exp(2.) -exp(0))/2.;
    cout << "Correct square norm of exp(x) is "<<solution<<endl;
    return 0;
}
