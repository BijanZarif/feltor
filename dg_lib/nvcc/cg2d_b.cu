#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "timer.cuh"
#include "evaluation.cuh"
#include "cg.cuh"
#include "arrvec2d.cuh"
#include "laplace.cuh"
#include "tensor.cuh"
#include "preconditioner.cuh"

const unsigned n = 3; //global relative error in L2 norm is O(h^P)

const unsigned Nx = 20;  //more N means less iterations for same error
const unsigned Ny = 20;  //more N means less iterations for same error
const double lx = 2.*M_PI;
const double ly = 2.*M_PI;

const double eps = 1e-9; //# of pcg iterations increases very much if 
 // eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;
typedef dg::ArrVec2d< double, n, HVec>  HArrVec;
typedef dg::ArrVec2d< double, n, DVec>  DArrVec;

typedef dg::T2D<double, n> Preconditioner;
typedef dg::S2D<double, n> Postconditioner;

typedef cusp::ell_matrix<int, double, cusp::host_memory> HMatrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> DMatrix;

double fct(double x, double y){ return sin(y)*sin(x);}
double laplace_fct( double x, double y) { return 2*sin(y)*sin(x);}
double initial( double x, double y) {return sin(0);}

using namespace std;

int main()
{
    dg::Timer t;
    const double hx = lx/(double)Nx;
    const double hy = ly/(double)Ny;
    cout<<"Expand initial condition\n";
    HArrVec x = dg::expand<double (&)(double, double), n> ( initial, 0,lx, 0, ly, Nx, Ny);

    cout << "Create Laplacian\n";
    t.tic();
    DMatrix A = dg::dgtensor<double, n>( dg::create::laplace1d_per<double, n>( Ny, hy), 
                               dg::S1D<double, n>( hx),
                               dg::S1D<double, n>( hy),
                               dg::create::laplace1d_per<double, n>( Nx, hx)); //dir does also work but is slow
    t.toc();
    cout<< "Creation took "<<t.diff()<<"s\n";
    //create conjugate gradient
    dg::CG<DMatrix, DVec, Preconditioner > pcg( x.data(), n*n*Nx*Ny);
    dg::CG<HMatrix, HVec, Preconditioner > pcg_host( x.data(), n*n*Nx*Ny);
    cout<<"Expand right hand side\n";
    const HArrVec solution = dg::expand<double (&)(double, double), n> ( fct, 0,lx, 0,ly, Nx, Ny);
    HArrVec b = dg::expand<double (&)(double, double), n> ( laplace_fct, 0,lx, 0,ly, Nx, Ny);
    //compute S b
    dg::blas2::symv( Postconditioner(hx, hy), b.data(), b.data());
    hipDeviceSynchronize();

    //copy data to device memory
    const DArrVec dsolution( solution);
    DArrVec db( b), dx( x);
    //////////////////////////////////////////////////////////////////////
    cout << "# of polynomial coefficients: "<< n <<endl;
    cout << "# of 2d cells                 "<< Nx*Ny <<endl;
    
    t.tic();
    cout << "Number of pcg iterations "<< pcg( A, dx.data(), db.data(), Preconditioner(hx, hy), eps)<<endl;
    t.toc();
    cout << "... for a precision of "<< eps<<endl;
    cout << "... on the device took "<< t.diff()<<"s\n";
    t.tic();
    cout << "Number of pcg iterations "<< pcg_host( A, x.data(), b.data(), Preconditioner(hx, hy), eps)<<endl;
    t.toc();
    cout << "... for a precision of "<< eps<<endl;
    cout << "... on the host took   "<< t.diff()<<"s\n";
    //compute error
    DArrVec derror( dsolution);
    dg::blas1::axpby( 1.,dx.data(),-1.,derror.data());

    double normerr = dg::blas2::dot( Postconditioner(hx, hy), derror.data());
    cout << "L2 Norm2 of Error is           " << normerr << endl;
    double norm = dg::blas2::dot( Postconditioner(hx, hy), dsolution.data());
    cout << "L2 Norm2 of Solution is        " << norm << endl;
    cout << "L2 Norm of relative error is   " <<sqrt( normerr/norm)<<endl;
    //Fehler der Integration des Sinus ist vernachlässigbar (vgl. evaluation_t)



    return 0;
}
