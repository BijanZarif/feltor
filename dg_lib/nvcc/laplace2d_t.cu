#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/print.h>
#include <cusp/ell_matrix.h>

#include "laplace.cuh"
#include "laplace2d.cuh"
#include "dgvec.cuh"
#include "dgmat.cuh"
#include "evaluation.cuh"
#include "preconditioner.cuh"
#include "blas.h"


const unsigned n = 3;
const unsigned Nx = 10; //minimum 3
const unsigned Ny = 10; //minimum 3

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;

using namespace dg;
using namespace std;

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;
typedef dg::ArrVec1d< double, n, HVec>  HArrVec;
typedef dg::ArrVec1d< double, n, DVec>  DArrVec;
typedef dg::ArrVec2d< double, n, HVec>  HArrMat;
typedef dg::ArrVec2d< double, n, DVec>  DArrMat;

typedef cusp::ell_matrix<int, double, cusp::host_memory> HMatrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> DMatrix;

double function( double x, double y) { return sin(y);}
double function( double x) { return sin(x);}

int main()
{
    cout<< "# of polynomial coeff per dim: "<<n<<"\n";
    cout<< "# of cells in x: "<<Nx<<"\n";
    cout<< "# of cells in y: "<<Ny<<"\n";
    const double hx = lx/(double)Nx;
    const double hy = ly/(double)Ny;
    HArrMat hv2d = expand< double(&)(double, double), n>( function, 0, lx, 0, ly, Nx, Ny), hw2d( hv2d);
    HArrVec hv1d = expand< double(&)(double), n>( function, 0, lx, Nx), hw1d( hv1d);
    cout << "Before multiplication: \n";
    double norm2 = blas2::dot( S1D<double, n>( hx), hw1d.data());
    cout << "Norm2 1D is : "<<norm2<<endl;
    cout << "yields in 2D: "<< norm2*ly<<endl;
    double norm2_= blas2::dot( S2D<double, n>( hx, hy), hw2d.data());
    cout << "Norm2 2D is : "<<norm2_<<endl;

    HMatrix laplace1d = create::laplace1d_per<n>(Nx, hx);
    HMatrix laplace2d = create::tensor<n>(create::laplace1d_per<n>( Ny, hy),
                                          create::laplace1d_per<n>( Nx, hx));
    HMatrix laplace2d_= create::laplace2d_per<n>(Nx, Ny,hx, hy);
    blas2::symv( laplace1d, hv1d.data(), hw1d.data() );
    blas2::symv( laplace2d, hv2d.data(), hw2d.data() );

    //cout << "hw1d: \n"<<hw1d<<endl;
    //cout << "hw2d: \n"<<hw2d<<endl;
    blas2::symv( laplace2d_, hv2d.data(), hw2d.data() );
    //cout << "hw2d_: \n"<<hw2d<<endl;

    cout << "After multiplication: \n";
    norm2 = blas2::dot( S1D<double, n>( hx), hw1d.data());
    cout << "Norm2 1D is : "<<norm2<<endl;
    cout << "yields in 2D: "<< norm2*ly<<endl;
    norm2_= blas2::dot( S2D<double, n>( hx, hy), hw2d.data());
    cout << "Norm2 2D is : "<<norm2_<<endl;

    return 0;
}

