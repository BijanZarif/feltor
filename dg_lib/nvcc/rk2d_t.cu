#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "preconditioner2d.cuh"
#include "evaluation.cuh"
#include "arakawa.cuh"
#include "rk.cuh"

//#include "cg.cuh"
//#include "laplace.cuh"
//#include "tensor.cuh"

using namespace std;
using namespace dg;

const unsigned n = 1;
const unsigned Nx = 40;
const unsigned Ny = 40;
const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
const double hx = lx/(double)Nx;
const double hy = ly/(double)Ny;

const unsigned k = 3;
const double T = 1.;
const unsigned NT = 2*T/0.01/hx; //respect courant condition
const double dt = T/(double)NT;

typedef thrust::device_vector<double> DVec;
//typedef thrust::host_vector<double> DVec;
typedef ArrVec2d<double, n, DVec > DArrVec;
typedef cusp::device_memory MemorySpace;

double initial( double x, double y) { return sin(x)*sin(y); }
double function( double x, double y){ return sin(y); }
double result( double x, double y)  { return initial( x-cos(y)*T, y); }
double arak   ( double x, double y) { return -cos(y)*sin(y)*cos(x); }



template< class Vector_Type, class MemorySpace>
struct RHS
{
    typedef Vector_Type Vector;
    RHS(): arakawa( Nx, Ny, hx, hy), phi( expand<double(&)(double, double), n>( function, 0, lx, 0, ly, Nx, Ny))
    {
        //typedef cusp::ell_matrix<int, double, MemorySpace> Matrix;
        //CG<Matrix, Vector_Type, T2D<double,n> > pcg( phi.data(), n*n*Nx*Ny);
        //Matrix A = dg::dgtensor<double, n>( 
        //                       dg::create::laplace1d_per<double, n>( Ny, hy), 
        //                       dg::S1D<double, n>( hx),
        //                       dg::S1D<double, n>( hy),
        //                       dg::create::laplace1d_per<double, n>( Nx, hx)); 
        //ArrVec2d<double, n, Vector> trick(phi);
        //blas2::symv( S2D<double, n>(hx,hy), phi.data(), trick.data());
        //cout << "Number of pcg iterations "<< pcg( A, phi.data(), trick.data(), T2D<double, n>(hx, hy), 1e-10)<<endl;

        //a = expand<double(&)(double, double), n>( arak, 0, lx, 0, ly, Nx, Ny);
        //cout << "phi \n" << phi<< endl;
    }
    void operator()( const Vector& y, Vector& yp)
    {
        ArrVec2d_View<double,n, const Vector> y_view( y, Nx), yp_view( yp, Nx);
        //cout << "Y \n"<<y_view;
        arakawa( phi.data(), y, yp);
        //cout << "YP \n"<<yp_view;
        //cout << "Norm "<< dg::blas2::dot( dg::S2D<double,n >( hx, hy), yp) << endl;
        //cout << "A \n" << a ;
        //cout << "Norm "<< dg::blas2::dot( dg::S2D<double,n >( hx, hy), a.data())<<endl;
        //double x;
        //cin >>  x;
    }
  private:
    Arakawa<double, n, Vector, MemorySpace> arakawa;
    ArrVec2d<double, n, Vector> phi;
};

int main()
{
    cout << "# of 2d cells                     " << Nx*Ny <<endl;
    cout << "# of Legendre nodes per dimension "<< n <<endl;
    cout << "# of timesteps                    "<< NT <<endl;
    cout <<fixed<< setprecision(2)<<endl;
    DArrVec init = expand< double(&)(double, double), n> ( initial, 0, lx, 0, ly, Nx, Ny), step(init);
    Arakawa<double, n, DVec, MemorySpace>( Nx, Ny, hx, hy, init.data());
    const DArrVec solution = expand< double(&)(double, double), n> ( result, 0, lx, 0, ly, Nx, Ny);
    
    RHS<DVec, MemorySpace> rhs;
    RK<3, RHS<DVec, MemorySpace> >  rk( init.data());
    for( unsigned i=0; i<NT; i++)
    {
        rk( rhs, init.data(), step.data(), dt);
        init = step;
    }

    blas1::axpby( 1., solution.data(), -1., init.data());
    hipDeviceSynchronize();
    cout << scientific;
    cout << "Norm of error is "<<blas2::dot( S2D<double, n>(hx, hy), init.data())<<"\n";
    //n = 1 -> p = 4 ?? weird (should be 2)
    //n = 2 -> p = 2 (is error dominated by error for dx(phi)?
    //n = 3 -> p = 6 
    //n = 4 -> p = 5.5
    //n = 5 -> p = 10 !!! ( is this because of "too good" functions??)


    return 0;
}
