#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "evaluation.cuh"
#include "cg.cuh"
#include "arrvec2d.cuh"
#include "laplace.cuh"
#include "tensor.cuh"
#include "preconditioner.cuh"

const unsigned n = 3; //global relative error in L2 norm is O(h^P)

const unsigned Nx = 20;  //more N means less iterations for same error
const unsigned Ny = 20;  //more N means less iterations for same error
const double lx = 2.*M_PI;
const double ly = 2.*M_PI;

const double eps = 1e-9; //# of pcg iterations increases very much if 
 // eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;
typedef dg::ArrVec2d< double, n, HVec>  HArrVec;
typedef dg::ArrVec2d< double, n, DVec>  DArrVec;

typedef dg::T2D<double, n> Preconditioner;
typedef dg::S2D<double, n> Postconditioner;

typedef cusp::ell_matrix<int, double, cusp::host_memory> HMatrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> DMatrix;

double fct(double x, double y){ return sin(y)*sin(x);}
double laplace_fct( double x, double y) { return 2*sin(y)*sin(x);}
double initial( double x, double y) {return sin(0);}
using namespace std;
int main()
{
    const double hx = lx/(double)Nx;
    const double hy = ly/(double)Ny;
    cout<<"Expand initial condition\n";
    HArrVec x = dg::expand<double (&)(double, double), n> ( initial, 0,lx, 0, ly, Nx, Ny);

    cout << "Create Laplacian\n";
    DMatrix A = dg::dgtensor<double, n>( 
                               dg::create::laplace1d_per<double, n>( Ny, hy), 
                               dg::S1D<double, n>( hx),
                               dg::S1D<double, n>( hy),
                               dg::create::laplace1d_per<double, n>( Nx, hx)); 
    dg::CG<DMatrix, DVec, Preconditioner > pcg( x.data(), n*n*Nx*Ny);
    //dg::CG<DMatrix, DVec> cg( x.data(), n*N);
    cout<<"Expand right hand side\n";
    HArrVec b = dg::expand<double (&)(double, double), n> ( laplace_fct, 0,lx, 0,ly, Nx, Ny);
    const HArrVec solution = dg::expand<double (&)(double, double), n> ( fct, 0,lx, 0,ly, Nx, Ny);

    //copy data to device memory
    const DArrVec dsolution( solution);
    DArrVec db( b), dx( x);
    //////////////////////////////////////////////////////////////////////
    cout << "# of polynomial coefficients: "<< n <<endl;
    cout << "# of 2d cells                 "<< Nx*Ny <<endl;
    //compute S b
    dg::blas2::symv( Postconditioner(hx, hy), db.data(), db.data());
    hipDeviceSynchronize();
    cout << "Number of pcg iterations "<< pcg( A, dx.data(), db.data(), Preconditioner(hx, hy), eps)<<endl;
    hipDeviceSynchronize();
    //std::cout << "Number of cg iterations "<< cg( A, dx.data(), db.data(), dg::Identity<double>(), eps)<<endl;
    cout << "For a precision of "<< eps<<endl;
    //compute error
    DArrVec derror( dsolution);
    dg::blas1::axpby( 1.,dx.data(),-1.,derror.data());

    DArrVec dAx(dx), res( db);
    dg::blas2::symv(  A, dx.data(), dAx.data());
    dg::blas1::axpby( 1.,dAx.data(),-1.,res.data());
    hipDeviceSynchronize();

    double xnorm = dg::blas2::dot( Postconditioner(hx, hy), dx.data());
    cout << "L2 Norm2 of x0 is              " << xnorm << endl;
    double eps = dg::blas2::dot( Postconditioner(hx, hy), derror.data());
    cout << "L2 Norm2 of Error is           " << eps << endl;
    double norm = dg::blas2::dot( Postconditioner(hx, hy), dsolution.data());
    cout << "L2 Norm2 of Solution is        " << norm << endl;
    double normres = dg::blas2::dot( Postconditioner(hx, hy), res.data());
    cout << "L2 Norm2 of Residuum is        " << normres << endl;
    cout << "L2 Norm of relative error is   " <<sqrt( eps/norm)<<endl;
    //Fehler der Integration des Sinus ist vernachlässigbar (vgl. evaluation_t)



    return 0;
}
