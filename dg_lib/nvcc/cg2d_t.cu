#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "evaluation.cuh"
#include "cg.cuh"
#include "dgmat.cuh"
#include "laplace2d.cuh"
#include "preconditioner.cuh"

const unsigned n = 2; //global relative error in L2 norm is O(h^P)

const unsigned Nx = 100;  //more N means less iterations for same error
const unsigned Ny = 100;  //more N means less iterations for same error
const double lx = 2*M_PI;
const double ly = 2*M_PI;

const double eps = 1e-7; //# of pcg iterations increases very much if 
 // eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;
typedef dg::ArrVec2d< double, n, HVec>  HArrVec;
typedef dg::ArrVec2d< double, n, DVec>  DArrVec;

typedef dg::T2D<double, n> Preconditioner;
typedef dg::S2D<double, n> Postconditioner;

typedef cusp::ell_matrix<int, double, cusp::host_memory> HMatrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> DMatrix;

double sine(double x, double y){ return sin( x)*sin(y);}
double initial( double x, double y) {return sin(0);}
using namespace std;
int main()
{
    const double hx = lx/(double)Nx;
    const double hy = ly/(double)Ny;
    HArrVec x = dg::expand<double (&)(double, double), n> ( initial, 0,lx, 0, ly, Nx, Ny);
    DMatrix A = dg::create::laplace2d_per<n>( Nx, Ny, hx, hy); 
    dg::CG<DMatrix, DVec, Preconditioner > pcg( x.data(), n*n*Nx*Ny);
    //dg::CG<DMatrix, DVec> cg( x.data(), n*N);
    HArrVec b = dg::expand<double (&)(double, double), n> ( sine, 0,lx, 0,ly, Nx, Ny);
    const HArrVec solution(b);

    //copy data to device memory
    const DArrVec dsolution( solution.data(), Nx);
    DArrVec db( b.data(), Nx), dx( db);

    cout << "# of polynomial coefficients: "<< n <<endl;
    cout << "# of 2d cells                 "<< Nx*Ny <<endl;
    //compute S b
    dg::blas2::symv( Postconditioner(hx, hy), db.data(), db.data());
    hipDeviceSynchronize();
    std::cout << "Number of pcg iterations "<< pcg( A, dx.data(), db.data(), Preconditioner(hx, hy), eps)<<endl;
    //std::cout << "Number of cg iterations "<< cg( A, dx.data(), db.data(), dg::Identity<double>(), eps)<<endl;
    cout << "For a precision of "<< eps<<endl;
    //compute error
    DArrVec derror( dsolution.data(), Nx);
    dg::blas1::axpby( 1.,dx.data(),-1.,derror.data());
    hipDeviceSynchronize();

    double eps = dg::blas2::dot( Postconditioner(hx, hy), derror.data());
    cout << "L2 Norm2 of Error is " << eps << endl;
    double norm = dg::blas2::dot( Postconditioner(hx, hy), dsolution.data());
    cout << "L2 Norm2 of Solution is " << norm << endl;
    std::cout << "L2 Norm of relative error is "<<sqrt( eps/norm)<<std::endl;
    //Fehler der Integration des Sinus ist vernachlässigbar (vgl. evaluation_t)



    return 0;
}
