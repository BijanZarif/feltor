#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <thrust/scatter.h>
#include <thrust/host_vector.h>

#include "cuda_texture.cuh"

#include "arrvec2d.cuh"
#include "evaluation.cuh"
#include "functions.h"
#include "functors.cuh"
#include "toefl.cuh"
#include "rk.cuh"


using namespace std;
using namespace dg;

const unsigned n = 3;
const unsigned Nx = 20;
const unsigned Ny = 10;
const double lx = 2.;
const double ly = 1.;

const double Pr = 10;
const double Ra = 5e5;

const unsigned k = 2;
const double dt = 1e-6;

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;
typedef dg::ArrVec2d< double, n, HVec>  HArrVec;
typedef dg::ArrVec2d< double, n, DVec>  DArrVec;
typedef cusp::ell_matrix<int, double, cusp::host_memory> HMatrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> DMatrix;

using namespace std;

double groundState( double x, double y) { return ly/2. - y;}

int main()
{
    dg::Window w(800, 400);
    glfwSetWindowTitle( "Behold the convection\n");

    const double hx = lx/ (double)Nx;
    const double hy = ly/ (double)Ny;

    /////////////////////////////////////////////////////////////////////////
    cout << "# of Legendre coefficients: " << n<<endl;
    cout << "# of grid cells:            " << Nx*Ny<<endl;
    cout << "Timestep                    " << dt << endl;

    //create initial vector
    dg::Gaussian g( lx/2., ly/2., .1, .1, 1);
    DArrVec theta = dg::expand<dg::Gaussian, n> ( g, 0.,lx, 0., ly, Nx, Ny);
    vector<DVec> y0(2, theta.data()), y1(y0);
    y0[1] = DVec( n*n*Nx*Ny, 0.); //omega is zero

    //create RHS and RK
    Toefl<double, n, DVec, cusp::device_memory> test( Nx, Ny, hx, hy, Ra, Pr, 1e-6); 
    RK< k, Toefl<double, n, DVec, cusp::device_memory> > rk( y0);

    //create equidistant backward transformation
    dg::Operator<double, n> backwardeq( dg::DLT<n>::backwardEQ);
    dg::Operator<double, n*n> backward2d = dg::tensor( backwardeq, backwardeq);
    HMatrix hbackward = dg::tensor( Nx*Ny, backward2d);
    DMatrix backward = hbackward;

    //create visualisation vectors
    int running = GL_TRUE;
    DVec visual( n*n*Nx*Ny);
    thrust::device_vector<int> map = dg::makePermutationMap<n>( Nx, Ny);
    DArrVec ground = expand< double(&)(double, double), n> ( groundState, 0, lx, 0, ly, Nx, Ny), temperature( ground);
    dg::ColorMapRedBlueExt colors( 1.);
    while (running)
    {
        //compute the total temperature
        blas1::axpby( 1., y0[0], 0., temperature.data());
        blas1::axpby( 1., ground.data(), 1., temperature.data());
        //transform field to an equidistant grid
        dg::blas2::symv( backward, temperature.data(), visual);
        thrust::scatter( visual.begin(), visual.end(), map.begin(), visual.begin());
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), -1., thrust::maximum<double>() );
        std::cout << "Color scale " << colors.scale() <<"\n";
        //draw and swap buffers
        w.draw( visual, n*Nx, n*Ny, colors);
        //step 
        rk( test, y0, y1, dt);
        for( unsigned i=0; i<2; i++)
            thrust::swap( y0[i], y1[i]);
        glfwWaitEvents();
        running = !glfwGetKey( GLFW_KEY_ESC) &&
                    glfwGetWindowParam( GLFW_OPENED);
    }
    ////////////////////////////////////////////////////////////////////

    return 0;

}
