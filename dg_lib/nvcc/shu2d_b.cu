#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <thrust/remove.h>
#include <thrust/host_vector.h>

#include "timer.cuh"

#include "functors.cuh"

#include "arrvec2d.cuh"
#include "evaluation.cuh"
#include "shu.cuh"
#include "rk.cuh"



using namespace std;
using namespace dg;

const unsigned n = 3;
const unsigned Nx = 33;
const unsigned Ny = 33;
const double lx = 1.;
const double ly = 1.;

const unsigned k = 3;
const double D = 0.0;
const double U = 1; //the dipole doesn't move with this velocity because box is not infinite
const double R = 0.2*lx;
const double T = 0.1;
const unsigned NT = (unsigned)(T*n*Nx/0.05/lx);
const double eps = 1e-3; //CG method

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;
typedef dg::ArrVec2d< double, n, HVec>  HArrVec;
typedef dg::ArrVec2d< double, n, DVec>  DArrVec;

typedef cusp::ell_matrix<int, double, cusp::host_memory> HMatrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> DMatrix;

typedef cusp::device_memory Memory;

using namespace std;

int main()
{
    Timer t;
    const double hx = lx/ (double)Nx;
    const double hy = ly/ (double)Ny;
    const double dt = T/(double)NT;
    ////////////////////////////////////////////////////////////
    cout << "# of Legendre coefficients: " << n<<endl;
    cout << "# of grid cells:            " << Nx*Ny<<endl;
    cout << "Timestep                    " << dt << endl;
    cout << "Diffusion                   " << D <<endl;
    dg::Lamb lamb( 0.5*lx, 0.5*ly, R, U);
    HArrVec omega = expand< dg::Lamb, n> ( lamb, 0, lx, 0, ly, Nx, Ny);
    DArrVec stencil = expand< double(&)(double, double), n> ( one, 0, lx, 0, ly, Nx, Ny);
    dg::Lamb lamb2( 0.5*lx, 0.5*ly-0.9755*U*T, R, U);
    HArrVec solh = expand< dg::Lamb, n> ( lamb2, 0, lx, 0, ly, Nx, Ny);
    DVec sol = solh.data();
    DVec y0( omega.data()), y1( y0);
    //make solver and stepper
    Shu<double, n, DVec, Memory> test( Nx, Ny, hx, hy, D, eps);
    RK< k, Shu<double, n, DVec, Memory> > rk( y0);

    t.tic();
    test( y0, y1);
    t.toc();
    cout << "Time for one rhs evaluation: "<<t.diff()<<"s\n";
    double vorticity = blas2::dot( stencil.data(), S2D<double, n>(hx, hy), y0);
    double enstrophy = 0.5*blas2::dot( y0, S2D<double, n>(hx, hy), y0);
    double energy =    0.5*blas2::dot( y0, S2D<double, n>(hx, hy), test.potential()) ;

    double time = 0;
    while( time < T)
    {
        //step 
        rk( test, y0, y1, dt);
        thrust::swap(y0, y1);
        time += dt;
    }
    ////////////////////////////////////////////////////////////////////
    cout << "Analytic formula enstrophy "<<lamb.enstrophy()<<endl;
    cout << "Analytic formula energy    "<<lamb.energy()<<endl;
    cout << "Total vorticity           is: "<<blas2::dot( stencil.data(), S2D<double, n>(hx, hy), y0) << "\n";
    cout << "Relative enstrophy error  is: "<<(0.5*blas2::dot( S2D<double, n>(hx, hy), y0) - enstrophy)/enstrophy<<"\n";
    test( y0, y1); //get the potential ready
    cout << "Relative energy error     is: "<<(0.5*blas2::dot( test.potential(), S2D<double, n>(hx, hy), y0) - energy)/energy<<"\n";

    //energy and enstrophy errrors are due to timestep only ( vorticity is exactly conserved)
    // k = 2 | p = 3
    // k = 3 | p = 4
    // k = 4 | p = 5

    return 0;

}
