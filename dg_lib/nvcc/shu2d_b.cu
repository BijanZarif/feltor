#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <thrust/remove.h>
#include <thrust/host_vector.h>

#include "timer.cuh"

#include "functors.cuh"

#include "arrvec2d.cuh"
#include "evaluation.cuh"
#include "shu.cuh"
#include "rk.cuh"



using namespace std;
using namespace dg;

const unsigned n = 3;
const double lx = 2.*M_PI;
const double ly = 2.*M_PI;

const unsigned k = 3;
const double U = 1.; //the dipole doesn't move with this velocity because box is not infinite
const double R = 0.2*lx;
const double T = 2.;
const double eps = 1e-6; //CG method

typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;
typedef dg::ArrVec2d< double, n, HVec>  HArrVec;
typedef dg::ArrVec2d< double, n, DVec>  DArrVec;

typedef cusp::ell_matrix<int, double, cusp::host_memory> HMatrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> DMatrix;


double D = 0.0;
unsigned Nx = 16;
unsigned Ny = 16;

double initial( double x, double y){ return 2.*sin(x)*sin(y);}
double solution( double x, double y){ return 2.*sin(x)*sin(y)*exp(-2.*D*T);}
using namespace std;

int main()
{
    Timer t;
    ////////////////////////////////////////////////////////////
    cout << "Type # of grid cells in one dimension!\n";
    cin >> Nx;
    Ny = Nx; 
    cout << "Type diffusion constant!\n";
    cin >> D;
    cout << "# of Legendre coefficients: " << n<<endl;
    cout << "# of grid cells:            " << Nx*Ny<<endl;
    cout << "Diffusion                   " << D <<endl;
    ////////////////////////////////////////////////////////////

    const double hx = lx/ (double)Nx;
    const double hy = ly/ (double)Ny;
    unsigned NT = (unsigned)(T*n*Nx/0.025/lx);
    cout << "Type # of timesteps\n";
    cin >> NT;
    const double dt = T/(double)NT;
    cout << "Runge Kutta stages          " << k <<endl;
    cout << "Timestep                    " << dt << endl;
    cout << "# of steps                  " << NT <<endl;
    ////////////////////////////////////////////////////////////

    DArrVec stencil = expand< double(&)(double, double), n> ( one, 0, lx, 0, ly, Nx, Ny);
    //dg::Lamb lamb( 0.5*lx, 0.5*ly, R, U);
    //HArrVec omega = expand< dg::Lamb, n> ( lamb, 0, lx, 0, ly, Nx, Ny);
    HArrVec omega = expand< double(&)(double, double), n> ( initial, 0, lx, 0, ly, Nx, Ny);

    //dg::Lamb lamb2( 0.5*lx, 0.5*ly-0.9755*U*T, R, U);
    //HArrVec solh = expand< dg::Lamb, n> ( lamb2, 0, lx, 0, ly, Nx, Ny);
    HArrVec solh = expand< double(&)(double, double), n> ( solution, 0, lx, 0, ly, Nx, Ny);

    DVec sol = solh.data();
    DVec y0( omega.data()), y1( y0);
    //make solver and stepper
    Shu<double, n, DVec> test( Nx, Ny, hx, hy, D, eps);
    RK< k, Shu<double, n, DVec> > rk( y0);
    AB< k, Shu<double, n, DVec> > ab( y0);

    t.tic();
    test( y0, y1);
    t.toc();
    cout << "Time for one rhs evaluation: "<<t.diff()<<"s\n";
    double vorticity = blas2::dot( stencil.data(), S2D<double, n>(hx, hy), y0);
    double enstrophy = 0.5*blas2::dot( y0, S2D<double, n>(hx, hy), y0);
    double energy =    0.5*blas2::dot( y0, S2D<double, n>(hx, hy), test.potential()) ;

    double time = 0;
    ab.init( test, y0, dt);
    while( time < T)
    {
        //step 
        ab( test, y0, y1, dt);
        thrust::swap(y0, y1);
        time += dt;
    }
    ////////////////////////////////////////////////////////////////////
    //cout << "Analytic formula enstrophy "<<lamb.enstrophy()<<endl;
    //cout << "Analytic formula energy    "<<lamb.energy()<<endl;
    cout << "Total vorticity           is: "<<blas2::dot( stencil.data(), S2D<double, n>(hx, hy), y0) << "\n";
    cout << "Relative enstrophy error  is: "<<(0.5*blas2::dot( S2D<double, n>(hx, hy), y0) - enstrophy)/enstrophy<<"\n";
    test( y0, y1); //get the potential ready
    cout << "Relative energy error     is: "<<(0.5*blas2::dot( test.potential(), S2D<double, n>(hx, hy), y0) - energy)/energy<<"\n";

    blas1::axpby( 1., sol, -1., y0);
    cout << "Distance to solution "<<sqrt( blas2::dot( S2D<double,n >(hx,hy), y0)) << endl;

    //energy and enstrophy errrors are due to timestep only ( vorticity is exactly conserved)
    // k = 2 | p = 3
    // k = 3 | p = 4
    // k = 4 | p = 5

    //solution to sin(x)sin(y) 
    // n = 1 
    // n = 2 | p = 2
    // n = 3 | p = 2.6
    // n = 4 | p = 4

    return 0;

}
