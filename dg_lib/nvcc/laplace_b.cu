#include <iostream>

#include <cusp/print.h>
#include <cusp/ell_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/dia_matrix.h>
#include <cusp/csr_matrix.h>

#include "../gcc/timer.h"
#include "laplace.cuh"
#include "laplace2d.cuh"
#include "dgvec.cuh"
#include "dgmat.cuh"
#include "blas.h"

const unsigned P = 3;
const unsigned N = 3e4;
const unsigned Nx = 1e2;
const unsigned Ny = 1e2;

using namespace dg;
using namespace std;
typedef thrust::device_vector< double>   DVec;
typedef thrust::host_vector< double>     HVec;

//ell and hyb matrices are fastest for 1d transforms
typedef cusp::ell_matrix<int, double, cusp::host_memory> HMatrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> DMatrix;

int main()
{
    Timer t;
    cout << "# of polynomial coefficients P is: "<< P <<endl;
    cout << "# of 1d intervals is:  "<<N<<"\n";
    cout << "# of 2d cells is:      "<<Nx*Ny<<"\n";
    ArrVec1d<double, P> hv( N,  1);
    for( unsigned k=0; k<N; k++)
        for( unsigned i=0; i<P; i++)
            hv( k, i) = i;

    ArrVec2d<double, P> hw2d( Nx, Ny, 0.);
    DVec dv = hv.data(), dw( dv);
    DVec dw2d = hw2d.data(), dv2d( dw2d);
    DMatrix laplace1d = create::laplace1d_per<P>( N, 2.);
    DMatrix laplace2d = create::tensorSum<P>( create::laplace1d_per<P>(Ny, 2.),
                                              S1D<double, P>( 2.),
                                              S1D<double, P>( 2.),
                                              create::laplace1d_per<P>(Nx, 2.) );
    t.tic();
    blas2::symv( laplace1d, dv, dw);
    hipDeviceSynchronize();
    t.toc();
    cout << "Multiplication with laplace1d took "<<t.diff()<<"s\n";
    t.tic();
    blas2::symv( laplace2d, dv2d, dw2d);
    hipDeviceSynchronize();
    t.toc();
    cout << "Multiplication with laplace2d took "<<t.diff()<<"s\n";
    return 0;
}

