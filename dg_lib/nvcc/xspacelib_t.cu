#include "hip/hip_runtime.h"
#include <iostream>

#include "xspacelib.cuh"

const unsigned n=3;


double sine( double x, double y){ return sin(x);}

int main()
{
    const dg::Grid<double, n> grid( 0, 2.*M_PI, 0, 2.*M_PI, 10, 10, dg::DIR, dg::PER);

    dg::HVec x = dg::evaluate( sine, grid);
    dg::ArakawaX<double, n, dg::DVec> arakawa( grid);

    dg::Polarisation2dX<double, n, dg::DVec> polarisation ( grid);
    return 0;
}
