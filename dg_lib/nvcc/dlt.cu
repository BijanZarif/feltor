#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cusp/ell_matrix.h>
#include <cusp/dia_matrix.h>

#include "blas.h"
#include "laplace.cuh"
#include "timer.cuh"
#include "array.cuh"
#include "dlt.h"
#include "dgvec.cuh"
#include "evaluation.cuh"
#include "operators.cuh"


using namespace std;
using namespace dg;

const unsigned n = 3; //thrust transform is always faster
const unsigned N = 1e5;

typedef thrust::device_vector<double>   DVec;
typedef thrust::host_vector<double>     HVec;

typedef thrust::device_vector< Array<double, n> > DArrVec;
typedef thrust::host_vector< Array<double, n> >   HArrVec;

typedef ArrVec1d< double, n, HVec> HArrVec_;
typedef ArrVec1d< double, n, DVec> DArrVec_;
typedef cusp::ell_matrix<int, double, cusp::host_memory>   HMatrix;
typedef cusp::ell_matrix<int, double, cusp::device_memory> DMatrix;

template< size_t n>
struct Forward{
    Forward(): forward( DLT<n>::forward){}
    
    __host__ __device__
    Array<double, n> operator() (const Array<double, n>& arr)
    {
        Array<double,n > temp(0.);
        for( unsigned i=0; i<n; i++)
            for( unsigned j=0; j<n; j++)
                temp[i] += forward(i,j)*arr[j];
        return temp;
               
    }
    private:
    Operator<double, n> forward;
};

template< size_t n>
cusp::coo_matrix<int, double, cusp::host_memory> createForward( unsigned N)
{
    cusp::coo_matrix<int, double, cusp::host_memory> A( n*N, n*N, n*n*N);
    Operator< double, n> forward(DLT<n>::forward);
    //std::cout << a << "\n"<<b <<std::endl;
    //assemble the matrix
    int number = 0;
    for( unsigned i=0; i<N; i++)
        for( unsigned k=0; k<n; k++)
            for( unsigned l=0; l<n; l++)
                create::detail::add_index<n>(A, number, i, i, k, l, forward(k,l));
    return A;
};

double function( double x) { return sin(x);}

void doSymv( double* ptr, thrust::input_device_iterator_tag )
{
    thrust::device_ptr<Array<double, n> > begin = thrust::device_pointer_cast( reinterpret_cast< Array< double, n>* >(ptr));
    thrust::device_ptr<Array<double, n> > end = begin + N-1;
    thrust::transform( begin, end, begin, Forward<n>());
}
void doSymv( double* ptr, thrust::input_host_iterator_tag)
{
    Array<double, n>* begin =  reinterpret_cast< Array< double, n>* >(ptr);
    Array<double, n>* end = begin + N-1;
    thrust::transform( begin, end, begin, Forward<n>());
}

template< class Vector>
void symv( Vector& v)
{
    doSymv( thrust::raw_pointer_cast(&v[0]), typename thrust::iterator_traits< typename Vector::iterator>::iterator_category());
}




int main()
{
    cout << "# of Legendre coefficients: " << n<<endl;
    cout << "# of grid cells:            " << N<<endl;
    Timer t;
    HArrVec_ hv_ = evaluate<double(&)(double), n>( function, 0, 2.*M_PI, N);
    HArrVec  hv( N);
    for( unsigned i=0; i<N; i++)
        for( unsigned j=0; j<n; j++)
            hv[i][j] = hv_(i,j);
    DArrVec  dv( hv);
    DArrVec  dv2( N);
    DArrVec_ dv_( hv_);
    DArrVec_ dv_2( N);
    DMatrix dm = createForward<n>( N);

    t.tic();
    symv( dv_.data());
    t.toc();
    cout << "Forward thrust transform took "<<t.diff()<<"s\n";
    t.tic();
    blas2::symv( dm, dv_.data(), dv_.data());
    t.toc();
    cout << "Foward cusp transform took    "<<t.diff()<<"s\n";

    //test for equality...
    /*
    hv_ = dv_;
    hv = dv;
    for( unsigned i=0; i<N; i++)
    {
        for( unsigned j=0; j<n; j++)
            cout << hv[i][j]  << " ";
        cout << "\n";
    }
    cout << endl;
    cout << hv_ <<endl;
    */

    
    return 0;
}
