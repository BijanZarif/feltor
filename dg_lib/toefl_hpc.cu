#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>

#include "hdf5.h"
#include "hdf5_hl.h"


#include "toeflR.cuh"
#include "parameters.h"
#include "rk.cuh"
#include "../lib/read_input.h"

#include "timer.cuh"

using namespace std;
using namespace dg;

const unsigned n = 3;
const unsigned k = 3;

using namespace std;

int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v;
    std::string input;
    if( argc != 3)
    {
        cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        v = toefl::read_input( argv[1]);
        input = toefl::read_file( argv[1]);
    }
    const Parameters p( v);
    if( p.n != n || p.k != k)
    {
        cerr << "ERROR: n or k doesn't match: "<<k<<" vs. "<<p.k<<" and "<<n<<" vs. "<<p.n<<"\n";
        return -1;
    }

    //set up computations
    dg::Grid<double,n > grid( 0, p.lx, 0, p.ly, p.Nx, p.Ny, p.bc_x, p.bc_y);
    //create RHS 
    dg::ToeflR<double, n, dg::DVec > test( grid, p.kappa, p.nu, p.tau, p.eps_pol, p.eps_gamma, p.global); 
    //create initial vector
    dg::Gaussian g( p.posX*grid.lx(), p.posY*grid.ly(), p.sigma, p.sigma, p.n0); 
    std::vector<dg::DVec> y0(2, dg::evaluate( g, grid)), y1(y0); // n_e' = gaussian
    blas2::symv( test.gamma(), y0[0], y0[1]); // n_e = \Gamma_i n_i -> n_i = ( 1+alphaDelta) n_e' + 1
    blas2::symv( V2D<double, n> ( grid), y0[1], y0[1]);
    if( p.global)
    {
        thrust::transform( y0[0].begin(), y0[0].end(), y0[0].begin(), dg::PLUS<double>(+1));
        thrust::transform( y0[1].begin(), y0[1].end(), y0[1].begin(), dg::PLUS<double>(+1));
        test.log( y0, y0); //transform to logarithmic values
    }
    //create timestepper
    dg::AB< k, std::vector<dg::DVec> > ab( y0);
    /////////////////////////////////////////////////////////////////////////
    //set up hdf5
    dg::HVec output( y1[0]); //intermediate transport location
    hid_t   file, grp;
    herr_t  status;
    hsize_t dims[2];
    dims[0] = n*grid.Ny(); 
    dims[1] = n*grid.Nx(); 
    file = H5Fcreate( argv[2], H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);
    std::stringstream title; 
    hsize_t size = input.size();
    status = H5LTmake_dataset_char( file, "inputfile", 1, &size, input.data()); //name should precede t so that reading is easier
    /////////////////////////////////////////////////////////////////////////
    double time = 0;
    ab.init( test, y0, p.dt);
    /////////////////////////////////////first output (with zero potential)
    if( p.global)
        test.exp( y0,y1); //transform to logarithmic values
    grp = H5Gcreate( file, "t=0", H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT  );
    //output all three fields
    output = y1[0];
    status = H5LTmake_dataset_double( grp, "electrons", 2,  dims, output.data());
    output = y1[1];
    status = H5LTmake_dataset_double( grp, "ions", 2,  dims, output.data());
    blas1::axpby( 0., output, 0., output); //set output zero as it should be
    status = H5LTmake_dataset_double( grp, "potential", 2,  dims, output.data());
    H5Gclose( grp);

    title << std::setfill('0');
    ///////////////////////////////////Timeloop////////////////////////////////
    for( unsigned i=0; i<p.maxout; i++)
    {
        for( unsigned i=0; i<p.itstp; i++)
        {
            ab( test, y0, y1, p.dt);
            y0.swap( y1);
        }
        time += p.itstp*p.dt;
        if( p.global)
            test.exp( y0,y1); //transform to logarithmic values
        
        title << "t=";
        title <<std::setw(6)<<std::right<<(unsigned)(floor(time))<<"."<<std::setw(6)<<std::left<<(unsigned)((time-floor(time))*1e6);
        std::cout << title.str()<<"\n";
        grp = H5Gcreate( file, title.str().c_str(), H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT  );

        title.str("");
        //output all three fields
        output = y1[0]; //electrons
        status = H5LTmake_dataset_double( grp, "electrons", 2,  dims, output.data());
        output = y1[1]; //ions
        status = H5LTmake_dataset_double( grp, "ions", 2,  dims, output.data());
        output = test.polarisation();
        status = H5LTmake_dataset_double( grp, "potential", 2,  dims, output.data());
        H5Gclose( grp);
    }

    //writing takes the same time as device-host transfers
    ////////////////////////////////////////////////////////////////////
    H5Fclose( file);

    return 0;

}
