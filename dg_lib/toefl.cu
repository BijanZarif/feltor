#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>

#include "draw/host_window.h"

#include "toefl.cuh"
#include "rk.cuh"
#include "../lib/read_input.h"

#include "timer.cuh"

using namespace std;
using namespace dg;

const unsigned n = 3;
const unsigned k = 3;

using namespace std;
void display( const std::vector<double>& v, std::ostream& os = std::cout )
{
    os << "Physical parameters are: \n"
        <<"    Viscosity:       = "<<v[13]<<"\n"
        <<"    Curvature_y:     = "<<v[14]<<"\n"
        <<"    Ion-temperature: = "<<v[15]<<"\n";
    char local[] = "LOCAL" , global[] = "GLOBAL";
    os  <<"Mode is:   \n"
        <<"    "<<(v[12]?global:local)<<"\n";
    char per[] = "PERIODIC", dir[] = "DIRICHLET";
    os << "Boundary parameters are: \n"
        <<"    lx = "<<v[8]<<"\n"
        <<"    ly = "<<v[9]<<"\n"
        <<"Boundary conditions in x are: \n"
        <<"    "<<(v[10] ? dir:per)<<"\n"
        <<"Boundary conditions in y are: \n"
        <<"    "<<(v[11] ? dir:per)<<"\n";
    os << "Algorithmic parameters are: \n"
        <<"    n  = "<<v[1]<<"\n"
        <<"    Nx = "<<v[2]<<"\n"
        <<"    Ny = "<<v[3]<<"\n"
        <<"    k  = "<<v[4]<<"\n"
        <<"    dt = "<<v[5]<<"\n";
    os  <<"Blob parameters are: \n"
        << "    width is:     "<<v[17]<<"\n"
        << "    amplitude is: "<<v[16]<<"\n"
        << "    posX:         "<<v[18]<<"\n"
        << "    posY:         "<<v[19]<<"\n";
    os << "Stopping for CG:         "<<v[6]<<"\n"
        <<"Stopping for Gamma CG:   "<<v[7]<<"\n";


}
int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v, v2;
    if( argc == 1)
    {
        v = toefl::read_input("input.txt");
    }
    else if( argc == 2)
    {
        v = toefl::read_input( argv[1]);
    }
    else
    {
        cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }
    v2 = toefl::read_input( "window_params.txt");
    draw::HostWindow w(v2[3], v2[4]);
    w.set_multiplot( v2[1], v2[2]);
    if( n != v[1] || k != v[4]) {
        cerr << "Order is wrong\n";
        return;
    }
    /////////////////////////////////////////////////////////////////////////
    display( v, std::cout);


    dg::bc bc_x = dg::PER, bc_y = dg::PER;
    if( v[10]) bc_x = dg::DIR;
    if( v[11]) bc_y = dg::DIR;
    unsigned Nx = v[2], Ny = v[3];
    double lx = v[8], ly = v[9];
    dg::Grid<double, n > grid( 0, lx, 0, ly, Nx, Ny, bc_x, bc_y);
    //create initial vector
    double n0 = v[16], sigma = v[17], posX = v[18], posY = v[19];
    dg::Gaussian g( posX*grid.lx(), posY*grid.ly(), sigma, sigma, n0); //gaussian width is in absolute values
    dg::DVec ne = dg::evaluate ( g, grid);
    bool global = v[12];
    if( global)
        thrust::transform( ne.begin(), ne.end(), ne.begin(), dg::PLUS<double>(1));
    std::vector<dg::DVec> y0(2, ne), y1(y0); // n_e = n_i 

    //create RHS and RK
    double eps_pol = v[6], nu = v[13], kappa = v[14];
    dg::Toefl<double, n, dg::DVec > test( grid, global, eps_pol , kappa, nu, bc_x, bc_y); 
    if( global)
        test.log( y0,y0); //transform to logarithmic values
    dg::RK< k, std::vector<dg::DVec> > rk( y0);
    dg::AB< k, std::vector<dg::DVec> > ab( y0);

    dg::DVec dvisual( grid.size());
    dg::HVec visual( grid.size());
    dg::DMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    Timer t;
    bool running = true;
    double time = 0, dt = v[5];
    unsigned itstp = v[20];
    ab.init( test, y0, dt);
    while (running)
    {
        //transform field to an equidistant grid
        /*
        if( global)
        {
            test.exp( y0, y1);
            thrust::transform( y1[0].begin(), y1[0].end(), y1[0].begin(), dg::PLUS<double>(-1));
            dg::blas2::gemv( equi, y1[0], y1[1]);
        }
        else
            dg::blas2::gemv( equi, y0[0], y1[1]);
        visual = y1[1]; //transfer to host
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw electrons
        w.title() <<"ne/ "<<colors.scale()<<"\t";
        w.draw( visual, n*v[1], n*v[2], colors, 0, 0);
        */
        //transform field to an equidistant grid
        if( global)
        {
            test.exp( y0, y1);
            thrust::transform( y1[1].begin(), y1[1].end(), y1[1].begin(), dg::PLUS<double>(-1));
            dg::blas2::gemv( equi, y1[1], y1[0]);
        }
        else
            dg::blas2::gemv( equi, y0[1], y1[0]);
        visual = y1[0]; //transfer to host
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        w.title() <<"ni/ "<<colors.scale()<<"\t";
        w.draw( visual, n*grid.Nx(), n*grid.Ny(), colors);

        //transform phi
        dg::blas2::gemv( test.laplacianM(), test.polarisation(), y1[1]);
        dg::blas2::gemv( equi, y1[1], dvisual);
        visual = dvisual; //transfer to host
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        w.title() <<"phi/ "<<colors.scale()<<"\t";
        w.title() << setprecision(2) << fixed;
        w.title() << " &&   time = "<<time;
        w.draw( visual, n*grid.Nx(), n*grid.Ny(), colors);

        //step 
        t.tic();
        for( unsigned i=0; i<itstp; i++)
        {
            ab( test, y0, y1, dt);
            y0.swap( y1); //attention on -O3 ?
            //for( unsigned i=0; i<y0.size(); i++)
            //    thrust::swap( y0[i], y1[i]);
        }
        time += (double)itstp*dt;
        t.toc();
        //glfwWaitEvents();
        running = !glfwGetKey( GLFW_KEY_ESC) &&
                    glfwGetWindowParam( GLFW_OPENED);
    }
    std::cout << "Average time for one step: "<<t.diff()/(double)itstp<<"s\n";
    ////////////////////////////////////////////////////////////////////

    return 0;

}
